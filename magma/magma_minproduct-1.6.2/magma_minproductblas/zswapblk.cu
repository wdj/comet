#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c

*/
#include "common_magma_minproduct.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
 *  Blocked version: swap several pairs of lines
 */
typedef struct {
    magma_minproductDoubleComplex *A;
    magma_minproductDoubleComplex *B;
    int n, ldda, lddb, npivots;
    short ipiv[BLOCK_SIZE];
} magma_minproductgpu_zswapblk_params_t;

__global__ void magma_minproductgpu_zswapblkrm( magma_minproductgpu_zswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        magma_minproductDoubleComplex *A = params.A + y - params.ldda;
        magma_minproductDoubleComplex *B = params.B + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A += params.ldda;
            if ( params.ipiv[i] == -1 )
                continue;
            magma_minproductDoubleComplex  tmp1 = *A;
            magma_minproductDoubleComplex *tmp2 = B + params.ipiv[i]*params.lddb;
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
}

__global__ void magma_minproductgpu_zswapblkcm( magma_minproductgpu_zswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.ldda;
    unsigned int offset2 = y*params.lddb;
    if( y < params.n )
    {
        magma_minproductDoubleComplex *A = params.A + offset1 - 1;
        magma_minproductDoubleComplex *B = params.B + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A++;
            if ( params.ipiv[i] == -1 )
                continue;
            magma_minproductDoubleComplex  tmp1 = *A;
            magma_minproductDoubleComplex *tmp2 = B + params.ipiv[i];
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
    __syncthreads();
}


/**
    @ingroup magma_minproduct_zblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_zswapblk_q(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductDoubleComplex_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductDoubleComplex_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset,
    magma_minproduct_queue_t queue )
{
    magma_minproduct_int_t  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_minproduct_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == Magma_minproductColMajor ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_zswapblk_params_t params = { dA+k, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_zswapblkcm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
    else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_zswapblk_params_t params = { dA+k*ldda, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_zswapblkrm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
}


/**
    @see magma_minproductblas_zswapblk_q
    @ingroup magma_minproduct_zblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_zswapblk(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductDoubleComplex_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductDoubleComplex_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset )
{
    magma_minproductblas_zswapblk_q(
        order, n, dA, ldda, dB, lddb, i1, i2, ipiv, inci, offset, magma_minproduct_stream );
}
