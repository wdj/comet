#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zmgecsrmv.cu normal z -> s, Sun May  3 11:22:58 2015

*/
#include "common_magma_tally3.h"

#define BLOCK_SIZE 512


__global__ void 
smgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    float alpha, 
    float * dval, 
    magma_tally3_index_t * drowptr, 
    magma_tally3_index_t * dcolind,
    float * dx,
    float beta, 
    float * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ float dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally3_S_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            float val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally3_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally3_int_t
                number of rows in A

    @param[in]
    n           magma_tally3_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    dval        magma_tally3Float_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_tally3Index_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_tally3Index_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_tally3Float_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[out]
    dy          magma_tally3Float_ptr
                input/output vector y

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_sblas
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_smgecsrmv(
    magma_tally3_trans_t transA,
    magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3_int_t num_vecs, 
    float alpha,
    magma_tally3Float_ptr dval,
    magma_tally3Index_ptr drowptr,
    magma_tally3Index_ptr dcolind,
    magma_tally3Float_ptr dx,
    float beta,
    magma_tally3Float_ptr dy,
    magma_tally3_queue_t queue )
{
    dim3 grid( magma_tally3_ceildiv( m, BLOCK_SIZE ), 1, 1);
    magma_tally3_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( float ); // num_vecs vectors 
    smgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_tally3_SUCCESS;
}



