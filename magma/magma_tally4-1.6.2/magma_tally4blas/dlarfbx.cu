#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarfbx.cu normal z -> d, Fri Jan 30 19:00:08 2015

*/
#include "common_magma_tally4.h"
#include "commonblas_d.h"
#include "magma_tally4_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512




//==============================================================================
extern "C"
__global__ void 
magma_tally4_dgemv_kernel1(int m, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ c, 
                    double *dwork)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally4_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally4_D_MUL( MAGMA_tally4_D_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_tally4_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_tally4_dgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        DGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
    ----------------------------------------------------------------------------- */
extern "C"
__global__ void
magma_tally4_dgemv_kernel3(int m, const double * __restrict__ V, int ldv, double *c,
                    double *dwork, double *tau)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    if (i==0)
       c[0] = MAGMA_tally4_D_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally4_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally4_D_MUL( MAGMA_tally4_D_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_tally4_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================
extern "C"
__global__ void
magma_tally4_dgemv_kernel2(int m, int n, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ x, double *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    double lsum;

    V += j;

    lsum = MAGMA_tally4_D_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_tally4_D_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a real block reflector H to a real vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the real k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a real block of
    k elementary reflectors. 
*/
extern "C" void
magma_tally4_dlarfbx_gpu(
    magma_tally4_int_t m, magma_tally4_int_t k,
    magma_tally4Double_ptr V,  magma_tally4_int_t ldv,
    magma_tally4Double_ptr dT, magma_tally4_int_t ldt,
    magma_tally4Double_ptr c,
    magma_tally4Double_ptr dwork)
{
    /* dwork = V**H c     */
    magma_tally4_dgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_tally4_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_tally4_dtrmv_tkernel<<< k, k, 0, magma_tally4_stream >>>( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_tally4_dgemv_kernel2<<< blocks3, threads3, 0, magma_tally4_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
