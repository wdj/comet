#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeelltmv.cu normal z -> d, Sun May  3 11:22:58 2015

*/

#include "common_magma_minproduct.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
__global__ void 
dgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double * dval, 
    magma_minproduct_index_t * dcolind,
    double * dx,
    double beta, 
    double * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_minproduct_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            double val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
dgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double lambda, 
    double * dval, 
    magma_minproduct_index_t * dcolind,
    double * dx,
    double beta, 
    int offset,
    int blocksize,
    magma_minproduct_index_t * addrows,
    double * dy)
{

    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_minproduct_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            double val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_minproduct_int_t
                number of elements in the longest row 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    dval        magma_minproductDouble_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_minproductDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[out]
    dy          magma_minproductDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_d
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_dgeelltmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t nnz_per_row,
    double alpha,
    magma_minproductDouble_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductDouble_ptr dx,
    double beta,
    magma_minproductDouble_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
    dgeelltmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_minproduct_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_minproduct_int_t
                number of elements in the longest row 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    lambda      double
                scalar multiplier

    @param[in]
    dval        magma_minproductDouble_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_minproductDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier
                
    @param[in]
    offset      magma_minproduct_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_minproduct_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_minproductIndex_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_minproductDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_dblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_dgeelltmv_shift(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t nnz_per_row,
    double alpha,
    double lambda,
    magma_minproductDouble_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductDouble_ptr dx,
    double beta,
    int offset,
    int blocksize,
    magma_minproductIndex_ptr addrows,
    magma_minproductDouble_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
    double tmp_shift;
    //magma_minproduct_dsetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    dgeelltmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


   return MAGMA_minproduct_SUCCESS;
}



