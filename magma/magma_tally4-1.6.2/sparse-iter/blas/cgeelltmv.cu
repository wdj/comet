#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeelltmv.cu normal z -> c, Sun May  3 11:22:58 2015

*/

#include "common_magma_tally4.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
__global__ void 
cgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally4FloatComplex alpha, 
    magma_tally4FloatComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4FloatComplex * dx,
    magma_tally4FloatComplex beta, 
    magma_tally4FloatComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally4FloatComplex dot = MAGMA_tally4_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally4FloatComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
cgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally4FloatComplex alpha, 
    magma_tally4FloatComplex lambda, 
    magma_tally4FloatComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4FloatComplex * dx,
    magma_tally4FloatComplex beta, 
    int offset,
    int blocksize,
    magma_tally4_index_t * addrows,
    magma_tally4FloatComplex * dy)
{

    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally4FloatComplex dot = MAGMA_tally4_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally4FloatComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally4_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally4FloatComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4FloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally4FloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4FloatComplex
                scalar multiplier

    @param[out]
    dy          magma_tally4FloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_c
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_cgeelltmv(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t nnz_per_row,
    magma_tally4FloatComplex alpha,
    magma_tally4FloatComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4FloatComplex_ptr dx,
    magma_tally4FloatComplex beta,
    magma_tally4FloatComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ) );
    magma_tally4_int_t threads = BLOCK_SIZE;
    cgeelltmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally4_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally4_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally4FloatComplex
                scalar multiplier

    @param[in]
    lambda      magma_tally4FloatComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4FloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally4FloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4FloatComplex
                scalar multiplier
                
    @param[in]
    offset      magma_tally4_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally4_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally4Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally4FloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_cblas
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_cgeelltmv_shift(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t nnz_per_row,
    magma_tally4FloatComplex alpha,
    magma_tally4FloatComplex lambda,
    magma_tally4FloatComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4FloatComplex_ptr dx,
    magma_tally4FloatComplex beta,
    int offset,
    int blocksize,
    magma_tally4Index_ptr addrows,
    magma_tally4FloatComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ) );
    magma_tally4_int_t threads = BLOCK_SIZE;
    magma_tally4FloatComplex tmp_shift;
    //magma_tally4_csetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    cgeelltmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


   return MAGMA_tally4_SUCCESS;
}



