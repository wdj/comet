#include "hip/hip_runtime.h"
/*
   -- MAGMA_tally4 (version 1.6.1) --
   Univ. of Tennessee, Knoxville
   Univ. of California, Berkeley
   Univ. of Colorado, Denver
   @date January 2015

   @author Azzam Haidar
   @author Tingxing Dong

   @generated from zgemv_batched.cu normal z -> s, Fri Jan 30 19:00:10 2015
 */
#include "common_magma_tally4.h"


#define sgemv_bs 32

extern __shared__ float shared_data[];


__global__ void
kernel_sgemvn_batched(
    int m, int n, float alpha,
    float **dA_array, int lda,
    float **x_array, int incx,
    float beta, float  **y_array, int incy)
{

    float *A = dA_array[blockIdx.x];
    float *x = x_array[blockIdx.x];
    float *y = y_array[blockIdx.x];

    int tx = threadIdx.x;

    float res = MAGMA_tally4_S_ZERO;

    float *buff = (float*)shared_data;

    if(tx < n)
    {
        buff[tx] = x[tx*incx];
    }
    __syncthreads();
   
    
    if(tx < m )
    {
        for(int j=0; j < n ; j++)
        {
            res += A[tx]*buff[j];
            A += lda;
        }
  
        y[tx*incy] = alpha * res + y[tx*incy] * beta;
    }

}

/*
    Matrix Non-transpose Vector Multiplication
    y := alpha*A*x + beta*y,
*/
extern "C"
void magma_tally4blas_sgemvn_batched(
    int m, int n, 
    float alpha, float **dA_array, int lda, 
    float **x_array,  int incx,
    float beta, float **y_array,  int incy, 
    int batchCount, magma_tally4_queue_t queue)
{

    if( m > 512 || n > 512)
    {
        fprintf( stderr, "m=%d, n=%d, sgemv_batched nontranspose assume row && column lower than %d. Plz call magma_tally4blas_sgemv instead", m, n, 512);
        return ;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(m,n), 1, 1);
   
    kernel_sgemvn_batched<<< grid, threads, n * sizeof(float), queue >>>( m, n, alpha,  dA_array, lda, x_array, incx,  
                                                                         beta, y_array, incy);
}



__global__ void
kernel_sgemvt_batched(
    int m, int n, int m1, float alpha,
    float **dA_array, int lda,
    float **x_array, int incx,
    float beta, float  **y_array, int incy)
{
  

    float *A_ptr = dA_array[blockIdx.x];
    float *x_ptr = x_array[blockIdx.x];
    float *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    float res = MAGMA_tally4_S_ZERO;

    if(tx<m)
    {  
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ float sdata[sgemv_bs];

    for(int i=0; i<m1; i+= sgemv_bs)
    {
        res += A_ptr[i] * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += A_ptr[m1] * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(sgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Transpose Vector Multiplication
    y := alpha* A**T *x + beta*y,
*/

extern "C"
void magma_tally4blas_sgemvt_batched(
    int m, int n, 
    float alpha, float **dA_array, int lda, 
    float **x_array,  int incx,
    float beta, float **y_array,  int incy, 
    int batchCount, magma_tally4_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(sgemv_bs, 1, 1);

    int m1 = (m / sgemv_bs) * sgemv_bs;

    kernel_sgemvt_batched <<< grid, threads,0, queue  >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);

}
   

#if defined(PRECISION_z) || defined (PRECISION_c)


__global__ void
kernel_sgemvc_batched(
    int m, int n, int m1, float alpha,
    float **dA_array, int lda,
    float **x_array, int incx,
    float beta, float  **y_array, int incy)
{
  

    float *A_ptr = dA_array[blockIdx.x];
    float *x_ptr = x_array[blockIdx.x];
    float *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    float res = MAGMA_tally4_S_ZERO;

    if(tx<m)
    {
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ float sdata[sgemv_bs];

    for(int i=0; i<m1; i+= sgemv_bs)
    {
        res += MAGMA_tally4_S_CNJG (A_ptr[i]) * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += MAGMA_tally4_S_CNJG(A_ptr[m1]) * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(sgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Conjugate Transpose Vector Multiplication
    y := alpha* A**H *x + beta*y,
*/

extern "C"
void magma_tally4blas_sgemvc_batched(
    int m, int n, 
    float alpha, float **dA_array, int lda, 
    float **x_array,  int incx,
    float beta, float **y_array,  int incy, 
    int batchCount, magma_tally4_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(sgemv_bs, 1, 1);

    int m1 = (m / sgemv_bs) * sgemv_bs;

    kernel_sgemvc_batched <<< grid, threads, 0, queue >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);
}
   
#endif // defined(PRECISION_z) || defined (PRECISION_c)


/**
    Purpose
    -------

    This routine computes Y = alpha opt(A) x + beta y, on the GPU, where
    A = dA_array[i],x = x_array[i] and y = y_array[i], i=[0,batchCount-1].
    This is a batched version.

    @param[in]
    trans  CHARACTER*1.
           On entry, TRANS specifies the form of op( A ) to be used in
           the matrix multiplication as follows:
           = 'N':  op( A ) = A.
           = 'T':  op( A ) = A**T.
           = 'C':  op( A ) = A**H.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix opt(A).

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix opt(A)

    @param[in]
    alpha   REAL.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA_array A = dA_array[i] 
            A: REAL array of dimension ( LDA, n ) on the GPU.
   
    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x_array x = x_array[i]
            x: REAL array of dimension.
            n if trans == Magma_tally4NoTrans.
            m if trans == Magma_tally4Trans or Magma_tally4ConjTrans.

    @param[in]
    incx    INTEGER.
            incx specifies the increment for the elments of x.
            incx must not be zero.
    
    @param[in]
    beta    REAL.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y_array y = y_array[i]:       
            On exit y = alpha opt(A) x + beta y.
            y: REAL array of dimension.
            m if trans == Magma_tally4NoTrans.
            n if trans == Magma_tally4Trans or Magma_tally4ConjTrans.

    @param[in]
    incy    INTEGER.
            incy specifies the increment for the elments of y.
            incy must not be zero.
    
    @param[in]
    batchCount INTEGER
            number of pointers contained in dA_array, x_array and y_array.

    @ingroup magma_tally4_sblas2
    *******************************************************************   */

extern "C"
void magma_tally4blas_sgemv_batched(
    magma_tally4_trans_t trans, magma_tally4_int_t m, magma_tally4_int_t n, 
    float alpha,
    magma_tally4Float_ptr dA_array[], magma_tally4_int_t ldda, 
    magma_tally4Float_ptr dx_array[], magma_tally4_int_t incx,
    float beta,
    magma_tally4Float_ptr dy_array[], magma_tally4_int_t incy, 
    magma_tally4_int_t batchCount, magma_tally4_queue_t queue)
{       
    magma_tally4_int_t info = 0;
    if ( trans != Magma_tally4NoTrans && trans != Magma_tally4Trans && trans != Magma_tally4ConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;

    if (info != 0) {
        magma_tally4_xerbla( __func__, -(info) );
        return;  //info;
    }

    if(m==0 || n ==0 ) return;

    if ( trans == Magma_tally4NoTrans ) {

        magma_tally4blas_sgemvn_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
            
    }
    else if ( trans == Magma_tally4Trans ) {
        magma_tally4blas_sgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
    }
    else if ( trans == Magma_tally4ConjTrans ) {
#if defined(PRECISION_z) || defined (PRECISION_c)
        magma_tally4blas_sgemvc_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#else
        magma_tally4blas_sgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#endif
    }
    else {
        fprintf( stderr, "trans = %c is invalid\n", lapacke_trans_const(trans) );
    }
}

#undef sgemv_bs 
