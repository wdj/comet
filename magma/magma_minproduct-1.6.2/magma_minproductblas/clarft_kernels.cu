#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated from zlarft_kernels.cu normal z -> c, Fri Jan 30 19:00:10 2015
       @author Azzam Haidar
*/

#include "common_magma_minproduct.h"
#include "magma_minproduct_templates.h"
#define cgemv_bs 32
#define BLOCK_SIZE 512

#define use_gemm_larft

extern __shared__ magma_minproductFloatComplex shared_data[];


//===================================================================================================
static __device__
void clarft_gemvcolwise_device( int m, magma_minproductFloatComplex *v, magma_minproductFloatComplex *tau,
                         magma_minproductFloatComplex *c, int ldc, magma_minproductFloatComplex *T, int ldt, int step )
{

    const int thblk =  blockIdx.x;
    if (thblk > step)
        return;
    /* if blockIdx.x<step step performs the z = V(tx:n,tx)' * V(tx:n,1:tx-1) used for computing T:*/

    if ( !MAGMA_minproduct_C_EQUAL(*tau, MAGMA_minproduct_C_ZERO) ) {
        if(thblk<step){    
            const int tx = threadIdx.x;
            magma_minproductFloatComplex *dc = c + blockIdx.x * ldc;
           
            __shared__ magma_minproductFloatComplex sum[ BLOCK_SIZE ];
            magma_minproductFloatComplex tmp;
           
            /* perform  {T_i}^H := V(:,i)' * V(:,1:i-1)  */
            if (tx==0)
                tmp = dc[0]; //since V[0] should be one
            else
                tmp = MAGMA_minproduct_C_ZERO;
            for( int j = tx+1; j < m; j += BLOCK_SIZE ){
                tmp +=  MAGMA_minproduct_C_CNJG( v[j] ) * dc[j];
            }
            sum[tx] = tmp;
            magma_minproduct_sum_reduce< BLOCK_SIZE >( tx, sum );
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_minproduct_C_CNJG(sum[0]);
            #else
            tmp = - MAGMA_minproduct_C_CNJG(*tau) * sum[0]; 
            *(T+thblk) = MAGMA_minproduct_C_CNJG(tmp); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            //*(T+thblk) = - MAGMA_minproduct_C_CNJG(sum[0]) * (*tau); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            #endif
        }
        else{
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_minproduct_C_ONE;
            #else
            *(T+thblk) = *tau;
            #endif
        }
    }// in case tau is zero put the corresponding column of T to zero
    else 
    {
        *(T+thblk) = MAGMA_minproduct_C_ZERO;
    }
}
//===================================================================================================
__global__
void clarft_gemvcolwise_kernel( int m, magma_minproductFloatComplex *v, int ldv, magma_minproductFloatComplex *tau,
                          magma_minproductFloatComplex *T, int ldt, int step )
{
    clarft_gemvcolwise_device(m, v+step+step*ldv, tau+step, v+step, ldv, T+step*ldt, ldt, step);
}
//===================================================================================================
__global__
void clarft_gemvcolwise_kernel_batched( int m, magma_minproductFloatComplex **v_array, int ldv, magma_minproductFloatComplex **tau_array,
                          magma_minproductFloatComplex **T_array, int ldt, int step )
{
    int batchid = blockIdx.z;
    clarft_gemvcolwise_device(m, v_array[batchid]+step+step*ldv, tau_array[batchid]+step, v_array[batchid]+step, ldv, T_array[batchid]+step*ldt, ldt, step);
}
//===================================================================================================
extern "C" 
void magma_minproductblas_clarft_gemvcolwise(
    magma_minproduct_int_t m,  magma_minproduct_int_t step,
    magma_minproductFloatComplex *v, magma_minproduct_int_t ldv, 
    magma_minproductFloatComplex *T,  magma_minproduct_int_t ldt,
    magma_minproductFloatComplex *tau)
{
    dim3 grid( step+1, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    clarft_gemvcolwise_kernel<<< grid, threads, 0, magma_minproduct_stream >>>( m, v, ldv, tau, T, ldt, step);

}
//===================================================================================================
extern "C" 
void magma_minproductblas_clarft_gemvcolwise_batched(
    magma_minproduct_int_t m,  magma_minproduct_int_t step,
    magma_minproductFloatComplex **v_array, magma_minproduct_int_t ldv, 
    magma_minproductFloatComplex **T_array,  magma_minproduct_int_t ldt,
    magma_minproductFloatComplex **tau_array, magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue )
{
    dim3 grid( step+1, 1, batchCount );
    dim3 threads( BLOCK_SIZE );
    clarft_gemvcolwise_kernel_batched<<< grid, threads, 0, queue >>>( m, v_array, ldv, tau_array, T_array, ldt, step);

}
//===================================================================================================




//===================================================================================================
// cgemv(y=alpha*A*x) interface: T/W=tau*v*x, 
static __device__ void
clarft_gemvrowwise_device(
    int m, int i,
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v_ptr, int ldv, 
    magma_minproductFloatComplex *x_ptr, int incx,
    magma_minproductFloatComplex *T_ptr, int ldt,
    magma_minproductFloatComplex *W, magma_minproductFloatComplex* sdata)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 


    if(tx ==0 && ty == 0)
    {
        T_ptr[0] = *tau;
    } 

    if(i <= 0) return;
    
    magma_minproductFloatComplex res = MAGMA_minproduct_C_ZERO;

    v_ptr += ldv * ty;
            

   
    if(tx < cgemv_bs)
    {
        for(int s=tx; s<m; s+= cgemv_bs)
        {
            res += MAGMA_minproduct_C_CNJG (v_ptr[s]) * x_ptr[s*incx];
        }
    
        sdata[ty * cgemv_bs + tx] = res;
    }
    __syncthreads();

    magma_minproduct_sum_reduce<cgemv_bs>(tx, &(sdata[ty*cgemv_bs+0]));

    #if defined (use_gemm_larft)
    if(tx == 0)
    {
            W[ty] = -sdata[ty * cgemv_bs + 0];
    } 
    #else
    if(tx == 0)
    {
            W[ty] = -sdata[ty * cgemv_bs + 0] * (*tau) ;
    }
    #endif 
}




//T(1:i-1,i) := - tau(i) * V(i:n,1:i-1)' * V(i:n,i)
//T(i,i) = tau(i)
//===================================================================================================
 __global__ void
clarft_gemvrowwise_kernel(
    int m, int i, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v, int ldv, 
    magma_minproductFloatComplex *T, int ldt)
{

    magma_minproductFloatComplex *W =  T +i*ldt;

    magma_minproductFloatComplex *sdata = (magma_minproductFloatComplex*)shared_data;

    clarft_gemvrowwise_device(m, i, tau+i, v+i, ldv,  v+i+i*ldv, 1,  
                           T+i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
__global__ void
clarft_gemvrowwise_kernel_batched(
    int m, int i,
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **v_array, int ldv, 
    magma_minproductFloatComplex **T_array, int ldt)
{

    int batchid = blockIdx.z;

    magma_minproductFloatComplex *W =  T_array[batchid] +i*ldt;

    magma_minproductFloatComplex *sdata = (magma_minproductFloatComplex*)shared_data;

    clarft_gemvrowwise_device(m, i, tau_array[batchid]+i, v_array[batchid]+i, ldv,  v_array[batchid]+i+i*ldv, 1,  
                           T_array[batchid] +i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
extern "C"
void magma_minproductblas_clarft_gemvrowwise(
    magma_minproduct_int_t m, magma_minproduct_int_t i, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v, magma_minproduct_int_t ldv, 
    magma_minproductFloatComplex *T, magma_minproduct_int_t ldt,
    magma_minproductFloatComplex *W)
{

    dim3 grid(1);


    dim3 threads(cgemv_bs, max(i,1), 1);


    clarft_gemvrowwise_kernel <<< grid, threads, sizeof(magma_minproductFloatComplex)*cgemv_bs*(i+1), magma_minproduct_stream>>>(m, i, tau, v, ldv, T, ldt);
}
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_gemvrowwise_batched(
    magma_minproduct_int_t m, magma_minproduct_int_t i, 
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **v_array, magma_minproduct_int_t ldv, 
    magma_minproductFloatComplex **T_array, magma_minproduct_int_t ldt,
    magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(cgemv_bs, max(i,1), 1);

    /*  cgemvrowwise used a bigger shared memory and has more data reuse and performs better
    */
    clarft_gemvrowwise_kernel_batched <<< grid, threads, sizeof(magma_minproductFloatComplex)*cgemv_bs*(i+1), queue>>>(m, i,  tau_array, v_array, ldv, T_array, ldt);
}
//===================================================================================================
   


//===================================================================================================
/*
   loop_inside
*/
static __device__ void
clarft_gemv_loop_inside_device(
    int n, int k, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v, int ldv, 
    magma_minproductFloatComplex *T, int ldt)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    
    int incx = 1;
    magma_minproductFloatComplex *sdata = (magma_minproductFloatComplex*)shared_data;

    magma_minproductFloatComplex res;

    // write the first elment
    if(tx ==0 && ty == 0)
    {
        T[0] = tau[0];
    } 
 
    for(int i=1; i<k;i++)
    {

        int m = n-i; 

        magma_minproductFloatComplex *v_ptr = v;

        v_ptr += i;

        magma_minproductFloatComplex *x_ptr = v_ptr + i * ldv;
            
        res = MAGMA_minproduct_C_ZERO;
            
        if(tx < cgemv_bs && ty < i)
        {
            v_ptr += ldv * ty;

            for(int s=tx; s<m; s+= cgemv_bs)
            {
                res += MAGMA_minproduct_C_CNJG (v_ptr[s]) * x_ptr[s*incx];
            }
    
            sdata[ty * cgemv_bs + tx] = res;
        }
        __syncthreads();

        magma_minproduct_sum_reduce<cgemv_bs>(tx, &(sdata[ty*cgemv_bs+0]));
        

       __syncthreads();
       #if defined (use_gemm_larft)
       if(tx < i && ty == 0)
       {
            T[i* ldt + tx] = sdata[tx * cgemv_bs + 0];  
       } 
       // not needed since it is overwritten in trmv
       /*
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       */
       #else
       if(tx < i && ty == 0)
       {
           T[i* ldt + tx] = -sdata[tx * cgemv_bs + 0] * (tau[i]) ;  
       } 
      
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       #endif
     
       v_ptr -= i;

    }// end of loop k
}
//===================================================================================================
__global__ void
clarft_gemv_loop_inside_kernel(
    int n, int k, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v, int ldv, 
    magma_minproductFloatComplex *T, int ldt)
{
    clarft_gemv_loop_inside_device(n, k, tau, v, ldv, T, ldt);
}
//===================================================================================================
__global__ void
clarft_gemv_loop_inside_kernel_batched(
    int n, int k, 
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **v_array, int ldv, 
    magma_minproductFloatComplex **T_array, int ldt)
{
    int batchid = blockIdx.z;
    clarft_gemv_loop_inside_device(n, k, tau_array[batchid], v_array[batchid], ldv, T_array[batchid], ldt);
}
//===================================================================================================
//===================================================================================================
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_gemv_loop_inside(
    int n, int k, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *v, int ldv, 
    magma_minproductFloatComplex *T, int ldt)
{

    dim3 grid(1);
    dim3 threads(cgemv_bs, max(k,1), 1);
    clarft_gemv_loop_inside_kernel<<<grid, threads, sizeof(magma_minproductFloatComplex) * (cgemv_bs*(k+1)), magma_minproduct_stream>>>(n, k, tau, v, ldv, T, ldt); 
}
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_gemv_loop_inside_batched(
    int n, int k, 
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **v_array, int ldv, 
    magma_minproductFloatComplex **T_array, int ldt, magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(cgemv_bs, max(k,1), 1);
    clarft_gemv_loop_inside_kernel_batched<<<grid, threads, sizeof(magma_minproductFloatComplex) * (cgemv_bs*(k+1)), queue>>>(n, k, tau_array, v_array, ldv, T_array, ldt); 
}
//===================================================================================================





//===================================================================================================
static  __device__ void 
clarft_ctrmv_sm32x32_device(
    int n, int k, magma_minproductFloatComplex *tau,
    magma_minproductFloatComplex *Tin, int ldtin,  magma_minproductFloatComplex *Tout, int ldtout )
{
    int tx = threadIdx.x; 
    magma_minproductFloatComplex *sdata = (magma_minproductFloatComplex*)shared_data;
    magma_minproductFloatComplex res;

    // this routine apply a sequence of trmv to update k column of the triangular
    // T starting at n-k to n where T is of size n by n and where the first n-k 
    // columns of T are supposed updated previously.
    // So the routine load all of T nxn to the shared memory 
    // and apply the sequence of trmv.
    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n-k; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
    
#if defined(use_gemm_larft)
    for(int s=n-k; s<n; s++)
    {
        if(tx == s)
            sdata[tx + s*n] = tau[s];
        else
            sdata[tx + s*n] = -tau[s] * Tin[tx + s * ldtin];
    }
#else
    for(int s=n-k; s<n; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
#endif

    // perform trmv
    for(int i=n-k; i<n;i++)
    {
       __syncthreads();  
       res = MAGMA_minproduct_C_ZERO;
       if(tx < i)
       {
           for(int j=tx; j<i; j++)
           {
               res += sdata[tx + j * n] * sdata[j+ i * n];      
           }
       }       
       __syncthreads();  
       if(tx < i)
       {
           sdata[tx + i * n] = res;
       }
    } 

    __syncthreads();  
    // write back the updated block of k column of T
    for(int s=n-k; s<n; s++)
    {
       Tout[tx + s * ldtout] = sdata[tx + s*n];
    }

}
//===================================================================================================
__global__ void 
clarft_ctrmv_sm32x32_kernel(
    int n, int k, magma_minproductFloatComplex *tau,
    magma_minproductFloatComplex *Tin, int ldtin,  magma_minproductFloatComplex *Tout, int ldtout )
{
    clarft_ctrmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
__global__ void 
clarft_ctrmv_sm32x32_kernel_batched(
    int n, int k, magma_minproductFloatComplex **tau_array,
    magma_minproductFloatComplex **Tin_array, int ldtin,  magma_minproductFloatComplex **Tout_array, int ldtout )
{
    int batchId = blockIdx.z;
    clarft_ctrmv_sm32x32_device( n, k, tau_array[batchId], Tin_array[batchId], ldtin, Tout_array[batchId], ldtout);
}
//===================================================================================================
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_ctrmv_sm32x32(
    magma_minproduct_int_t m, magma_minproduct_int_t n, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *Tin, magma_minproduct_int_t ldtin, 
    magma_minproductFloatComplex *Tout, magma_minproduct_int_t ldtout)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    clarft_ctrmv_sm32x32_kernel <<< grid, threads, sizeof(magma_minproductFloatComplex)*(m*m), magma_minproduct_stream >>> (m, n,  tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_ctrmv_sm32x32_batched(
    magma_minproduct_int_t m, magma_minproduct_int_t n, 
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **Tin_array, magma_minproduct_int_t ldtin, 
    magma_minproductFloatComplex **Tout_array, magma_minproduct_int_t ldtout,
    magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    clarft_ctrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magma_minproductFloatComplex)*(m*m), queue >>> (m, n,  tau_array, Tin_array, ldtin, Tout_array, ldtout);
}
//===================================================================================================




//===================================================================================================
//===================================================================================================
static __device__ void 
clarft_recctrmv_sm32x32_device(
    int m, int n, magma_minproductFloatComplex *tau,
    magma_minproductFloatComplex *Trec, int ldtrec, magma_minproductFloatComplex *Ttri, int ldttri)
{
    int tx = threadIdx.x; 
    magma_minproductFloatComplex *sdata = (magma_minproductFloatComplex*)shared_data;
    magma_minproductFloatComplex res;

    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n; s++)
    {
        sdata[tx + s*n] = Trec[tx + s * ldtrec];
    }
    __syncthreads();  
    
    // perform sequence of n-1 gemv
    for(int i=0; i<n;i++)
    {
       res = MAGMA_minproduct_C_ZERO;
       for(int j=0; j<i; j++)
       {
           res += sdata[tx + j * n] * Ttri[j+ i * ldttri];      
       }
       __syncthreads();   // a enlever
       sdata[tx + i * n] = -tau[i] * (sdata[tx + i * n] + res);
       __syncthreads();  
    } 

    // write back the updated block of k column of T  multiplying by -tau
    for(int s=0; s<n; s++)
    {
       Trec[tx + s * ldtrec] = sdata[tx + s*n];
    }

}

//===================================================================================================
__global__ void 
clarft_recctrmv_sm32x32_kernel(
    int m, int n, magma_minproductFloatComplex *tau,
    magma_minproductFloatComplex *Trec, int ldtrec, magma_minproductFloatComplex *Ttri, int ldttri)
{
    clarft_recctrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
__global__ void 
clarft_recctrmv_sm32x32_kernel_batched(
    int m, int n, magma_minproductFloatComplex **tau_array,
    magma_minproductFloatComplex **Trec_array, int ldtrec, magma_minproductFloatComplex **Ttri_array, int ldttri)
{
    int batchId = blockIdx.z;
    clarft_recctrmv_sm32x32_device(m, n, tau_array[batchId], Trec_array[batchId], ldtrec, Ttri_array[batchId], ldttri);
}
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_recctrmv_sm32x32(
    magma_minproduct_int_t m, magma_minproduct_int_t n, 
    magma_minproductFloatComplex *tau, 
    magma_minproductFloatComplex *Trec, magma_minproduct_int_t ldtrec, 
    magma_minproductFloatComplex *Ttri, magma_minproduct_int_t ldttri)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    clarft_recctrmv_sm32x32_kernel <<< grid, threads, sizeof(magma_minproductFloatComplex)*(m*n), magma_minproduct_stream >>> (m, n,  tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
extern "C"
void magma_minproductblas_clarft_recctrmv_sm32x32_batched(
    magma_minproduct_int_t m, magma_minproduct_int_t n, 
    magma_minproductFloatComplex **tau_array, 
    magma_minproductFloatComplex **Trec_array, magma_minproduct_int_t ldtrec, 
    magma_minproductFloatComplex **Ttri_array, magma_minproduct_int_t ldttri,
    magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    clarft_recctrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magma_minproductFloatComplex)*(m*n), queue >>> (m, n,  tau_array, Trec_array, ldtrec, Ttri_array, ldttri);
}
//===================================================================================================


