#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from ztrtri_diag.cu normal z -> d, Fri Jan 30 19:00:09 2015

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       File named dtrtri_diag.cu to avoid name conflict with src/dtrtri.o
       in the library. The actual kernels are in dtrtri_lower.cu and dtrtri_upper.cu
*/

#include "common_magma_tally3.h"
#include "dtrtri.h"


/**
    Inverts the NB x NB diagonal blocks of a triangular matrix.
    This routine is used in dtrsm.
    
    Same as dtrtri_diag, but adds queue argument.
    
    @ingroup magma_tally3_dblas3
    ********************************************************************/
/**
    Purpose
    -------
    dtrtri_diag inverts the NB x NB diagonal blocks of A.

    Arguments
    ----------
    @param[in]
    uplo    magma_tally3_uplo_t.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:
      -     = Magma_tally3Upper:  A is an upper triangular matrix.
      -     = Magma_tally3Lower:  A is a  lower triangular matrix.

    @param[in]
    diag    magma_tally3_diag_t.
            On entry, diag specifies whether or not A is unit triangular
            as follows:
      -     = Magma_tally3Unit:     A is assumed to be unit triangular.
      -     = Magma_tally3NonUnit:  A is not assumed to be unit triangular.

    @param[in]
    n       INTEGER.
            On entry, n specifies the order of the matrix A. N >= 0.

    @param[in]
    dA      DOUBLE_PRECISION array of dimension ( ldda, n )
            The triangular matrix A.
    \n
            If UPLO = 'U', the leading N-by-N upper triangular part of A
            contains the upper triangular matrix, and the strictly lower
            triangular part of A is not referenced.
    \n
            If UPLO = 'L', the leading N-by-N lower triangular part of A
            contains the lower triangular matrix, and the strictly upper
            triangular part of A is not referenced.
    \n
            If DIAG = 'U', the diagonal elements of A are also not referenced
            and are assumed to be 1.

    @param[in]
    ldda    INTEGER.
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    d_dinvA DOUBLE_PRECISION array of dimension (NB, ((n+NB-1)/NB)*NB),
            where NB = 128.
            On exit, contains inverses of the NB-by-NB diagonal blocks of A.

    @param[in]
    queue   magma_tally3_queue_t
            Queue to execute in.

    @ingroup magma_tally3_dblas3
    ********************************************************************/
extern "C" void
magma_tally3blas_dtrtri_diag_q(
    magma_tally3_uplo_t uplo, magma_tally3_diag_t diag, magma_tally3_int_t n,
    magma_tally3Double_const_ptr dA, magma_tally3_int_t ldda,
    magma_tally3Double_ptr d_dinvA,
    magma_tally3_queue_t queue)
{
    magma_tally3_int_t info = 0;
    if (uplo != Magma_tally3Lower && uplo != Magma_tally3Upper)
        info = -1;
    else if (diag != Magma_tally3NonUnit && diag != Magma_tally3Unit)
        info = -2;
    else if (n < 0)
        info = -3;
    else if (ldda < n)
        info = -5;

    if (info != 0) {
        magma_tally3_xerbla( __func__, -(info) );
        return;  //info
    }
    
    int nblocks = (n + IB - 1)/IB;

    hipMemset( d_dinvA, 0, ((n+NB-1)/NB)*NB*NB * sizeof(double) );
    
    if ( uplo == Magma_tally3Lower ) {
        // invert diagonal IB x IB inner blocks
        dtrtri_diag_lower_kernel<<< nblocks, IB, 0, queue >>>( diag, n, dA, ldda, d_dinvA );

        // build up NB x NB blocks (assuming IB=16 here):
        // use   16 x 16  blocks to build  32 x 32  blocks,  1 x (1 x npages) grid,  4 x 4 threads;
        // then  32 x 32  blocks to build  64 x 64  blocks,  1 x (2 x npages) grid,  8 x 4 threads;
        // then  64 x 64  blocks to build 128 x 128 blocks,  1 x (4 x npages) grid, 16 x 4 threads;
        // then 128 x 128 blocks to build 256 x 256 blocks,  2 x (8 x npages) grid, 16 x 4 threads.
        for( int jb=IB; jb < NB; jb *= 2 ) {
            int kb = jb*2;
            int npages = (n + kb - 1)/kb;
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            //printf( "n %d, jb %d, grid %d x %d (%d x %d)\n", n, jb, grid.x, grid.y, grid.y / npages, npages );
            switch (jb) {
                case 16:
                    triple_dgemm16_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm16_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_dgemm32_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm32_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_dgemm64_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm64_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_dgemm_above64_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part3_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
    else {
        dtrtri_diag_upper_kernel<<< nblocks, IB, 0, queue >>>( diag, n, dA, ldda, d_dinvA );

        // update the inverse up to the size of IB
        for( int jb=IB; jb < NB; jb*=2 ) {
            int kb = jb*2;
            int npages = (n + kb - 1)/kb;
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            switch (jb) {
                case 16:
                    triple_dgemm16_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm16_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_dgemm32_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm32_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_dgemm64_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm64_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_dgemm_above64_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part3_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
}

/**
    @see magma_tally3blas_dtrtri_diag_q
    @ingroup magma_tally3_dblas3
    ********************************************************************/
extern "C" void
magma_tally3blas_dtrtri_diag(
    magma_tally3_uplo_t uplo, magma_tally3_diag_t diag, magma_tally3_int_t n,
    magma_tally3Double_const_ptr dA, magma_tally3_int_t ldda,
    magma_tally3Double_ptr d_dinvA)
{
    magma_tally3blas_dtrtri_diag_q( uplo, diag, n, dA, ldda, d_dinvA, magma_tally3_stream );
}
