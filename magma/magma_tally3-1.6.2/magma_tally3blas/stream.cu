/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Mark Gates
*/

#include "common_magma_tally3.h"

magma_tally3_queue_t magma_tally3_stream = 0;


/**
    Purpose
    -------
    magma_tally3blasSetKernelStream sets the CUDA stream that MAGMA_tally3 BLAS and
    CUBLAS (v1) routines use (unless explicitly given a stream).
    
    In a multi-threaded application, be careful to avoid race conditions
    when using this. For instance, if calls are executed in this order:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  magma_tally3blasSetKernelStream( s1 )         
    2.                                      magma_tally3blasSetKernelStream( s2 )
    3.  magma_tally3_dgemm( ... )
    4.                                      magma_tally3_dgemm( ... )
    @endverbatim
    
    both magma_tally3_dgemm would occur on stream s2. A lock should be used to prevent
    this, so the dgemm in thread 1 uses stream s1, and the dgemm in thread 2
    uses s2:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  lock()                                  
    2.  magma_tally3blasSetKernelStream( s1 )          
    3.  magma_tally3_dgemm( ... )                      
    4.  unlock()                                
    5.                                      lock()
    6.                                      magma_tally3blasSetKernelStream( s2 )
    7.                                      magma_tally3_dgemm( ... )
    8.                                      unlock()
    @endverbatim
    
    Most BLAS calls in MAGMA_tally3, such as magma_tally3_dgemm, are asynchronous, so the lock
    will only have to wait until dgemm is queued, not until it is finished.
    
    Arguments
    ---------
    @param[in]
    stream  magma_tally3_queue_t
            The CUDA stream.

    @ingroup magma_tally3_util
    ********************************************************************/
extern "C"
hipblasStatus_t magma_tally3blasSetKernelStream( magma_tally3_queue_t stream )
{
    magma_tally3_stream = stream;
    return cublasSetKernelStream( stream );
}


/**
    Purpose
    -------
    magma_tally3blasGetKernelStream gets the CUDA stream that MAGMA_tally3 BLAS
    routines use.

    Arguments
    ---------
    @param[out]
    stream  magma_tally3_queue_t
            The CUDA stream.

    @ingroup magma_tally3_util
    ********************************************************************/
extern "C"
hipblasStatus_t magma_tally3blasGetKernelStream( magma_tally3_queue_t *stream )
{
    *stream = magma_tally3_stream;
    return HIPBLAS_STATUS_SUCCESS;
}
