#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma_tally3.h"

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
zsymmetrize_lower( int m, magma_tally3DoubleComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    magma_tally3DoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magma_tally3DoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dAT = hipConj(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
zsymmetrize_upper( int m, magma_tally3DoubleComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    magma_tally3DoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magma_tally3DoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dA = hipConj(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
    }
}


/**
    Purpose
    -------
    
    ZSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_tally3_uplo_t
            Specifies the part of the matrix dA that is valid on input.
      -     = Magma_tally3Upper:      Upper triangular part
      -     = Magma_tally3Lower:      Lower triangular part
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            The m by m matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    queue   magma_tally3_queue_t
            Queue to execute in.

    @ingroup magma_tally3_zaux2
    ********************************************************************/
extern "C" void
magma_tally3blas_zsymmetrize_q(
    magma_tally3_uplo_t uplo, magma_tally3_int_t m,
    magma_tally3DoubleComplex_ptr dA, magma_tally3_int_t ldda,
    magma_tally3_queue_t queue )
{
    magma_tally3_int_t info = 0;
    if ( uplo != Magma_tally3Lower && uplo != Magma_tally3Upper )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( ldda < max(1,m) )
        info = -4;
    
    if ( info != 0 ) {
        magma_tally3_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 )
        return;
    
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    if ( uplo == Magma_tally3Upper ) {
        zsymmetrize_upper<<< grid, threads, 0, queue >>>( m, dA, ldda );
    }
    else {
        zsymmetrize_lower<<< grid, threads, 0, queue >>>( m, dA, ldda );
    }
}


/**
    @see magma_tally3blas_zsymmetrize_q
    @ingroup magma_tally3_zaux2
    ********************************************************************/
extern "C" void
magma_tally3blas_zsymmetrize(
    magma_tally3_uplo_t uplo, magma_tally3_int_t m,
    magma_tally3DoubleComplex_ptr dA, magma_tally3_int_t ldda )
{
    magma_tally3blas_zsymmetrize_q( uplo, m, dA, ldda, magma_tally3_stream );
}
