#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zmgeelltmv.cu normal z -> c, Sun May  3 11:22:58 2015

*/

#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


__global__ void 
cmgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        magma_tally2FloatComplex alpha, 
        magma_tally2FloatComplex * dval, 
        magma_tally2_index_t * dcolind,
        magma_tally2FloatComplex * dx,
        magma_tally2FloatComplex beta, 
        magma_tally2FloatComplex * dy)
{
    extern __shared__ magma_tally2FloatComplex dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally2_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally2FloatComplex val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_tally2FloatComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2FloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally2FloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2FloatComplex
                scalar multiplier

    @param[out]
    dy          magma_tally2FloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_cblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_cmgeelltmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t num_vecs,
    magma_tally2_int_t nnz_per_row,
    magma_tally2FloatComplex alpha,
    magma_tally2FloatComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2FloatComplex_ptr dx,
    magma_tally2FloatComplex beta,
    magma_tally2FloatComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( magma_tally2FloatComplex ); // num_vecs vectors 
    cmgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_tally2_SUCCESS;
}



