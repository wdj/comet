#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
/*!
 * \file   linalg_cuda.cu
 * \author Wayne Joubert
 * \date   Tue May 15 12:03:55 EDT 2018
 * \brief  Supporting CUDA functions.
 * \note   Copyright (C) 2018 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//-----------------------------------------------------------------------------

#include "linalg_cuda.hh"

//-----------------------------------------------------------------------------

#define TRANSPOSE

// If TRANSPOSE, then copied matrices are stored as A, B^T, for speed on
// the tensor cores.  Otherwise, store as the rest of the code: A^T, B.

#ifdef USE_TC

#include "hip/hip_fp16.h"

//TODO: get rid of multiplier
#define MULTIPLIER 1

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_fp16_kernel_(
  int num_way,
  bool is_sparse,
  bool is_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvlea,
  int nvle,
  int nvle2,
  int nvleX2,
  int nfl,
  int nfl2,
  int nfl2_step,
  int fl2_min,
  GMUInt32* vo) {
//TODO: rename nvle->nvl, nvlea->nvla . . .
//TODO: vi32

  // Two fields (seminibbles) map to two halves of 32-bit word

#ifdef TRANSPOSE
  const int vlX2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int fl2_step = blockIdx.y;
#else
  const int fl2_step = threadIdx.x + blockIdx.x * blockDim.x;
  const int vlX2 = blockIdx.y;
#endif

  if (vlX2 >= nvleX2 || fl2_step >= nfl2_step) {
    return;
  }

  const int i01 = vlX2 % 2; // count either 0 bits or 1 bits.
  const int vl = vlX2 / 2;

  const int fl2 = fl2_min + fl2_step;

  // Output array as floats has nfl/2 rows, as halfs has nfl rows.

//TODO: cast vi_dim0 to size_t
  const GMUInt32* const vi_col = vi + vl * vi_dim0;

  // Pick up two consecutive field values:
  // first field seminibble0, second field seminibble1
  // Set to zero if outside of active range.

  const int nibble = vl<nvlea ? (vi_col[fl2/8] >> (4*(fl2%8))) & 15 : 0;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  // Count number of 0 (or 1) bits in respective seminibble.
  // Determine whether to skip (1,0) null indicator value.

  //CHECK
  const bool skip_10 = is_sparse || (num_way == 3 && ! is_right);

  // Possible counts, represented as FP16.
  const GMUInt16 zero = 0x0000;
  const GMUInt16 one = 0x3c00;
  const GMUInt16 two = 0x4000;
  //const GMUInt16 zero = *(GMUInt16*)&__float2half(0.);
  //const GMUInt16 one = *(GMUInt16*)&__float2half(1.);
  //const GMUInt16 two = *(GMUInt16*)&__float2half(2.);

  const GMUInt16 out0 = seminibble0 == 3*i01     ? two :
                        seminibble0 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble0 == 1         ? one :
                                                   zero;

  const GMUInt16 out1 = seminibble1 == 3*i01     ? two :
                        seminibble1 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble1 == 1         ? one :
                                                   zero;
  // Always keep pair of cols together, corresponding to the two i01 values.
  // Right case: straight copy of cols to cols in sequence.
  // Left case: interleave to make later swizzling of metrics array work:
  // [ A A B B C C D D E E F F ] -> [ A A D D B B E E C C F F]

//TODO: vl_index = is_right ? vl : vl < nvle2 ? 2*vl : 2*vl - nvle + 1
  const int vlX2_index = is_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvle2 ? 2*vl : 2*vl - nvle + 1 );

  const int fl2_index = fl2_step;

#ifdef TRANSPOSE
  const int fl_index_0 = 0 + 2 * fl2_index;
  const int fl_index_1 = 1 + 2 * fl2_index;

  const int vlX2_dim = nvleX2;

//TODO: void* vo, GMUInt16* vo16;
  ((GMUInt16*)vo)[vlX2_index + vlX2_dim * (size_t)fl_index_0] = out0;
  ((GMUInt16*)vo)[vlX2_index + vlX2_dim * (size_t)fl_index_1] = out1;

#else
  const int fl2_dim = nfl2_step;

  // Combine two halfs into one 32-bit value.

  const GMUInt32 out01 = ((GMUInt32)out0) + ( ((GMUInt32)out1) << 16 );

//TODO: GMUInt32* vo32
//TODO: cast vlX2_index to size_t
  vo[fl2_index + fl2_dim * vlX2_index] = out01;
#endif
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_int8_kernel_(
  int num_way,
  bool is_sparse,
  bool is_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvlea,
  int nvle,
  int nvle2,
  int nvleX2,
  int nfl,
  int nfl2,
  int nfl2_step,
  int fl2_min,
  GMUInt16* vo) {

  // Two fields (seminibbles) map to two halves of 16-bit word

#ifdef TRANSPOSE
  const int vlX2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int fl2_step = blockIdx.y;
#else
  const int fl2_step = threadIdx.x + blockIdx.x * blockDim.x;
  const int vlX2 = blockIdx.y;
#endif

  if (vlX2 >= nvleX2 || fl2_step >= nfl2_step) {
    return;
  }

  const int i01 = vlX2 % 2; // count either 0 bits or 1 bits.
  const int vl = vlX2 / 2;

  const int fl2 = fl2_min + fl2_step;

  // Output array as shorts has nfl/2 rows, as chars has nfl rows.

// ISSUE; int32 vs int16
  const GMUInt32* const vi_col = vi + vl * vi_dim0;

  // Pick up two consecutive field values:
  // first field seminibble0, second field seminibble1
  // Set to zero if outside of active range.

  const int nibble = vl<nvlea ? (vi_col[fl2/8] >> (4*(fl2%8))) & 15 : 0; 

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  // Count number of 0 (or 1) bits in respective seminibble.
  // Determine whether to skip (1,0) null indicator value.

  //CHECK
  const bool skip_10 = is_sparse || (num_way == 3 && ! is_right);

  // Possibe counts, represented as Int8.
  const GMUInt8 zero = 0;
  const GMUInt8 one = 1;
  const GMUInt8 two = 2;

  const GMUInt8 out0 = seminibble0 == 3*i01     ? two :
                       seminibble0 == 3*(1-i01) ? zero :
                                      !skip_10  ? one :
                       seminibble0 == 1         ? one :
                                                  zero;

  const GMUInt8 out1 = seminibble1 == 3*i01     ? two :
                       seminibble1 == 3*(1-i01) ? zero :
                                      !skip_10  ? one :
                       seminibble1 == 1         ? one :
                                                  zero;
  // Always keep pair of cols together, corresponding to the two i01 values.
  // Right case: straight copy of cols to cols in sequence.
  // Left case: interleave to make later swizzling of metrics array work:
  // [ A A B B C C D D E E F F ] -> [ A A D D B B E E C C F F]

  const int vlX2_index = is_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvle2 ? 2*vl : 2*vl - nvle + 1 );

  const int fl2_index = fl2_step;

#ifdef TRANSPOSE
  //CHECK
  const int fl_index_0 = 0 + 2 * fl2_index;
  const int fl_index_1 = 1 + 2 * fl2_index;

  const int vlX2_dim = nvleX2;

  ((GMUInt8*)vo)[ vlX2_index + vlX2_dim * fl_index_0 ] = out0;
  ((GMUInt8*)vo)[ vlX2_index + vlX2_dim * fl_index_1 ] = out1;
#else
  const int fl2_dim = nfl2_step;

  // Combine two chars into one short int.

  const GMUInt16 out01 = ((GMUInt16)out0) + ( ((GMUInt16)out1) << 8 );

  vo[ fl2_index + fl2_dim * vlX2_index ] = out01;
#endif
}

//-----------------------------------------------------------------------------
// Convert matrix stored as packed 2-bit values into matrix of FP16 (or Int8).

void gm_tc_buf_write_(
  bool is_right,
  int I_max,
  int I_max_dim,
  int nvl,
  int npvfl,
  int npvfl_step,
  int pvfl_min,
  void* vi_ptr,
  GMEnv* env) {
  GMInsist(env && vi_ptr);
  GMInsist(I_max_dim >= 0 && I_max_dim <= nvl);
  GMInsist(I_max >= 0 && I_max <= I_max_dim);
  GMInsist(nvl >= 0);
  GMInsist(npvfl >= 0);
//TODO: more assertions
//TODO: void: vi, int32* vi32

  const bool is_int8 = env->tc == 2;

  const int nvle = is_right ? nvl : I_max_dim; // effective nvl dimension
  const int nvle2 = nvle / 2;
  const int nvleX2 = 2 * nvle;
  const int nvlea = is_right ? nvl : I_max; // num active nvle; others zeroed

  const int nfl = npvfl * 64;
  const int nfl2 = nfl / 2;
  const int nfl_step = npvfl_step * 64;
  const int nfl2_step = nfl_step / 2;
  const int fl_min = pvfl_min * 64;
  const int fl2_min = fl_min / 2;

// ISSUE: is this right for int16 case
  const int vi_dim0 = npvfl * 4; // 4 = sizeof(doublecomplex) / sizeof(int32)

  GMInsistInterface(env, nvle % 2 == 0 && nvl % 2 == 0 &&
                    "tc method here requires num_vector_local multiple of 2.");

  const int threadblocksize = 256;
#ifdef TRANSPOSE
  const int num_threadblocks_0 = gm_ceil_i8(nvleX2, threadblocksize);
  const int num_threadblocks_1 = nfl2_step;
#else
  const int num_threadblocks_0 = gm_ceil_i8(nfl2_step, threadblocksize);
  const int num_threadblocks_1 = nvleX2;
#endif

  const void* tc_buf = is_right ? env->tc_buf_right : env->tc_buf_left;

  if (! is_int8) {

    gm_tc_buf_write_fp16_kernel_<<<
        dim3(num_threadblocks_0, num_threadblocks_1, 1),
        dim3(threadblocksize, 1, 1),
        0,
        env->stream_compute_>>>(
      GMEnv_num_way(env),
      env->sparse,
      is_right,
      (GMUInt32*)vi_ptr,
      vi_dim0,
      nvlea,
      nvle,
      nvle2,
      nvleX2,
      nfl,
      nfl2,
      nfl2_step,
      fl2_min,
      (GMUInt32*)tc_buf);

  } else {

    gm_tc_buf_write_int8_kernel_<<<
        dim3(num_threadblocks_0, num_threadblocks_1, 1),
        dim3(threadblocksize, 1, 1),
        0,
        env->stream_compute_>>>(
      GMEnv_num_way(env),
      env->sparse,
      is_right,
      (GMUInt32*)vi_ptr,
      vi_dim0,
      nvlea,
      nvle,
      nvle2,
      nvleX2,
      nfl,
      nfl2,
      nfl2_step,
      fl2_min,
      (GMUInt16*)tc_buf);

  }

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
// Call tensor core enabled cuBLAS function to tally bits for CCC.

void gm_tc_solve_(
  bool is_first,
  int nvll,
  int nvl,
  int npvfl_step,
  void* dA,
  void* dB,
  void* dC,
  GMEnv* env) {
  GMInsist(env && dA && dB && dC);
  GMInsist(nvll >= 0);
  GMInsist(nvl >= 0);
  GMInsist(nvll <= nvl);
  GMInsist(npvfl_step >= 0);
  GMInsist(env->tc == 1 || env->tc == 2);

  const int nfl_step = npvfl_step * 64;

  const int m = 2 * nvll; // metrics array dim
  const int n = 2 * nvl; // metrics array dim
  const int k = nfl_step; // vectors array (as halfs/bytes) dim

  const float alpha = 1;
  const float beta = is_first ? 0 : 1;

  const bool is_int8 = env->tc == 2;

  // See https://devblogs.nvidia.com/programming-tensor-cores-cuda-9/
  // "Invoke the GEMM, ensuring k, lda, ldb, and ldc are all multiples of 8, 
  // and m is a multiple of 4"
  // "GEMMs that do not satisfy the above rules will fall back
  // to a non-Tensor Core implementation"
  // See also https://docs.nvidia.com/cuda/cublas/index.html#cublas-gemmEx

  GMInsist(k % 8 == 0); // nfl is derived from padded-up npvfl, so always ok.
  GMInsist(m % 8 == 0); // need I_max_dim % 4 == 0
  GMInsist(n % 8 == 0); // need nvl % 4 == 0

  // Make BLAS call.

  hipblasStatus_t status = hipblasGemmEx(
    env->cublas_handle,
#ifdef TRANSPOSE
    HIPBLAS_OP_N, HIPBLAS_OP_T,
#else
    HIPBLAS_OP_T, HIPBLAS_OP_N,
#endif
    m, n, k,
    &alpha,
    env->tc_buf_left, is_int8 ? HIP_R_8I : HIP_R_16F,
#ifdef TRANSPOSE
    m,
#else
    k,
#endif
    env->tc_buf_right, is_int8 ? HIP_R_8I : HIP_R_16F,
#ifdef TRANSPOSE
    n,
#else
    k,
#endif
    &beta,
    dC, HIP_R_32F, m,
    HIP_R_32F,
#ifdef TRANSPOSE
    //CUBLAS_GEMM_ALGO3_TENSOR_OP // best timing, for cuda 9.1.85, transpose
    //CUBLAS_GEMM_DFALT_TENSOR_OP // good timing, for cuda 9.2.88, transpose
    CUBLAS_GEMM_ALGO4_TENSOR_OP // best timing, for cuda 9.2.88, transpose
#else
    CUBLAS_GEMM_ALGO4_TENSOR_OP // best timing, for cuda 9.1.85, non-transpose
#endif
    //CUBLAS_GEMM_DFALT_TENSOR_OP
  );

  if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
    printf("Error: HIPBLAS_STATUS_NOT_INITIALIZED\n");
  } else if (status == HIPBLAS_STATUS_ARCH_MISMATCH) {
    printf("Error: HIPBLAS_STATUS_ARCH_MISMATCH\n");
  } else if (status == HIPBLAS_STATUS_NOT_SUPPORTED) {
    printf("Error: HIPBLAS_STATUS_NOT_SUPPORTED\n");
  } else if (status == HIPBLAS_STATUS_INVALID_VALUE) {
    printf("Error: HIPBLAS_STATUS_INVALID_VALUE\n");
  } else if (status == HIPBLAS_STATUS_EXECUTION_FAILED) {
    printf("Error: HIPBLAS_STATUS_EXECUTION_FAILED\n");
  }

  GMInsist(status == HIPBLAS_STATUS_SUCCESS);

  env->ops_local += 2 * m * (double)n * (double)k;
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_fix_metrics_kernel_(
  int nvl,
  int nvll,
  int nvll2,
  float* vo,
  float multiplier) {

  // Row and column of metrics array.

  const int thread_r = threadIdx.x + blockIdx.x * blockDim.x;
  const int thread_c = blockIdx.y;

  if (thread_r >= nvll2 || thread_c >= nvl) {
    return;
  }

  // Considered as an array of floats, array is 2*nvl rows X 2*nvl cols.
  // Each thread manipulates a block of 4 rows and 2 cols.
  // Thus the dimensions of the metrics array in blocks is nvll2 X nvl.
  // Each block viewed as an array of doubles is 2 X 2.

  // Two col numbers being processed of this (float) array.

//TODO: use only single pointer instead . . . ?

//TODO: make size_t; cast 4*nvll to size_t
  const int fc_offset0 = thread_c * (4*nvll);
  const int fc_offset1 = thread_c * (4*nvll) + 2*nvll;
//TODO: fcr_offset0/1

  // Read the 8 floats.

//TODO: variable fvo
  const float f00 = vo[fc_offset0+0+4*thread_r] * multiplier;
  const float f01 = vo[fc_offset0+1+4*thread_r] * multiplier;
  const float f02 = vo[fc_offset0+2+4*thread_r] * multiplier;
  const float f03 = vo[fc_offset0+3+4*thread_r] * multiplier;

  const float f10 = vo[fc_offset1+0+4*thread_r] * multiplier;
  const float f11 = vo[fc_offset1+1+4*thread_r] * multiplier;
  const float f12 = vo[fc_offset1+2+4*thread_r] * multiplier;
  const float f13 = vo[fc_offset1+3+4*thread_r] * multiplier;

  // Apply the permutation:

  // [ A  A ]  ->  [ A  B ]
  // [ A  A ]  ->  [ A  B ]
  // [ B  B ]  ->  [ A  B ]
  // [ B  B ]  ->  [ A  B ]

  const float f00p = f00;
  const float f01p = f01;

  const float f02p = f10;
  const float f03p = f11;

  const float f10p = f02;
  const float f11p = f03;

  const float f12p = f12;
  const float f13p = f13;

  // Use helper value to move value to upper half of mantissa.

  const double shifter = (((GMUInt32)1) << GM_TALLY1_MAX_VALUE_BITS);

  // Pack two 25-bit integers into mantissa of double.

// TODO: explicitly cast float values to double
  const double d00 = f00p + f02p * shifter;
  const double d01 = f01p + f03p * shifter;

  const double d10 = f10p + f12p * shifter;
  const double d11 = f11p + f13p * shifter;

  // Overwrite block with the new values.
  // All is isolated to a single thread, should be thread safe.

//TODO: make size_t; cast 2*nvll to size_t
  const int dc_offset0 = thread_c * (2*nvll);
  const int dc_offset1 = thread_c * (2*nvll) + nvll;
//TODO: dcr_offset0/1

//TODO: variable dvo
  ((double*)vo)[dc_offset0+0+2*thread_r] = d00;
  ((double*)vo)[dc_offset0+1+2*thread_r] = d01;

  ((double*)vo)[dc_offset1+0+2*thread_r] = d10;
  ((double*)vo)[dc_offset1+1+2*thread_r] = d11;
}

//-----------------------------------------------------------------------------
// Swizzle/cast values from the CUBLAS call into required double complex format.

void gm_tc_fix_metrics_(
  int nvll,
  int nvl,
  void* vo_ptr,
  GMEnv* env) {
  GMInsist(env && vo_ptr);
  GMInsist(nvll >= 0);
  GMInsist(nvl >= 0);
  GMInsist(nvll <= nvl);

  const int nvll2 = nvll / 2;

  const int threadblocksize = 256;
  const int vll2_threadblocks = gm_ceil_i8(nvll2, threadblocksize);

  const bool is_int8 = env->tc == 2;

  gm_tc_fix_metrics_kernel_<<<
      dim3(vll2_threadblocks, nvl, 1),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
    nvl,
    nvll,
    nvll2,
    (float*)vo_ptr,
    is_int8 ? 1 : MULTIPLIER * MULTIPLIER
  );

  GMEnv_cuda_last_call_succeeded(env);
}
#endif

//-----------------------------------------------------------------------------

void gm_tc_gemm_start(int m, int n, int k,
                      void* dA, int ldda,
                      void* dB, int lddb,
                      void* dC, int lddc,
                      GMEnv* env) {
  GMInsist(dA && dB && dC && env);
  GMInsist(m >= 0 && n >= 0 && k >= 0);
  GMInsist(ldda >= 0 && lddb >= 0 && lddc >= 0);
  GMInsist(k <= ldda);
  GMInsist(k <= lddb);
  GMInsist(m <= lddc);
  GMInsist(env->tc);
  GMInsist(GMEnv_metric_type(env) == GM_METRIC_TYPE_CCC);
  GMInsist(GMEnv_compute_method(env) == GM_COMPUTE_METHOD_GPU);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  GMInsist(deviceProp.major >= 7);

#ifdef USE_TC
  const int I_max = m;
  const int I_max_dim = lddc;
  const int nvll = I_max_dim; // effective nvl for left matrix
  const int nvl = n;
  const int npvfl = k;

  const int num_steps = env->num_tc_steps;

  for (int step_num = 0; step_num < num_steps; ++step_num) {
    const int pvfl_min = ((step_num+0) * npvfl) / num_steps;
    const int pvfl_max = ((step_num+1) * npvfl) / num_steps;
    const int npvfl_step = pvfl_max - pvfl_min;
    GMAssert(npvfl_step <= env->npvfl_step_max);

    if (npvfl_step == 0) {
      continue;
    }

    const bool left_matrix = false; // A
    const bool right_matrix = true; // B
    gm_tc_buf_write_(left_matrix, I_max, I_max_dim, nvl, npvfl,
                     npvfl_step, pvfl_min, dA, env);
    gm_tc_buf_write_(right_matrix, I_max, I_max_dim, nvl, npvfl,
                     npvfl_step, pvfl_min, dB, env);

    //for (int i=0; i<20; ++i)
    gm_tc_solve_(pvfl_min==0, nvll, nvl, npvfl_step, dA, dB, dC, env);
  }

  gm_tc_fix_metrics_(nvll, nvl, dC, env);

#else
  GMInsistInterface(env,
                    false && "TC option not implemented for this platform.");
#endif
}

//-----------------------------------------------------------------------------
