#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally3.h"

#define BLOCK_SIZE 512


__global__ void 
zmgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        magma_tally3DoubleComplex alpha, 
        magma_tally3DoubleComplex * dval, 
        magma_tally3_index_t * dcolind,
        magma_tally3DoubleComplex * dx,
        magma_tally3DoubleComplex beta, 
        magma_tally3DoubleComplex * dy)
{
    extern __shared__ magma_tally3DoubleComplex dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally3_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally3DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally3_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally3_int_t
                number of rows in A

    @param[in]
    n           magma_tally3_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_tally3_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_tally3DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally3DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally3Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally3DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally3DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally3DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_zblas
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_zmgeelltmv(
    magma_tally3_trans_t transA,
    magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3_int_t num_vecs,
    magma_tally3_int_t nnz_per_row,
    magma_tally3DoubleComplex alpha,
    magma_tally3DoubleComplex_ptr dval,
    magma_tally3Index_ptr dcolind,
    magma_tally3DoubleComplex_ptr dx,
    magma_tally3DoubleComplex beta,
    magma_tally3DoubleComplex_ptr dy,
    magma_tally3_queue_t queue )
{
    dim3 grid( magma_tally3_ceildiv( m, BLOCK_SIZE ) );
    magma_tally3_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( magma_tally3DoubleComplex ); // num_vecs vectors 
    zmgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_tally3_SUCCESS;
}



