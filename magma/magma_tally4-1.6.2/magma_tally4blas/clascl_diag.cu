#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlascl_diag.cu normal z -> c, Fri Jan 30 19:00:09 2015
*/
#include "common_magma_tally4.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
clascl_diag_lower(int m, int n, magma_tally4FloatComplex_const_ptr D, int ldd, 
                                      magma_tally4FloatComplex_ptr A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] /= D[j+j*ldd];
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
clascl_diag_upper(int m, int n, magma_tally4FloatComplex_const_ptr D, int ldd, 
                                      magma_tally4FloatComplex_ptr A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] /= D[ind+ind*ldd];
    }
}


/**
    Purpose
    -------
    CLASCL2 scales the M by N complex matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_tally4_type_t
            TYPE indices the storage type of the input matrix A.
            = Magma_tally4Full:   full matrix.
            = Magma_tally4Lower:  lower triangular matrix.
            = Magma_tally4Upper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA_tally4 does not currently support.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in]
    dD      REAL vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    \param[in,out]
    dA      COMPLEX array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @ingroup magma_tally4_caux2
    ********************************************************************/
extern "C" void
magma_tally4blas_clascl_diag_q(
    magma_tally4_type_t type, magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4FloatComplex_const_ptr dD, magma_tally4_int_t lddd, 
          magma_tally4FloatComplex_ptr dA, magma_tally4_int_t ldda, 
    magma_tally4_int_t *info, magma_tally4_queue_t queue )
{
    *info = 0;
    if ( type != Magma_tally4Lower && type != Magma_tally4Upper && type != Magma_tally4Full )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -5;
    
    if (*info != 0) {
        magma_tally4_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    if (type == Magma_tally4Lower) {
        clascl_diag_lower <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
    else if (type == Magma_tally4Upper) {
        clascl_diag_upper <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
}


/**
    @see magma_tally4blas_clascl2_q
    @ingroup magma_tally4_caux2
    ********************************************************************/
extern "C" void
magma_tally4blas_clascl_diag(
    magma_tally4_type_t type, magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4FloatComplex_const_ptr dD, magma_tally4_int_t lddd, 
          magma_tally4FloatComplex_ptr dA, magma_tally4_int_t ldda, 
    magma_tally4_int_t *info )
{
    magma_tally4blas_clascl_diag_q( type, m, n, dD, lddd, dA, ldda, info, magma_tally4_stream );
}
