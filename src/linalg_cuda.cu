#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
/*!
 * \file   linalg_cuda.cu
 * \author Wayne Joubert
 * \date   Tue May 15 12:03:55 EDT 2018
 * \brief  Supporting CUDA functions.
 * \note   Copyright (C) 2018 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//-----------------------------------------------------------------------------

#include "linalg_cuda.hh"

//-----------------------------------------------------------------------------

#if 0
__device__ static const GMUInt32 MGEMM2_table1[] = {
  0x00000000,
  0x00003c00,
  0x00003c00,
  0x00004000,

  0x3c000000,
  0x3c003c00,
  0x3c003c00,
  0x3c004000,

  0x3c000000,
  0x3c003c00,
  0x3c003c00,
  0x3c004000,

  0x40000000,
  0x40003c00,
  0x40003c00,
  0x40004000
  };

__device__ static const GMUInt32 MGEMM2_table0[] = {
  0x40004000,
  0x40003c00,
  0x40003c00,
  0x40000000,

  0x3c004000,
  0x3c003c00,
  0x3c003c00,
  0x3c000000,

  0x3c004000,
  0x3c003c00,
  0x3c003c00,
  0x3c000000,

  0x00004000,
  0x00003c00,
  0x00003c00,
  0x00000000
  };

struct MGEMM2 {
  static const GMUInt32* table[2];
};

const GMUInt32* MGEMM2::table[] = {MGEMM2_table0, MGEMM2_table1};

#endif

#define TRANSPOSE

//-----------------------------------------------------------------------------

// https://docs.nvidia.com/cuda/cublas/index.html#cublas-gemmEx

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_fp16_(
  int num_way,
  bool is_sparse,
  int left_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvl_in,
  int nvl,
  int nvl2,
  int nfl,
  int nfl2,
  GMUInt32* vo) {

  // Two fields (seminibbles) map to two half words of 32-bit word

  const int fl2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int i01 = blockIdx.y; // count either 0 bits or 1 bits.
  const int vl = blockIdx.z;
  const int vl_in = vl <= nvl_in-1 ? vl : nvl_in-1;

  if (fl2 >= nfl2) {
    return;
  }

  if (vl < 0 || vl >= nvl) {
    return;
  }

  // Output array as floats has nfl/2 rows, as halfs has nfl rows.

  // Column offset in input array.

  const GMUInt32 * const vi_col = vi + vl_in * vi_dim0;

  // Pick up two consecutive field values.
  // NOTE: first field seminibble0, second field seminibble1

  const int nibble = (vi_col[fl2/8] >> (4 * (fl2%8))) & 15;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  // Count number of 0 (or 1) bits in respective seminibble.

  const bool is_right = left_right != 0;

  const bool skip_10 = is_sparse || (num_way == 3 && ! is_right);

  const GMUInt16 zero = 0x0000;
  const GMUInt16 one = 0x3c00;
  const GMUInt16 two = 0x4000;

  const GMUInt16 out0 = seminibble0 == 3*i01     ? two :
                        seminibble0 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble0 == 1         ? one :
                                                   zero;

  const GMUInt16 out1 = seminibble1 == 3*i01     ? two :
                        seminibble1 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble1 == 1         ? one :
                                                   zero;
  // Combine two halfs into one float.

  const GMUInt32 out01 = ((GMUInt32)out0 + ( ((GMUInt32out1) << 16);

  // Always keep pair of cols together, corresponding to the two i01 values.
  // Right case: straight copy of cols to cols in sequence.
  // Left case: interleave to make swizzling of result array work:
  // [ A A B B C C D D E E F F ] -> [ A A D D B B E E C C F F]

  const int vlX2_index = is_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvl2 ? 2*vl : 2*vl - nvl + 1 );

  const int fl2_index = fl2;
  const int fl2_dim = nfl2;

  vo[ fl2_index + fl2_dim * vlX2_index ] = out01;

#ifdef TRANSPOSE
// switch?
  const int fl_index_0 = 0 + 2 * fl2_index;
  const int fl_index_1 = 1 + 2 * fl2_index;

  const int vlX2_dim = 2 * nvl;

  ((GMUInt16*)vo)[ vlX2_index + vlX2_dim * fl_index_0 ] = out0;
  ((GMUInt16*)vo)[ vlX2_index + vlX2_dim * fl_index_1 ] = out1;
#endif

//if (fl2==0) printf("%s vec %i field %i  %i\n", left_right ? "r" : "l", vl, 2*fl2+0, seminibble0);
//if (fl2==0) printf("lr %i vec %i field %i  %i\n", left_right, vl, 2*fl2+1, seminibble1);
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_int8_(
  int num_way,
  bool is_sparse,
  int left_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvl_in,
  int nvl,
  int nvl2,
  int nfl,
  int nfl2,
  GMUInt16* vo) {

  // Two fields (seminibbles) map to two half words of 32-bit word

  const int fl2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int i01 = blockIdx.y; // count either 0 bits or 1 bits.
  const int vl = blockIdx.z;
  const int vl_in = vl <= nvl_in-1 ? vl : nvl_in-1;

  if (fl2 >= nfl2) {
    return;
  }

  if (vl < 0 || vl >= nvl) {
    return;
  }

  // Output array as shorts has nfl/2 rows, as chars has nfl rows.

  // Column offset in input array.

  const GMUInt32 * const vi_col = vi + vl_in * vi_dim0;

  // Pick up two consecutive field values.
  // NOTE: first field seminibble0, second field seminibble1

  const int nibble = (vi_col[fl2/8] >> (4 * (fl2%8))) & 15;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  // Count number of 0 (or 1) bits in respective seminibble.

  const bool is_right = left_right != 0;

  // this needs work
  const bool skip_10 = is_sparse || (num_way == 3 && ! is_right);

  const GMUInt8 zero = 0;
  const GMUInt8 one = 1;
  const GMUInt8 two = 2;

  const GMUInt8 out0 = seminibble0 == 3*i01     ? two :
                       seminibble0 == 3*(1-i01) ? zero :
                                      !skip_10  ? one :
                       seminibble0 == 1         ? one :
                                                  zero;

  const GMUInt8 out1 = seminibble1 == 3*i01     ? two :
                       seminibble1 == 3*(1-i01) ? zero :
                                      !skip_10  ? one :
                       seminibble1 == 1         ? one :
                                                  zero;
  // Combine two chars into one short int.

  const GMUInt16 out01 = ((GMUInt16)out0) + ( ((GMUInt16)out1) << 8);

  // Always keep pair of cols together, corresponding to the two i01 values.
  // Right case: straight copy of cols to cols in sequence.
  // Left case: interleave to make swizzling of result array work:
  // [ A A B B C C D D E E F F ] -> [ A A D D B B E E C C F F]

  const int vlX2_index = is_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvl2 ? 2*vl : 2*vl - nvl + 1 );

  const int fl2_index = fl2;
  const int fl2_dim = nfl2;

  vo[ fl2_index + fl2_dim * vlX2_index ] = out01;

// byte address {0,1} + 2 * fl2 

#ifdef TRANSPOSE
// switch?
  const int fl_index_0 = 0 + 2 * fl2_index;
  const int fl_index_1 = 1 + 2 * fl2_index;

  const int vlX2_dim = 2 * nvl;

  ((GMUInt8*)vo)[ vlX2_index + vlX2_dim * fl_index_0 ] = out0;
  ((GMUInt8*)vo)[ vlX2_index + vlX2_dim * fl_index_1 ] = out1;
#endif

//  if (fl2==0 && vl==0 && i01==0) printf("//////  %i\n", (int) *(GMUInt16*)vo);

//  if (fl2==0)
//    printf("////// %s %i %i  %i %i %i %i  %i %i %i  %i %i\n", left_right ? "r" : "l", (int)nvl, (int)nfl,  fl0, fl1, i01, vl,  fl0, fl1, col,  (int)(col+2*nvl*fl0), (int)(col+2*nvl*fl1));

//printf("%i %i\n", (int)out0, (int)out1);

//if (fl2==0) printf("%s i01 %i vec %i field %i  %i  %i\n", left_right ? "r" : "l", i01, vl, 2*fl2+0, seminibble0, (int)out0);

//if (seminibble0) printf("vec %i field %i  %i\n", vl, 2*fl2+0, seminibble0);
//if (seminibble1) printf("vec %i field %i  %i\n", vl, 2*fl2+1, seminibble1);
}

//-----------------------------------------------------------------------------
// Convert packed-bits fields of input vectors into halfs (or chars).

void gm_tc_buf_write(
  int left_right,
  int I_max,
  int num_vector_local,
  int num_packedval_field_local,
  void* bufd,
  GMEnv* env) {
  GMInsist(left_right == 0 || left_right == 1);
  GMInsist(env && bufd);
  GMInsist(I_max >= 0);
  GMInsist(num_vector_local >= 0);
  GMInsist(num_packedval_field_local >= 0);
  GMInsist(GMEnv_metric_type(env) == GM_METRIC_TYPE_CCC);
  GMInsist(GMEnv_compute_method(env) == GM_COMPUTE_METHOD_GPU);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  GMInsist(deviceProp.major >= 7);

  //GMInsistInterface(env, GMEnv_num_way(env) == GM_NUM_WAY_2 &&
  //                  "Not yet implemented.");
  //GMInsistInterface(env, !env->sparse && "Not yet implemented.");

  const int nvl = num_vector_local;
  const int nvl2 = nvl / 2;
  const int npvfl = num_packedval_field_local;
  const int nfl = npvfl * 64;
  const int nfl2 = nfl / 2;

  GMInsistInterface(env, num_vector_local % 2 == 0 &&
                    "tc method requires num_vector_local multiple of 2.");

  const int threadblocksize = 256;
  const int fl2_threadblocks = (nfl2+threadblocksize-1) / threadblocksize;

  const bool is_right = left_right != 0;
  const bool is_int8 = env->tc == 2;

  if (is_int8) {

    gm_tc_buf_write_int8_<<<
      dim3(fl2_threadblocks, 2, nvl),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
      GMEnv_num_way(env),
      env->sparse,
      left_right,
      (GMUInt32*)bufd,
      npvfl * 4,
      left_right ? nvl : I_max,
      nvl,
      nvl2,
      nfl,
      nfl2,
      is_right ? (GMUInt16*)env->tc_buf_right : (GMUInt16*)env->tc_buf_left);

  } else {

    gm_tc_buf_write_fp16_<<<
      dim3(fl2_threadblocks, 2, nvl),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
      GMEnv_num_way(env),
      env->sparse,
      left_right,
      (GMUInt32*)bufd,
      npvfl * 4,
      left_right ? nvl : I_max,
      nvl,
      nvl2,
      nfl,
      nfl2,
      is_right ? (GMUInt32*)env->tc_buf_right : (GMUInt32*)env->tc_buf_left);

  }

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
// Call tensor core enabled cuBLAS function to tally bits for CCC.

void gm_tc_solve(
  int I_max,
  int num_vector_local,
  int num_packedval_field_local,
  void* dA,
  int ldda,
  void* dB,
  int lddb,
  void* dC,
  int lddc,
  GMEnv* env) {
  GMInsist(env && dA && dB && dC);
  GMInsist(I_max >= 0);
  GMInsist(num_vector_local >= 0);
  GMInsist(num_packedval_field_local >= 0);
  GMInsist(ldda >= 0 && lddb >= 0 && lddc >= 0);
  GMInsist(env->tc == 1 || env->tc == 2);

  const int nvl = num_vector_local;
  const int npvfl = num_packedval_field_local;
  const int nfl = npvfl * 64;

  const int m = 2 * nvl; // metrics array dim
  const int n = 2 * nvl; // metrics array dim
  const int k = nfl; // vectors array (as halfs) dim

  const float alpha = 1;
  const float beta = 0;

  const bool is_int8 = env->tc == 2;

  GMInsist(k % 8 == 0); // nfl is derived from padded-up npvfl, so ok.
  GMInsist(m % 8 == 0); // need nvl % 4 == 0

#if 0
  hipblasStatus_t status = cublasSgemmEx(
    env->cublas_handle,
    HIPBLAS_OP_T, HIPBLAS_OP_N,
    m, n, k,
    &alpha,
    env->tc_buf_left, HIP_R_16F, k,
    env->tc_buf_right, HIP_R_16F, k,
    &beta,
    dC, HIP_R_32F, m);
#endif

  hipblasStatus_t status = hipblasGemmEx(
    env->cublas_handle,
#ifdef TRANSPOSE
    HIPBLAS_OP_N, HIPBLAS_OP_T,
#else
    HIPBLAS_OP_T, HIPBLAS_OP_N,
#endif
    m, n, k,
    &alpha,
    env->tc_buf_left, is_int8 ? HIP_R_8I : HIP_R_16F,
#ifdef TRANSPOSE
    m,
#else
    k,
#endif
    env->tc_buf_right, is_int8 ? HIP_R_8I : HIP_R_16F,
#ifdef TRANSPOSE
    n,
#else
    k,
#endif
    &beta,
    dC, HIP_R_32F, m,
    HIP_R_32F,
    CUBLAS_GEMM_DFALT_TENSOR_OP);
    //CUBLAS_GEMM_ALGO4_TENSOR_OP); // best, for cuda 9.1.85 non-transpose

  if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
    printf("Error: HIPBLAS_STATUS_NOT_INITIALIZED\n");
  }
  if (status == HIPBLAS_STATUS_ARCH_MISMATCH) {
    printf("Error: HIPBLAS_STATUS_ARCH_MISMATCH\n");
  }
  if (status == HIPBLAS_STATUS_NOT_SUPPORTED) {
    printf("Error: HIPBLAS_STATUS_NOT_SUPPORTED\n");
  }
  if (status == HIPBLAS_STATUS_INVALID_VALUE) {
    printf("Error: HIPBLAS_STATUS_INVALID_VALUE\n");
  }
  if (status == HIPBLAS_STATUS_EXECUTION_FAILED) {
    printf("Error: HIPBLAS_STATUS_EXECUTION_FAILED\n");
  }

  GMInsist(status == HIPBLAS_STATUS_SUCCESS);

  env->ops_local += 2 * m * (double)n * (double)k;
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_fix_metrics_(
  int nvl,
  int nvl2,
  float* bufd) {

  // Row and column of metrics array.

  const int thread_r = threadIdx.x + blockIdx.x * blockDim.x;
  const int thread_c = blockIdx.y;

  if (thread_r >= nvl2 || thread_c >= nvl) {
    return;
  }

  // Considered as an array of floats, array is 2*nvl rows X 2*nvl cols.
  // Each thread manipulates a block of 4 rows and 2 cols.
  // Thus the dimensions of the metrics array in blocks is nvl2 X nvl.
  // Each block viewed as an array of doubles is 2 X 2.

  // Two col numbers being processed of this (float) array.

  const int fc0 = thread_c * (4*nvl);
  const int fc1 = thread_c * (4*nvl) + 2*nvl;

  // Read the 8 floats.

  const float f00 = bufd[fc0+0+4*thread_r];
  const float f01 = bufd[fc0+1+4*thread_r];
  const float f02 = bufd[fc0+2+4*thread_r];
  const float f03 = bufd[fc0+3+4*thread_r];

  const float f10 = bufd[fc1+0+4*thread_r];
  const float f11 = bufd[fc1+1+4*thread_r];
  const float f12 = bufd[fc1+2+4*thread_r];
  const float f13 = bufd[fc1+3+4*thread_r];

  // Apply the permutation.
  //
  // [ A  A ]      [ A  B ]
  // [ A  A ]  ->  [ A  B ]
  // [ B  B ]  ->  [ A  B ]
  // [ B  B ]      [ A  B ]

  const float f00p = f00;
  const float f01p = f01;

  const float f02p = f10;
  const float f03p = f11;

  const float f10p = f02;
  const float f11p = f03;

  const float f12p = f12;
  const float f13p = f13;

  // Pack two 25-bit integers into mantissa of double.

  const double shifter = (((GMUInt32)1)<<GM_TALLY1_MAX_VALUE_BITS);

  const double d00 = f00p + f02p * shifter;
  const double d01 = f01p + f03p * shifter;

  const double d10 = f10p + f12p * shifter;
  const double d11 = f11p + f13p * shifter;

  // Overwrite block with the new values.

  const int dc0 = thread_c * (2*nvl);
  const int dc1 = thread_c * (2*nvl) + nvl;

  ((double*)bufd)[dc0+0+2*thread_r] = d00;
  ((double*)bufd)[dc0+1+2*thread_r] = d01;

  ((double*)bufd)[dc1+0+2*thread_r] = d10;
  ((double*)bufd)[dc1+1+2*thread_r] = d11;

//printf("%i %i %f %f %f %f\n", 2*thread_r+0, thread_c, f00p, f01p, f02p, f03p);
//printf("%i %i %f %f %f %f\n", 2*thread_r+1, thread_c, f10p, f11p, f12p, f13p);
}

//-----------------------------------------------------------------------------
// Swizzle/cast values from the CUBLAS call into required double complex format.

void gm_tc_fix_metrics(
  int I_max,
  int num_vector_local,
  void* bufd,
  GMEnv* env) {
  GMInsist(env && bufd);
  GMInsist(I_max >= 0);
  GMInsist(num_vector_local >= 0);

  const int nvl = num_vector_local;
  const int nvl2 = nvl / 2;

  const int threadblocksize = 256;
  const int vl2_threadblocks = (nvl2+threadblocksize-1) / threadblocksize;

  gm_tc_fix_metrics_<<<
    dim3(vl2_threadblocks, nvl, 1),
    dim3(threadblocksize, 1, 1),
    0,
    env->stream_compute_>>>(
    nvl,
    nvl2,
    (float*)bufd
  );

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
