#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zswapdblk.cu normal z -> c, Fri Jan 30 19:00:09 2015

*/
#include "common_magma_minproduct.h"


/*********************************************************/
/*
 *  Swap diagonal blocks of two matrices.
 *  Each thread block swaps one diagonal block.
 *  Each thread iterates across one row of the block.
 */

__global__ void 
cswapdblk_kernel( int nb,
                  magma_minproductFloatComplex *dA, int ldda, int inca,
                  magma_minproductFloatComplex *dB, int lddb, int incb )
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    dA += tx + bx * nb * (ldda + inca);
    dB += tx + bx * nb * (lddb + incb);

    magma_minproductFloatComplex tmp;

    #pragma unroll
    for( int i = 0; i < nb; i++ ){
        tmp        = dA[i*ldda];
        dA[i*ldda] = dB[i*lddb];
        dB[i*lddb] = tmp;
    }
}


/**
    Purpose
    -------
    cswapdblk swaps diagonal blocks of size nb x nb between matrices
    dA and dB on the GPU. It swaps nblocks = n/nb blocks.
    For i = 1 .. nblocks, submatrices
    dA( i*nb*inca, i*nb ) and
    dB( i*nb*incb, i*nb ) are swapped.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns of the matrices dA and dB.  N >= 0.

    @param[in]
    nb      INTEGER
            The size of diagonal blocks.
            NB > 0 and NB <= maximum threads per CUDA block (512 or 1024).

    @param[in,out]
    dA      COMPLEX array, dimension (LDDA,N)
            The matrix dA.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.
            LDDA >= (nblocks - 1)*nb*inca + nb.

    @param[in]
    inca    INTEGER
            The row increment between diagonal blocks of dA. inca >= 0. For example,
            inca = 1 means blocks are stored on the diagonal at dA(i*nb, i*nb),
            inca = 0 means blocks are stored side-by-side    at dA(0,    i*nb).

    @param[in,out]
    dB      COMPLEX array, dimension (LDDB,N)
            The matrix dB.

    @param[in]
    lddb    INTEGER
            The leading dimension of the array db.
            LDDB >= (nblocks - 1)*nb*incb + nb.

    @param[in]
    incb    INTEGER
            The row increment between diagonal blocks of dB. incb >= 0. See inca.
    
    @param[in]
    queue   magma_minproduct_queue_t
            Queue to execute in.

    @ingroup magma_minproduct_caux2
    ********************************************************************/
extern "C" void 
magma_minproductblas_cswapdblk_q(
    magma_minproduct_int_t n, magma_minproduct_int_t nb,
    magma_minproductFloatComplex_ptr dA, magma_minproduct_int_t ldda, magma_minproduct_int_t inca,
    magma_minproductFloatComplex_ptr dB, magma_minproduct_int_t lddb, magma_minproduct_int_t incb,
    magma_minproduct_queue_t queue )
{
    magma_minproduct_int_t nblocks = n / nb;
    
    magma_minproduct_int_t info = 0;
    if (n < 0) {
        info = -1;
    } else if (nb < 1 || nb > 1024) {
        info = -2;
    } else if (ldda < (nblocks-1)*nb*inca + nb) {
        info = -4;
    } else if (inca < 0) {
        info = -5;
    } else if (lddb < (nblocks-1)*nb*incb + nb) {
        info = -7;
    } else if (incb < 0) {
        info = -8;
    }

    if (info != 0) {
        magma_minproduct_xerbla( __func__, -(info) );
        return;  //info;
    }

    if ( nblocks > 0 ) {
        cswapdblk_kernel<<< nblocks, nb, 0, queue >>>
            ( nb, dA, ldda, inca,
                  dB, lddb, incb );
    }
}


/**
    @see magma_minproductblas_cswapdblk_q
    @ingroup magma_minproduct_caux2
    ********************************************************************/
extern "C" void 
magma_minproductblas_cswapdblk(
    magma_minproduct_int_t n, magma_minproduct_int_t nb,
    magma_minproductFloatComplex_ptr dA, magma_minproduct_int_t ldda, magma_minproduct_int_t inca,
    magma_minproductFloatComplex_ptr dB, magma_minproduct_int_t lddb, magma_minproduct_int_t incb )
{
    magma_minproductblas_cswapdblk_q( n, nb, dA, ldda, inca, dB, lddb, incb, magma_minproduct_stream );
}
