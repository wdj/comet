#include "hip/hip_runtime.h"
/*
   -- MAGMA_minproduct (version 1.6.1) --
   Univ. of Tennessee, Knoxville
   Univ. of California, Berkeley
   Univ. of Colorado, Denver
   @date January 2015

   @author Azzam Haidar
   @author Tingxing Dong

   @precisions normal z -> s d c
 */
#include "common_magma_minproduct.h"


#define zgemv_bs 32

extern __shared__ magma_minproductDoubleComplex shared_data[];


__global__ void
kernel_zgemvn_batched(
    int m, int n, magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex **dA_array, int lda,
    magma_minproductDoubleComplex **x_array, int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex  **y_array, int incy)
{

    magma_minproductDoubleComplex *A = dA_array[blockIdx.x];
    magma_minproductDoubleComplex *x = x_array[blockIdx.x];
    magma_minproductDoubleComplex *y = y_array[blockIdx.x];

    int tx = threadIdx.x;

    magma_minproductDoubleComplex res = MAGMA_minproduct_Z_ZERO;

    magma_minproductDoubleComplex *buff = (magma_minproductDoubleComplex*)shared_data;

    if(tx < n)
    {
        buff[tx] = x[tx*incx];
    }
    __syncthreads();
   
    
    if(tx < m )
    {
        for(int j=0; j < n ; j++)
        {
            res += A[tx]*buff[j];
            A += lda;
        }
  
        y[tx*incy] = alpha * res + y[tx*incy] * beta;
    }

}

/*
    Matrix Non-transpose Vector Multiplication
    y := alpha*A*x + beta*y,
*/
extern "C"
void magma_minproductblas_zgemvn_batched(
    int m, int n, 
    magma_minproductDoubleComplex alpha, magma_minproductDoubleComplex **dA_array, int lda, 
    magma_minproductDoubleComplex **x_array,  int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex **y_array,  int incy, 
    int batchCount, magma_minproduct_queue_t queue)
{

    if( m > 512 || n > 512)
    {
        fprintf( stderr, "m=%d, n=%d, zgemv_batched nontranspose assume row && column lower than %d. Plz call magma_minproductblas_zgemv instead", m, n, 512);
        return ;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(m,n), 1, 1);
   
    kernel_zgemvn_batched<<< grid, threads, n * sizeof(magma_minproductDoubleComplex), queue >>>( m, n, alpha,  dA_array, lda, x_array, incx,  
                                                                         beta, y_array, incy);
}



__global__ void
kernel_zgemvt_batched(
    int m, int n, int m1, magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex **dA_array, int lda,
    magma_minproductDoubleComplex **x_array, int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex  **y_array, int incy)
{
  

    magma_minproductDoubleComplex *A_ptr = dA_array[blockIdx.x];
    magma_minproductDoubleComplex *x_ptr = x_array[blockIdx.x];
    magma_minproductDoubleComplex *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    magma_minproductDoubleComplex res = MAGMA_minproduct_Z_ZERO;

    if(tx<m)
    {  
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ magma_minproductDoubleComplex sdata[zgemv_bs];

    for(int i=0; i<m1; i+= zgemv_bs)
    {
        res += A_ptr[i] * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += A_ptr[m1] * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(zgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Transpose Vector Multiplication
    y := alpha* A**T *x + beta*y,
*/

extern "C"
void magma_minproductblas_zgemvt_batched(
    int m, int n, 
    magma_minproductDoubleComplex alpha, magma_minproductDoubleComplex **dA_array, int lda, 
    magma_minproductDoubleComplex **x_array,  int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex **y_array,  int incy, 
    int batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(zgemv_bs, 1, 1);

    int m1 = (m / zgemv_bs) * zgemv_bs;

    kernel_zgemvt_batched <<< grid, threads,0, queue  >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);

}
   

#if defined(PRECISION_z) || defined (PRECISION_c)


__global__ void
kernel_zgemvc_batched(
    int m, int n, int m1, magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex **dA_array, int lda,
    magma_minproductDoubleComplex **x_array, int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex  **y_array, int incy)
{
  

    magma_minproductDoubleComplex *A_ptr = dA_array[blockIdx.x];
    magma_minproductDoubleComplex *x_ptr = x_array[blockIdx.x];
    magma_minproductDoubleComplex *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    magma_minproductDoubleComplex res = MAGMA_minproduct_Z_ZERO;

    if(tx<m)
    {
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ magma_minproductDoubleComplex sdata[zgemv_bs];

    for(int i=0; i<m1; i+= zgemv_bs)
    {
        res += MAGMA_minproduct_Z_CNJG (A_ptr[i]) * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += MAGMA_minproduct_Z_CNJG(A_ptr[m1]) * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(zgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Conjugate Transpose Vector Multiplication
    y := alpha* A**H *x + beta*y,
*/

extern "C"
void magma_minproductblas_zgemvc_batched(
    int m, int n, 
    magma_minproductDoubleComplex alpha, magma_minproductDoubleComplex **dA_array, int lda, 
    magma_minproductDoubleComplex **x_array,  int incx,
    magma_minproductDoubleComplex beta, magma_minproductDoubleComplex **y_array,  int incy, 
    int batchCount, magma_minproduct_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(zgemv_bs, 1, 1);

    int m1 = (m / zgemv_bs) * zgemv_bs;

    kernel_zgemvc_batched <<< grid, threads, 0, queue >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);
}
   
#endif // defined(PRECISION_z) || defined (PRECISION_c)


/**
    Purpose
    -------

    This routine computes Y = alpha opt(A) x + beta y, on the GPU, where
    A = dA_array[i],x = x_array[i] and y = y_array[i], i=[0,batchCount-1].
    This is a batched version.

    @param[in]
    trans  CHARACTER*1.
           On entry, TRANS specifies the form of op( A ) to be used in
           the matrix multiplication as follows:
           = 'N':  op( A ) = A.
           = 'T':  op( A ) = A**T.
           = 'C':  op( A ) = A**H.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix opt(A).

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix opt(A)

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA_array A = dA_array[i] 
            A: COMPLEX*16 array of dimension ( LDA, n ) on the GPU.
   
    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x_array x = x_array[i]
            x: COMPLEX*16 array of dimension.
            n if trans == Magma_minproductNoTrans.
            m if trans == Magma_minproductTrans or Magma_minproductConjTrans.

    @param[in]
    incx    INTEGER.
            incx specifies the increment for the elments of x.
            incx must not be zero.
    
    @param[in]
    beta    DOUBLE PRECISION.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y_array y = y_array[i]:       
            On exit y = alpha opt(A) x + beta y.
            y: COMPLEX*16 array of dimension.
            m if trans == Magma_minproductNoTrans.
            n if trans == Magma_minproductTrans or Magma_minproductConjTrans.

    @param[in]
    incy    INTEGER.
            incy specifies the increment for the elments of y.
            incy must not be zero.
    
    @param[in]
    batchCount INTEGER
            number of pointers contained in dA_array, x_array and y_array.

    @ingroup magma_minproduct_zblas2
    *******************************************************************   */

extern "C"
void magma_minproductblas_zgemv_batched(
    magma_minproduct_trans_t trans, magma_minproduct_int_t m, magma_minproduct_int_t n, 
    magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex_ptr dA_array[], magma_minproduct_int_t ldda, 
    magma_minproductDoubleComplex_ptr dx_array[], magma_minproduct_int_t incx,
    magma_minproductDoubleComplex beta,
    magma_minproductDoubleComplex_ptr dy_array[], magma_minproduct_int_t incy, 
    magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{       
    magma_minproduct_int_t info = 0;
    if ( trans != Magma_minproductNoTrans && trans != Magma_minproductTrans && trans != Magma_minproductConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;

    if (info != 0) {
        magma_minproduct_xerbla( __func__, -(info) );
        return;  //info;
    }

    if(m==0 || n ==0 ) return;

    if ( trans == Magma_minproductNoTrans ) {

        magma_minproductblas_zgemvn_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
            
    }
    else if ( trans == Magma_minproductTrans ) {
        magma_minproductblas_zgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
    }
    else if ( trans == Magma_minproductConjTrans ) {
#if defined(PRECISION_z) || defined (PRECISION_c)
        magma_minproductblas_zgemvc_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#else
        magma_minproductblas_zgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#endif
    }
    else {
        fprintf( stderr, "trans = %c is invalid\n", lapacke_trans_const(trans) );
    }
}

#undef zgemv_bs 
