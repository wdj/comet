#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally4.h"

#define BLOCK_SIZE 512


__global__ void 
zmgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        magma_tally4DoubleComplex alpha, 
        magma_tally4DoubleComplex * dval, 
        magma_tally4_index_t * dcolind,
        magma_tally4DoubleComplex * dx,
        magma_tally4DoubleComplex beta, 
        magma_tally4DoubleComplex * dy)
{
    extern __shared__ magma_tally4DoubleComplex dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally4DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_tally4_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_tally4DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally4DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally4DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zblas
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zmgeelltmv(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t num_vecs,
    magma_tally4_int_t nnz_per_row,
    magma_tally4DoubleComplex alpha,
    magma_tally4DoubleComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4DoubleComplex_ptr dx,
    magma_tally4DoubleComplex beta,
    magma_tally4DoubleComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ) );
    magma_tally4_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( magma_tally4DoubleComplex ); // num_vecs vectors 
    zmgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_tally4_SUCCESS;
}



