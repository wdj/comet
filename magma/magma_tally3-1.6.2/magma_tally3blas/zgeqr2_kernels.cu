#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @precisions normal z -> s d c
*/



#include "common_magma_tally3.h"
#include "batched_kernel_param.h"



static    magma_tally3DoubleComplex neg_one = MAGMA_tally3_Z_NEG_ONE;
static    magma_tally3DoubleComplex one  = MAGMA_tally3_Z_ONE;
static    magma_tally3DoubleComplex zero  = MAGMA_tally3_Z_ZERO;

__global__ void
zgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magma_tally3DoubleComplex **dV_array,    int ldv,
                  magma_tally3DoubleComplex **dR_array,    int ldr)
{

    magma_tally3DoubleComplex *dV = dV_array[blockIdx.x];
    magma_tally3DoubleComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if( tid < n && column < n) 
    {
       for(int i=column; i<n; i++)
       {
          dR[tid + i * ldr]  =  dV[tid + i * ldv];  
       }
    }
}

void zgeqrf_copy_upper_batched(                
                  magma_tally3_int_t n, magma_tally3_int_t nb,
                  magma_tally3DoubleComplex **dV_array,    magma_tally3_int_t ldv,
                  magma_tally3DoubleComplex **dR_array,    magma_tally3_int_t ldr,
          magma_tally3_int_t batchCount, magma_tally3_queue_t queue)
{
   /* 
        copy some data in dV to dR
   */

      if( nb >= n) return ;

      zgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);

}



extern "C" magma_tally3_int_t
magma_tally3_zlarfb_zgemm_batched(
                  hipblasHandle_t myhandle,
                  magma_tally3_int_t m, magma_tally3_int_t n, magma_tally3_int_t k,
                  magma_tally3DoubleComplex **dV_array,    magma_tally3_int_t ldv,
                  magma_tally3DoubleComplex **dT_array,    magma_tally3_int_t ldt,
                  magma_tally3DoubleComplex **dA_array,    magma_tally3_int_t lda,
                  magma_tally3DoubleComplex **W_array,     magma_tally3_int_t ldw,
                  magma_tally3DoubleComplex **W2_array,    magma_tally3_int_t ldw2,
                  magma_tally3_int_t batchCount, magma_tally3_queue_t queue)

{

    // W is workspace size of W is nb * n 
    // W = V^H * A. V is stored in A(i:m, i:ib)

    
    if( m <=0 || n <= 0 || k <=0 ) return 1;

#if 1  // CUBLAS is faster than MAGMA_tally3BLAS by 17GFLOP/S at size 512 batchCount = 2000
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, m,
                             &one, (const magma_tally3DoubleComplex**) dV_array, ldv,
                                    (const magma_tally3DoubleComplex**) dA_array, lda,
                             &zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, k,
                             &one, (const magma_tally3DoubleComplex**) dT_array, ldt,
                                    (const magma_tally3DoubleComplex**) W_array, ldw,
                             &zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
                             &neg_one, (const magma_tally3DoubleComplex**) dV_array, ldv,
                                    (const magma_tally3DoubleComplex**) W2_array, ldw2,
                             &one,  dA_array, lda, batchCount );

#else 

    magma_tally3blas_zgemm_batched(Magma_tally3ConjTrans, Magma_tally3NoTrans, k, n, m,
                             one, (const magma_tally3DoubleComplex**) dV_array, ldv,
                                    (const magma_tally3DoubleComplex**) dA_array, lda,
                             zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    magma_tally3blas_zgemm_batched(Magma_tally3ConjTrans, Magma_tally3NoTrans, k, n, k,
                             one, (const magma_tally3DoubleComplex**) dT_array, ldt,
                                    (const magma_tally3DoubleComplex**) W_array, ldw,
                             zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    magma_tally3blas_zgemm_batched(Magma_tally3NoTrans, Magma_tally3NoTrans, m, n, k,
                             neg_one, (const magma_tally3DoubleComplex**) dV_array, ldv,
                                    (const magma_tally3DoubleComplex**) W2_array, ldw2,
                             one,  dA_array, lda, batchCount );
          
#endif       
    return 0;

}



