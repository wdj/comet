#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
/*!
 * \file   linalg_cuda.cu
 * \author Wayne Joubert
 * \date   Tue May 15 12:03:55 EDT 2018
 * \brief  Supporting CUDA functions.
 * \note   Copyright (C) 2018 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//-----------------------------------------------------------------------------

#include "linalg_cuda.hh"

//-----------------------------------------------------------------------------

#if 0
__device__ static const GMUInt32 MGEMM2_table1[] = {
  0x00000000,
  0x00003c00,
  0x00003c00,
  0x00004000,

  0x3c000000,
  0x3c003c00,
  0x3c003c00,
  0x3c004000,

  0x3c000000,
  0x3c003c00,
  0x3c003c00,
  0x3c004000,

  0x40000000,
  0x40003c00,
  0x40003c00,
  0x40004000
  };

__device__ static const GMUInt32 MGEMM2_table0[] = {
  0x40004000,
  0x40003c00,
  0x40003c00,
  0x40000000,

  0x3c004000,
  0x3c003c00,
  0x3c003c00,
  0x3c000000,

  0x3c004000,
  0x3c003c00,
  0x3c003c00,
  0x3c000000,

  0x00004000,
  0x00003c00,
  0x00003c00,
  0x00000000
  };

struct MGEMM2 {
  static const GMUInt32* table[2];
};

const GMUInt32* MGEMM2::table[] = {MGEMM2_table0, MGEMM2_table1};

#endif

//-----------------------------------------------------------------------------

// https://docs.nvidia.com/cuda/cublas/index.html#cublas-gemmEx

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_(
  int left_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvl,
  int nvl2,
  int nfl,
  int nfl2,
  GMUInt32* vo) {

  // Two fields (seminibbles) map to two half words of 32-bit word

  const int fl2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int i01 = blockIdx.y;
  const int vl = blockIdx.z;

  if (fl2 >= nfl2) {
    return;
  }

  const GMUInt32 * const vi_col = vi + vl * vi_dim0;

  // NOTE: first field seminibble0, second field seminibble1
  const int nibble = (vi_col[fl2/8] >> (4 * (fl2%8))) & 15;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  const GMUInt32 out0 = seminibble0 ==     3*i01 ? 0x4000 :
                        seminibble0 == 3 - 3*i01 ? 0x0000 :
                                                   0x3c00;

  const GMUInt32 out1 = seminibble1 ==     3*i01 ? 0x4000 :
                        seminibble1 == 3 - 3*i01 ? 0x0000 :
                                                   0x3c00;
  const GMUInt32 out01 = out0 + ( out1 << 16);

  const int col = left_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvl2 ? 2*vl : 2*vl - nvl + 1 );

  vo[fl2 + nfl2*col] = out01;

//if (seminibble0) printf("vec %i field %i  %i\n", vl, 2*fl2+0, seminibble0);
//if (seminibble1) printf("vec %i field %i  %i\n", vl, 2*fl2+1, seminibble1);
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_buf_write_8_(
  int left_right,
  GMUInt32* vi,
  int vi_dim0,
  int nvl,
  int nvl2,
  int nfl,
  int nfl2,
  GMUInt16* vo) {

  // Two fields (seminibbles) map to two half words of 32-bit word

  const int fl2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int i01 = blockIdx.y;
  const int vl = blockIdx.z;

  if (fl2 >= nfl2) {
    return;
  }

  const GMUInt32 * const vi_col = vi + vl * vi_dim0;

  // NOTE: first field seminibble0, second field seminibble1
  const int nibble = (vi_col[fl2/8] >> (4 * (fl2%8))) & 15;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  const GMUInt16 out0 = seminibble0 ==     3*i01 ? 2 :
                        seminibble0 == 3 - 3*i01 ? 0 :
                                                   1;

  const GMUInt16 out1 = seminibble1 ==     3*i01 ? 2 :
                        seminibble1 == 3 - 3*i01 ? 0 :
                                                   1;
  const GMUInt16 out01 = out0 + ( out1 << 8);

  const int col = left_right ? i01 + 2*vl :
                  i01 + 2*( vl < nvl2 ? 2*vl : 2*vl - nvl + 1 );

  vo[fl2 + nfl2*col] = out01;

//printf("%i %i\n", (int)out0, (int)out1);

//if (seminibble0) printf("vec %i field %i  %i\n", vl, 2*fl2+0, seminibble0);
//if (seminibble1) printf("vec %i field %i  %i\n", vl, 2*fl2+1, seminibble1);
}

//-----------------------------------------------------------------------------

void gm_tc_buf_write(
  int left_right,
  int num_vector_local,
  int num_packedval_field_local,
  void* bufd,
  GMEnv* env) {
  GMInsist(left_right ==0 || left_right == 1);
  GMInsist(env && bufd);
  GMInsist(num_vector_local >= 0);
  GMInsist(num_packedval_field_local >= 0);
  GMInsist(GMEnv_metric_type(env) == GM_METRIC_TYPE_CCC);
  GMInsist(GMEnv_compute_method(env) == GM_COMPUTE_METHOD_GPU);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  GMInsist(deviceProp.major >= 7);

  GMInsistInterface(env, GMEnv_num_way(env) == GM_NUM_WAY_2 &&
                    "Not yet implemented.");
  GMInsistInterface(env, !env->sparse && "Not yet implemented.");

  const int nvl = num_vector_local;
  const int nvl2 = nvl / 2;
  const int npvfl = num_packedval_field_local;
  const int nfl = npvfl * 64;
  const int nfl2 = nfl / 2;

  GMInsistInterface(env, num_vector_local % 2 == 0 &&
                    "tc method requires num_vector_local multiple of 2.");

  const int threadblocksize = 256;
  const int fl2_threadblocks = (nfl2+threadblocksize-1) / threadblocksize;

  if (env->tc == 2) {

    gm_tc_buf_write_8_<<<
      dim3(fl2_threadblocks, 2, nvl),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
      left_right,
      (GMUInt32*)bufd,
      npvfl * 4,
      nvl,
      nvl2,
      nfl,
      nfl2,
      left_right ? (GMUInt16*)env->tc_buf_right : (GMUInt16*)env->tc_buf_left);

  } else {

    gm_tc_buf_write_<<<
      dim3(fl2_threadblocks, 2, nvl),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
      left_right,
      (GMUInt32*)bufd,
      npvfl * 4,
      nvl,
      nvl2,
      nfl,
      nfl2,
      left_right ? (GMUInt32*)env->tc_buf_right : (GMUInt32*)env->tc_buf_left);

  }

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------

void gm_tc_solve(
  int num_vector_local,
  int num_vector_local_copy,
  int num_packedval_field_local,
  void* dA,
  int ldda,
  void* dB,
  int lddb,
  void* dC,
  int lddc,
  GMEnv* env) {
  GMInsist(env && dA && dB && dC);
  GMInsist(num_vector_local >= 0);
  GMInsist(num_vector_local_copy >= 0);
  GMInsist(num_packedval_field_local >= 0);
  GMInsist(ldda >= 0 && lddb >= 0 && lddc >= 0);

  const int nvl = num_vector_local;
  const int npvfl = num_packedval_field_local;
  const int nfl = npvfl * 64;

  const int m = nvl * 2;
  const int n = nvl * 2;
  const int k = nfl;

  const float alpha = 1;
  const float beta = 0;

#if 0
  hipblasStatus_t status = cublasSgemmEx(
    env->cublas_handle,
    HIPBLAS_OP_T, HIPBLAS_OP_N,
    m, n, k,
    &alpha,
    env->tc_buf_left, HIP_R_16F, k,
    env->tc_buf_right, HIP_R_16F, k,
    &beta,
    dC, HIP_R_32F, m);
#endif

  hipblasStatus_t status = hipblasGemmEx(
    env->cublas_handle,
    HIPBLAS_OP_T, HIPBLAS_OP_N,
    m, n, k,
    &alpha,
    env->tc_buf_left, env->tc == 2 ? HIP_R_8I : HIP_R_16F, k,
    env->tc_buf_right, env->tc == 2 ? HIP_R_8I : HIP_R_16F, k,
    &beta,
    dC, HIP_R_32F, m,
    HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);

  GMInsist(status == HIPBLAS_STATUS_SUCCESS);

  env->ops_local += 2 * m * (double)n * (double)k;
}

//-----------------------------------------------------------------------------

__global__ void gm_tc_fix_metrics_(
  int nvl,
  int nvl2,
  float* bufd) {

  const int thread_r = threadIdx.x + blockIdx.x * blockDim.x;
  const int thread_c = blockIdx.y;

  if (thread_r >= nvl2 || thread_c >= nvl) {
    return;
  }

  const int fc0 = thread_c * (4*nvl);
  const int fc1 = thread_c * (4*nvl) + 2*nvl;

  const float f00 = bufd[fc0+0+4*thread_r];
  const float f01 = bufd[fc0+1+4*thread_r];
  const float f02 = bufd[fc0+2+4*thread_r];
  const float f03 = bufd[fc0+3+4*thread_r];

  const float f10 = bufd[fc1+0+4*thread_r];
  const float f11 = bufd[fc1+1+4*thread_r];
  const float f12 = bufd[fc1+2+4*thread_r];
  const float f13 = bufd[fc1+3+4*thread_r];

  const float f00p = f00;
  const float f01p = f01;

  const float f02p = f10;
  const float f03p = f11;

  const float f10p = f02;
  const float f11p = f03;

  const float f12p = f12;
  const float f13p = f13;

  const double shifter = (((GMUInt32)1)<<GM_TALLY1_MAX_VALUE_BITS);

  const double d00 = f00p + f02p * shifter;
  const double d01 = f01p + f03p * shifter;

  const double d10 = f10p + f12p * shifter;
  const double d11 = f11p + f13p * shifter;

  const int dc0 = thread_c * (2*nvl);
  const int dc1 = thread_c * (2*nvl) + nvl;

  ((double*)bufd)[dc0+0+2*thread_r] = d00;
  ((double*)bufd)[dc0+1+2*thread_r] = d01;

  ((double*)bufd)[dc1+0+2*thread_r] = d10;
  ((double*)bufd)[dc1+1+2*thread_r] = d11;

//printf("%i %i %f %f %f %f\n", 2*thread_r+0, thread_c, f00p, f01p, f02p, f03p);
//printf("%i %i %f %f %f %f\n", 2*thread_r+1, thread_c, f10p, f11p, f12p, f13p);
}

//-----------------------------------------------------------------------------

void gm_tc_fix_metrics(
  int num_vector_local,
  void* bufd,
  GMEnv* env) {
  GMInsist(env && bufd);
  GMInsist(num_vector_local >= 0);

  const int nvl = num_vector_local;
  const int nvl2 = nvl / 2;

  const int threadblocksize = 256;
  const int vl2_threadblocks = (nvl2+threadblocksize-1) / threadblocksize;

  gm_tc_fix_metrics_<<<
    dim3(vl2_threadblocks, nvl, 1),
    dim3(threadblocksize, 1, 1),
    0,
    env->stream_compute_>>>(
    nvl,
    nvl2,
    (float*)bufd
  );

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
