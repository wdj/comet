#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/
#include "common_magma_minproduct.h"

#define BLOCK_SIZE 512


__global__ void 
zmgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    magma_minproductDoubleComplex alpha, 
    magma_minproductDoubleComplex * dval, 
    magma_minproduct_index_t * drowptr, 
    magma_minproduct_index_t * dcolind,
    magma_minproductDoubleComplex * dx,
    magma_minproductDoubleComplex beta, 
    magma_minproductDoubleComplex * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magma_minproductDoubleComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_minproduct_Z_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            magma_minproductDoubleComplex val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       magma_minproductDoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductDoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_minproductIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_minproductDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductDoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_minproductDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_zblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_zmgecsrmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t num_vecs, 
    magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex_ptr dval,
    magma_minproductIndex_ptr drowptr,
    magma_minproductIndex_ptr dcolind,
    magma_minproductDoubleComplex_ptr dx,
    magma_minproductDoubleComplex beta,
    magma_minproductDoubleComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ), 1, 1);
    magma_minproduct_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magma_minproductDoubleComplex ); // num_vecs vectors 
    zmgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_minproduct_SUCCESS;
}



