#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @precisions normal z -> s d c
*/



#include "common_magma_minproduct.h"
#include "batched_kernel_param.h"



static    magma_minproductDoubleComplex neg_one = MAGMA_minproduct_Z_NEG_ONE;
static    magma_minproductDoubleComplex one  = MAGMA_minproduct_Z_ONE;
static    magma_minproductDoubleComplex zero  = MAGMA_minproduct_Z_ZERO;

__global__ void
zgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magma_minproductDoubleComplex **dV_array,    int ldv,
                  magma_minproductDoubleComplex **dR_array,    int ldr)
{

    magma_minproductDoubleComplex *dV = dV_array[blockIdx.x];
    magma_minproductDoubleComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if( tid < n && column < n) 
    {
       for(int i=column; i<n; i++)
       {
          dR[tid + i * ldr]  =  dV[tid + i * ldv];  
       }
    }
}

void zgeqrf_copy_upper_batched(                
                  magma_minproduct_int_t n, magma_minproduct_int_t nb,
                  magma_minproductDoubleComplex **dV_array,    magma_minproduct_int_t ldv,
                  magma_minproductDoubleComplex **dR_array,    magma_minproduct_int_t ldr,
          magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)
{
   /* 
        copy some data in dV to dR
   */

      if( nb >= n) return ;

      zgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);

}



extern "C" magma_minproduct_int_t
magma_minproduct_zlarfb_zgemm_batched(
                  hipblasHandle_t myhandle,
                  magma_minproduct_int_t m, magma_minproduct_int_t n, magma_minproduct_int_t k,
                  magma_minproductDoubleComplex **dV_array,    magma_minproduct_int_t ldv,
                  magma_minproductDoubleComplex **dT_array,    magma_minproduct_int_t ldt,
                  magma_minproductDoubleComplex **dA_array,    magma_minproduct_int_t lda,
                  magma_minproductDoubleComplex **W_array,     magma_minproduct_int_t ldw,
                  magma_minproductDoubleComplex **W2_array,    magma_minproduct_int_t ldw2,
                  magma_minproduct_int_t batchCount, magma_minproduct_queue_t queue)

{

    // W is workspace size of W is nb * n 
    // W = V^H * A. V is stored in A(i:m, i:ib)

    
    if( m <=0 || n <= 0 || k <=0 ) return 1;

#if 1  // CUBLAS is faster than MAGMA_minproductBLAS by 17GFLOP/S at size 512 batchCount = 2000
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, m,
                             &one, (const magma_minproductDoubleComplex**) dV_array, ldv,
                                    (const magma_minproductDoubleComplex**) dA_array, lda,
                             &zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, k,
                             &one, (const magma_minproductDoubleComplex**) dT_array, ldt,
                                    (const magma_minproductDoubleComplex**) W_array, ldw,
                             &zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    hipblasZgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
                             &neg_one, (const magma_minproductDoubleComplex**) dV_array, ldv,
                                    (const magma_minproductDoubleComplex**) W2_array, ldw2,
                             &one,  dA_array, lda, batchCount );

#else 

    magma_minproductblas_zgemm_batched(Magma_minproductConjTrans, Magma_minproductNoTrans, k, n, m,
                             one, (const magma_minproductDoubleComplex**) dV_array, ldv,
                                    (const magma_minproductDoubleComplex**) dA_array, lda,
                             zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    magma_minproductblas_zgemm_batched(Magma_minproductConjTrans, Magma_minproductNoTrans, k, n, k,
                             one, (const magma_minproductDoubleComplex**) dT_array, ldt,
                                    (const magma_minproductDoubleComplex**) W_array, ldw,
                             zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    magma_minproductblas_zgemm_batched(Magma_minproductNoTrans, Magma_minproductNoTrans, m, n, k,
                             neg_one, (const magma_minproductDoubleComplex**) dV_array, ldv,
                                    (const magma_minproductDoubleComplex**) W2_array, ldw2,
                             one,  dA_array, lda, batchCount );
          
#endif       
    return 0;

}



