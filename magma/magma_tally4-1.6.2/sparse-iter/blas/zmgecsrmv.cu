#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/
#include "common_magma_tally4.h"

#define BLOCK_SIZE 512


__global__ void 
zmgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * drowptr, 
    magma_tally4_index_t * dcolind,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    magma_tally4DoubleComplex * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magma_tally4DoubleComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            magma_tally4DoubleComplex val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       magma_tally4DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4DoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_tally4Index_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_tally4DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally4DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zblas
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zmgecsrmv(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t num_vecs, 
    magma_tally4DoubleComplex alpha,
    magma_tally4DoubleComplex_ptr dval,
    magma_tally4Index_ptr drowptr,
    magma_tally4Index_ptr dcolind,
    magma_tally4DoubleComplex_ptr dx,
    magma_tally4DoubleComplex beta,
    magma_tally4DoubleComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ), 1, 1);
    magma_tally4_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magma_tally4DoubleComplex ); // num_vecs vectors 
    zmgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_tally4_SUCCESS;
}



