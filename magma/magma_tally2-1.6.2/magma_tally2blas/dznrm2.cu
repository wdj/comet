#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c

*/
#include "common_magma_tally2.h"
#include "commonblas_z.h"
#include "magma_tally2_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512
#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

#define PRECISION_z


//==============================================================================

__global__ void
magma_tally2blas_dznrm2_kernel( int m, magma_tally2DoubleComplex *dA, int ldda, double *dxnorm )
{
    const int tx = threadIdx.x;
    magma_tally2DoubleComplex *dx = dA + blockIdx.x * ldda;

    __shared__ double sum[ BLOCK_SIZE ];
    double re, lsum;

    // get norm of dx
    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_tally2_Z_REAL( dx[j] );
        double im = MAGMA_tally2_Z_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_tally2_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}


//==============================================================================
__global__ void
magma_tally2blas_dznrm2_check_kernel( int m, magma_tally2DoubleComplex *dA, int ldda, double *dxnorm, 
                               double *lsticc )
{
    const int tx = threadIdx.x;
    magma_tally2DoubleComplex *dx = dA + blockIdx.x * ldda;

    __shared__ double sum[ BLOCK_SIZE ];
    double re, lsum;

    // get norm of dx only if lsticc[blockIdx+1] != 0
    if ( lsticc[blockIdx.x + 1] == 0 )
        return;

    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_tally2_Z_REAL( dx[j] );
        double im = MAGMA_tally2_Z_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_tally2_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}

extern "C" void
magma_tally2blas_dznrm2_check(
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2DoubleComplex_ptr dA, magma_tally2_int_t ldda, 
    magma_tally2Double_ptr dxnorm,
    magma_tally2Double_ptr dlsticc) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magma_tally2blas_dznrm2_check_kernel<<< blocks, threads >>>( m, dA, ldda, dxnorm, dlsticc );
}


//==============================================================================
__global__ void
magma_tally2blas_dznrm2_smkernel( int m, int n, magma_tally2DoubleComplex *dA, int ldda,
                           double *dxnorm )
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    __shared__ double sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
    double re, lsum;

    for( int k = ty; k < n; k += BLOCK_SIZEy ) {
        magma_tally2DoubleComplex *dx = dA + k * ldda;

        // get norm of dx
        lsum = 0;
        for( int j = tx; j < m; j += BLOCK_SIZEx ) {
            #if (defined(PRECISION_s) || defined(PRECISION_d))
            re = dx[j];
            lsum += re*re;
            #else
            re = MAGMA_tally2_Z_REAL( dx[j] );
            double im = MAGMA_tally2_Z_IMAG( dx[j] );
            lsum += re*re + im*im;
            #endif
        }
        sum[tx][ty] = lsum;
        magma_tally2_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( tx, ty, sum );

        if (tx == 0)
            dxnorm[k] = sqrt(sum[0][ty]);
        __syncthreads();
    }
}


//==============================================================================
/*
    Compute the dznrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array.
    This routine uses only one SM (block).
*/
extern "C" void
magma_tally2blas_dznrm2_sm(
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2DoubleComplex_ptr dA, magma_tally2_int_t ldda,
    double *dxnorm)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_tally2blas_dznrm2_smkernel<<< blocks, threads, 0, magma_tally2_stream >>>( m, n, dA, ldda, dxnorm );
}

//==============================================================================
extern "C"
__global__ void
magma_tally2_dznrm2_adjust_kernel(double *xnorm, magma_tally2DoubleComplex *c)
{
    const int tx = threadIdx.x;

    __shared__ double sum[ BLOCK_SIZE ];
    double temp;

    temp = MAGMA_tally2_Z_ABS( c[tx] ) / xnorm[0];
    sum[tx] = -temp * temp;
    magma_tally2_sum_reduce_n( blockDim.x, tx, sum );

    __syncthreads();
    if (tx == 0)
        xnorm[0] = xnorm[0] * sqrt(1+sum[0]);
}


/*
    Adjust the norm of c to give the norm of c[k+1:], assuming that
    c was changed with orthogonal transformations.
*/
extern "C" void
magma_tally2blas_dznrm2_adjust(magma_tally2_int_t k, magma_tally2Double_ptr dxnorm, magma_tally2DoubleComplex_ptr dc)
{
    magma_tally2_dznrm2_adjust_kernel<<< 1, k, 0, magma_tally2_stream >>> (dxnorm, dc);
}

//==============================================================================

#define BS 256

__global__ void
magma_tally2_dznrm2_row_check_adjust_kernel(
    int n, double tol, double *xnorm, double *xnorm2, 
    magma_tally2DoubleComplex *C, int ldc, double *lsticc)
{
    const int tx = threadIdx.x + blockIdx.x*BS;
    lsticc[tx+1] = 0;

    if (tx < n) {
        double temp = MAGMA_tally2_Z_ABS( C[tx*ldc] ) / xnorm[tx];
        temp = max( 0.0, ((1.0 + temp) * (1.0 - temp)) );
        
        
        double temp2 = xnorm[tx] / xnorm2[tx];
        temp2 = temp * (temp2 * temp2);
        
        if (temp2 <= tol) {
            lsticc[tx+1] = 1;
        } else {
            xnorm[tx] *= sqrt(temp);
        }
    }
    if (tx == 0)
        lsticc[0] = 0;
    magma_tally2_sum_reduce_n( blockDim.x, tx, lsticc );
}

/*
    Adjust the norm of C[,1:k] to give the norm of C[k+1:,1:k], assuming that
    C was changed with orthogonal transformations.
    It also do checks for QP3
*/
extern "C" void
magma_tally2blas_dznrm2_row_check_adjust(
    magma_tally2_int_t k, double tol,
    magma_tally2Double_ptr dxnorm,
    magma_tally2Double_ptr dxnorm2, 
    magma_tally2DoubleComplex_ptr dC, magma_tally2_int_t lddc,
    magma_tally2Double_ptr dlsticc)
{
    int nblocks = (k+BS-1)/BS;
    magma_tally2_dznrm2_row_check_adjust_kernel<<< nblocks, BS >>> (k, tol, dxnorm, dxnorm2, dC, lddc, dlsticc);
}

//==============================================================================

/*
    Compute the dznrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array. 
    The computation can be done using n blocks (default) or on one SM (commented).
*/
extern "C" void
magma_tally2blas_dznrm2_cols(
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2DoubleComplex_ptr dA, magma_tally2_int_t ldda, 
    magma_tally2Double_ptr dxnorm) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magma_tally2blas_dznrm2_kernel<<< blocks, threads, 0, magma_tally2_stream >>>( m, dA, ldda, dxnorm );

    // The following would do the computation on one SM
    // magma_tally2blas_dznrm2_sm(m, n, dA, ldda, dxnorm);
}

//==============================================================================
