#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally4.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
__global__ void 
zgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    magma_tally4DoubleComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally4DoubleComplex dot = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally4DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
zgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex lambda, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    int offset,
    int blocksize,
    magma_tally4_index_t * addrows,
    magma_tally4DoubleComplex * dy)
{

    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally4DoubleComplex dot = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally4DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally4_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally4DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally4DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally4DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_z
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zgeelltmv(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t nnz_per_row,
    magma_tally4DoubleComplex alpha,
    magma_tally4DoubleComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4DoubleComplex_ptr dx,
    magma_tally4DoubleComplex beta,
    magma_tally4DoubleComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ) );
    magma_tally4_int_t threads = BLOCK_SIZE;
    zgeelltmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally4_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_tally4_int_t
                number of rows in A

    @param[in]
    n           magma_tally4_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally4_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally4DoubleComplex
                scalar multiplier

    @param[in]
    lambda      magma_tally4DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally4DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally4Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally4DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4DoubleComplex
                scalar multiplier
                
    @param[in]
    offset      magma_tally4_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally4_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally4Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally4DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zblas
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zgeelltmv_shift(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t nnz_per_row,
    magma_tally4DoubleComplex alpha,
    magma_tally4DoubleComplex lambda,
    magma_tally4DoubleComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4DoubleComplex_ptr dx,
    magma_tally4DoubleComplex beta,
    int offset,
    int blocksize,
    magma_tally4Index_ptr addrows,
    magma_tally4DoubleComplex_ptr dy,
    magma_tally4_queue_t queue )
{
    dim3 grid( magma_tally4_ceildiv( m, BLOCK_SIZE ) );
    magma_tally4_int_t threads = BLOCK_SIZE;
    magma_tally4DoubleComplex tmp_shift;
    //magma_tally4_zsetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    zgeelltmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


   return MAGMA_tally4_SUCCESS;
}



