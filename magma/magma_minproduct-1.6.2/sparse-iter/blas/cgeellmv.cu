#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeellmv.cu normal z -> c, Sun May  3 11:22:58 2015

*/

#include "common_magma_minproduct.h"

#define BLOCK_SIZE 512


// ELLPACK SpMV kernel
//Michael Garland
__global__ void 
cgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_minproductFloatComplex alpha, 
    magma_minproductFloatComplex * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproductFloatComplex * dx,
    magma_minproductFloatComplex beta, 
    magma_minproductFloatComplex * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_minproductFloatComplex dot = MAGMA_minproduct_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            magma_minproductFloatComplex val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELLPACK SpMV kernel
//Michael Garland
__global__ void 
cgeellmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_minproductFloatComplex alpha, 
    magma_minproductFloatComplex lambda, 
    magma_minproductFloatComplex * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproductFloatComplex * dx,
    magma_minproductFloatComplex beta, 
    int offset,
    int blocksize,
    magma_minproduct_index_t * addrows,
    magma_minproductFloatComplex * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_minproductFloatComplex dot = MAGMA_minproduct_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            magma_minproductFloatComplex val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}





/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLPACK.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_minproduct_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductFloatComplex_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_minproductFloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductFloatComplex
                scalar multiplier

    @param[out]
    dy          magma_minproductFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_cblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_cgeellmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t nnz_per_row,
    magma_minproductFloatComplex alpha,
    magma_minproductFloatComplex_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductFloatComplex_ptr dx,
    magma_minproductFloatComplex beta,
    magma_minproductFloatComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
   cgeellmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_minproduct_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELLPACK.
    It is the shifted version of the ELLPACK SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
    @param[in]
    nnz_per_row magma_minproduct_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_minproductFloatComplex
                scalar multiplier
                
    @param[in]
    lambda      magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductFloatComplex_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_minproductFloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductFloatComplex
                scalar multiplier
                
    @param[in]
    offset      magma_minproduct_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_minproduct_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_minproductIndex_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_minproductFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_cblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_cgeellmv_shift(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t nnz_per_row,
    magma_minproductFloatComplex alpha,
    magma_minproductFloatComplex lambda,
    magma_minproductFloatComplex_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductFloatComplex_ptr dx,
    magma_minproductFloatComplex beta,
    int offset,
    int blocksize,
    magma_minproductIndex_ptr addrows,
    magma_minproductFloatComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
   cgeellmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, lambda, dval, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy );


   return MAGMA_minproduct_SUCCESS;
}



