#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgecsrmv.cu normal z -> c, Sun May  3 11:22:58 2015

*/
#include "common_magma_minproduct.h"

#define BLOCK_SIZE 256


// CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    magma_minproductFloatComplex alpha, 
    magma_minproductFloatComplex * dval, 
    magma_minproduct_index_t * drowptr, 
    magma_minproduct_index_t * dcolind,
    magma_minproductFloatComplex * dx,
    magma_minproductFloatComplex beta, 
    magma_minproductFloatComplex * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magma_minproductFloatComplex dot = MAGMA_minproduct_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel_shift( 
    int num_rows, 
    int num_cols, 
    magma_minproductFloatComplex alpha, 
    magma_minproductFloatComplex lambda, 
    magma_minproductFloatComplex * dval, 
    magma_minproduct_index_t * drowptr, 
    magma_minproduct_index_t * dcolind,
    magma_minproductFloatComplex * dx,
    magma_minproductFloatComplex beta, 
    int offset,
    int blocksize,
    magma_minproduct_index_t * addrows,
    magma_minproductFloatComplex * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magma_minproductFloatComplex dot = MAGMA_minproduct_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                        * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                        * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 

    @param[in]
    alpha       magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductFloatComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_minproductIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_minproductFloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductFloatComplex
                scalar multiplier

    @param[out]
    dy          magma_minproductFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_cblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_cgecsrmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproductFloatComplex alpha,
    magma_minproductFloatComplex_ptr dval,
    magma_minproductIndex_ptr drowptr,
    magma_minproductIndex_ptr dcolind,
    magma_minproductFloatComplex_ptr dx,
    magma_minproductFloatComplex beta,
    magma_minproductFloatComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
    cgecsrmv_kernel<<< grid, threads, 0, queue >>>
                    (m, n, alpha, dval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_minproduct_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 

    @param[in]
    alpha       magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    lambda      magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductFloatComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_minproductIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_minproductFloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductFloatComplex
                scalar multiplier

    @param[in]
    offset      magma_minproduct_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_minproduct_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_minproductIndex_ptr
                in case the matrixpowerskernel is used
                
    @param[out]
    dy          magma_minproductFloatComplex_ptr
                output vector y  
    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_cblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_cgecsrmv_shift(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproductFloatComplex alpha,
    magma_minproductFloatComplex lambda,
    magma_minproductFloatComplex_ptr dval,
    magma_minproductIndex_ptr drowptr,
    magma_minproductIndex_ptr dcolind,
    magma_minproductFloatComplex_ptr dx,
    magma_minproductFloatComplex beta,
    int offset,
    int blocksize,
    magma_minproduct_index_t * addrows,
    magma_minproductFloatComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
    cgecsrmv_kernel_shift<<< grid, threads, 0, queue >>>
                         (m, n, alpha, lambda, dval, drowptr, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy);

    return MAGMA_minproduct_SUCCESS;
}



