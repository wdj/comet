#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from dznrm2.cu normal z -> c, Fri Jan 30 19:00:09 2015

*/
#include "common_magma_tally4.h"
#include "commonblas_c.h"
#include "magma_tally4_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512
#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

#define PRECISION_c


//==============================================================================

__global__ void
magma_tally4blas_scnrm2_kernel( int m, magma_tally4FloatComplex *dA, int ldda, float *dxnorm )
{
    const int tx = threadIdx.x;
    magma_tally4FloatComplex *dx = dA + blockIdx.x * ldda;

    __shared__ float sum[ BLOCK_SIZE ];
    float re, lsum;

    // get norm of dx
    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_tally4_C_REAL( dx[j] );
        float im = MAGMA_tally4_C_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_tally4_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}


//==============================================================================
__global__ void
magma_tally4blas_scnrm2_check_kernel( int m, magma_tally4FloatComplex *dA, int ldda, float *dxnorm, 
                               float *lsticc )
{
    const int tx = threadIdx.x;
    magma_tally4FloatComplex *dx = dA + blockIdx.x * ldda;

    __shared__ float sum[ BLOCK_SIZE ];
    float re, lsum;

    // get norm of dx only if lsticc[blockIdx+1] != 0
    if ( lsticc[blockIdx.x + 1] == 0 )
        return;

    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_tally4_C_REAL( dx[j] );
        float im = MAGMA_tally4_C_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_tally4_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}

extern "C" void
magma_tally4blas_scnrm2_check(
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4FloatComplex_ptr dA, magma_tally4_int_t ldda, 
    magma_tally4Float_ptr dxnorm,
    magma_tally4Float_ptr dlsticc) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magma_tally4blas_scnrm2_check_kernel<<< blocks, threads >>>( m, dA, ldda, dxnorm, dlsticc );
}


//==============================================================================
__global__ void
magma_tally4blas_scnrm2_smkernel( int m, int n, magma_tally4FloatComplex *dA, int ldda,
                           float *dxnorm )
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    __shared__ float sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
    float re, lsum;

    for( int k = ty; k < n; k += BLOCK_SIZEy ) {
        magma_tally4FloatComplex *dx = dA + k * ldda;

        // get norm of dx
        lsum = 0;
        for( int j = tx; j < m; j += BLOCK_SIZEx ) {
            #if (defined(PRECISION_s) || defined(PRECISION_d))
            re = dx[j];
            lsum += re*re;
            #else
            re = MAGMA_tally4_C_REAL( dx[j] );
            float im = MAGMA_tally4_C_IMAG( dx[j] );
            lsum += re*re + im*im;
            #endif
        }
        sum[tx][ty] = lsum;
        magma_tally4_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( tx, ty, sum );

        if (tx == 0)
            dxnorm[k] = sqrt(sum[0][ty]);
        __syncthreads();
    }
}


//==============================================================================
/*
    Compute the scnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array.
    This routine uses only one SM (block).
*/
extern "C" void
magma_tally4blas_scnrm2_sm(
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4FloatComplex_ptr dA, magma_tally4_int_t ldda,
    float *dxnorm)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_tally4blas_scnrm2_smkernel<<< blocks, threads, 0, magma_tally4_stream >>>( m, n, dA, ldda, dxnorm );
}

//==============================================================================
extern "C"
__global__ void
magma_tally4_scnrm2_adjust_kernel(float *xnorm, magma_tally4FloatComplex *c)
{
    const int tx = threadIdx.x;

    __shared__ float sum[ BLOCK_SIZE ];
    float temp;

    temp = MAGMA_tally4_C_ABS( c[tx] ) / xnorm[0];
    sum[tx] = -temp * temp;
    magma_tally4_sum_reduce_n( blockDim.x, tx, sum );

    __syncthreads();
    if (tx == 0)
        xnorm[0] = xnorm[0] * sqrt(1+sum[0]);
}


/*
    Adjust the norm of c to give the norm of c[k+1:], assuming that
    c was changed with orthogonal transformations.
*/
extern "C" void
magma_tally4blas_scnrm2_adjust(magma_tally4_int_t k, magma_tally4Float_ptr dxnorm, magma_tally4FloatComplex_ptr dc)
{
    magma_tally4_scnrm2_adjust_kernel<<< 1, k, 0, magma_tally4_stream >>> (dxnorm, dc);
}

//==============================================================================

#define BS 256

__global__ void
magma_tally4_scnrm2_row_check_adjust_kernel(
    int n, float tol, float *xnorm, float *xnorm2, 
    magma_tally4FloatComplex *C, int ldc, float *lsticc)
{
    const int tx = threadIdx.x + blockIdx.x*BS;
    lsticc[tx+1] = 0;

    if (tx < n) {
        float temp = MAGMA_tally4_C_ABS( C[tx*ldc] ) / xnorm[tx];
        temp = max( 0.0, ((1.0 + temp) * (1.0 - temp)) );
        
        
        float temp2 = xnorm[tx] / xnorm2[tx];
        temp2 = temp * (temp2 * temp2);
        
        if (temp2 <= tol) {
            lsticc[tx+1] = 1;
        } else {
            xnorm[tx] *= sqrt(temp);
        }
    }
    if (tx == 0)
        lsticc[0] = 0;
    magma_tally4_sum_reduce_n( blockDim.x, tx, lsticc );
}

/*
    Adjust the norm of C[,1:k] to give the norm of C[k+1:,1:k], assuming that
    C was changed with orthogonal transformations.
    It also do checks for QP3
*/
extern "C" void
magma_tally4blas_scnrm2_row_check_adjust(
    magma_tally4_int_t k, float tol,
    magma_tally4Float_ptr dxnorm,
    magma_tally4Float_ptr dxnorm2, 
    magma_tally4FloatComplex_ptr dC, magma_tally4_int_t lddc,
    magma_tally4Float_ptr dlsticc)
{
    int nblocks = (k+BS-1)/BS;
    magma_tally4_scnrm2_row_check_adjust_kernel<<< nblocks, BS >>> (k, tol, dxnorm, dxnorm2, dC, lddc, dlsticc);
}

//==============================================================================

/*
    Compute the scnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array. 
    The computation can be done using n blocks (default) or on one SM (commented).
*/
extern "C" void
magma_tally4blas_scnrm2_cols(
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4FloatComplex_ptr dA, magma_tally4_int_t ldda, 
    magma_tally4Float_ptr dxnorm) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magma_tally4blas_scnrm2_kernel<<< blocks, threads, 0, magma_tally4_stream >>>( m, dA, ldda, dxnorm );

    // The following would do the computation on one SM
    // magma_tally4blas_scnrm2_sm(m, n, dA, ldda, dxnorm);
}

//==============================================================================
