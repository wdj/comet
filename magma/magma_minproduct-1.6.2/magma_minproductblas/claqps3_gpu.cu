#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlaqps3_gpu.cu normal z -> c, Fri Jan 30 19:00:09 2015

*/

#include "common_magma_minproduct.h"
#include "commonblas_c.h"
#include "magma_minproduct_templates.h"

#define PRECISION_c

#define BLOCK_SIZE 512


/* --------------------------------------------------------------------------- */

#define BLOCK_SIZE1 192

__global__ void
magma_minproduct_cswap_gemv_kernel(int m, int rk, int n, const magma_minproductFloatComplex * __restrict__ V, int ldv,
                     const magma_minproductFloatComplex * __restrict__ x, int ldx, magma_minproductFloatComplex *c, magma_minproductFloatComplex *b)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE1 * blockIdx.x;
    magma_minproductFloatComplex lsum, tmp;

    V += j;

    lsum = MAGMA_minproduct_C_ZERO;
    if (j < m){
       tmp  = b[j];
       b[j] = c[j];
       if (j>=rk) 
          for(int k=0; k<n; k++)
              lsum += MAGMA_minproduct_C_MUL( V[k*ldv], MAGMA_minproduct_C_CNJG(x[k*ldx]));

       c[j] = tmp - lsum;
    }
}

__global__ void
magma_minproduct_cgemv_kernel(int m, int n, const magma_minproductFloatComplex * __restrict__ V, int ldv,
                     const magma_minproductFloatComplex * __restrict__ x, magma_minproductFloatComplex *b, magma_minproductFloatComplex *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE1 * blockIdx.x;
    magma_minproductFloatComplex lsum;

    V += j;

    lsum = MAGMA_minproduct_C_ZERO;
    if (j < m){
        for(int k=0; k<n; k++)
            lsum += MAGMA_minproduct_C_MUL( V[k*ldv], x[k]);

       c[j] = b[j] - lsum;
    }
}


__global__
void magma_minproduct_cscale_kernel(int n, magma_minproductFloatComplex* dx0,
                         magma_minproductFloatComplex *dtau, float *dxnorm, magma_minproductFloatComplex* dAkk)
{
   const int i = threadIdx.x;
   magma_minproductFloatComplex tmp;
   __shared__ magma_minproductFloatComplex scale;

   /* === Compute the norm of dx0 === */
   magma_minproductFloatComplex *dx = dx0;
   __shared__ float sum[ BLOCK_SIZE ];
   float re, lsum;

   lsum = 0;
   for( int k = i; k < n; k += BLOCK_SIZE ) {

        #if (defined(PRECISION_s) || defined(PRECISION_d))
             re = dx[k];
             lsum += re*re;
        #else
             re = MAGMA_minproduct_C_REAL( dx[k] );
             float im = MAGMA_minproduct_C_IMAG( dx[k] );
             lsum += re*re + im*im;
        #endif
   }
   sum[i] = lsum;
   magma_minproduct_sum_reduce< BLOCK_SIZE >( i, sum );

   /* === Compute the scaling factor === */
   if (i==0){
            float beta = sqrt(sum[0]);
            if ( beta == 0 ) {
              *dtau = MAGMA_minproduct_C_ZERO;
            }
            else {
               tmp = dx0[0];
#if (defined(PRECISION_s) || defined(PRECISION_d))
               beta  = -copysign( beta, tmp );

               // todo: deal with badly scaled vectors (see lapack's larfg)
               *dtau    = (beta - tmp) / beta;
               *dAkk    = beta;

               scale = 1. / (tmp - beta);
#else
               float alphar =  MAGMA_minproduct_C_REAL(tmp), alphai = MAGMA_minproduct_C_IMAG(tmp);
               beta  = -copysign( beta, alphar );

               // todo: deal with badly scaled vectors (see lapack's larfg)
               *dtau = MAGMA_minproduct_C_MAKE((beta - alphar)/beta, -alphai/beta);
               *dAkk = MAGMA_minproduct_C_MAKE(beta, 0.);

               tmp = MAGMA_minproduct_C_MAKE( alphar - beta, alphai);
               scale = MAGMA_minproduct_C_DIV( MAGMA_minproduct_C_ONE, tmp);
#endif
            }
   }

   __syncthreads();

   /* === Scale the vector === */
   for(int j=i; j<n; j+=BLOCK_SIZE)
      dx0[j] = MAGMA_minproduct_C_MUL(dx0[j], scale);

   /* === Make temporary the first element to 1; value is stored in dAkk === */
   if (i==0)
     dx0[0] = MAGMA_minproduct_C_ONE;
}


#define BLOCK_SIZE2 192
#if (defined(PRECISION_z) || defined(PRECISION_d))
  #define TOL 1.e-8
#else
  #define TOL 1.e-4
#endif

__global__ void
magma_minproduct_cgemv_kernel_adjust(int n, int k, magma_minproductFloatComplex * A, int lda, 
                          magma_minproductFloatComplex *B, int ldb, magma_minproductFloatComplex *C,
                          float *xnorm, float *xnorm2, magma_minproductFloatComplex *Akk, int *lsticc, int *lsticcs)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE2 * blockIdx.x;
    magma_minproductFloatComplex sum;
    float temp, oldnorm;

    if (j<n) {
      B += j;
      sum = MAGMA_minproduct_C_CNJG( B[(k-1)*ldb] );
      // sum = MAGMA_minproduct_C_ZERO;
      for(int m=0; m<k-1; m++) {
         sum += MAGMA_minproduct_C_MUL( MAGMA_minproduct_C_CNJG( B[m*ldb] ), A[m*lda] );
      }
      C[j*lda] -= sum;

      oldnorm = xnorm[j];
      temp = MAGMA_minproduct_C_ABS( C[j*lda] ) / oldnorm;
      temp  = (1.0 + temp) * (1.0 - temp);
      temp  = oldnorm * sqrt(temp);

      xnorm[j] = temp;

      // Below 'j' was 'i'; was that a bug?
      float temp2 = xnorm[j] / xnorm2[j];
      temp2 = temp*(temp2 * temp2);
      if (temp2 <= TOL){
         *lsticc = 1;
         lsticcs[j] = 1;
      }
    }

   if (j==0)
       A[(k-1)*lda] = *Akk;
  
/*
    __syncthreads();
    // Check if the norm has to be recomputed 
    if (blockIdx.x==0) {
       //if (2.*temp < oldnorm) {
           //printf("recompute norm\n");
           magma_minproductFloatComplex *dx = C+blockIdx.x*lda+1;
           __shared__ float sum[ BLOCK_SIZE2 ];
           float re, lsum;
 
           // get norm of dx
           lsum = 0;
           for( int k = i; k < n1; k += BLOCK_SIZE2 ) {

               #if (defined(PRECISION_s) || defined(PRECISION_d))
                   re = dx[k];
                   lsum += re*re;
               #else
                   re = MAGMA_minproduct_C_REAL( dx[k] );
                   float im = MAGMA_minproduct_C_IMAG( dx[k] );
                   lsum += re*re + im*im;
               #endif
           }
           sum[i] = lsum;
           magma_minproduct_sum_reduce< BLOCK_SIZE2 >( i, sum );

           if (i==0){
             printf("adjusted = %f recomputed = %f\n", xnorm[blockIdx.x], sqrt(sum[0])); 
             xnorm[blockIdx.x] = sqrt(sum[0]);
           }
      }
 //   }
*/
}

__global__ void
magma_minproductblas_scnrm2_check_kernel(int m, magma_minproductFloatComplex *da, int ldda, 
                              float *dxnorm, float *dxnorm2, 
                              int *dlsticc, int *dlsticcs)
{
    const int i = threadIdx.x;
    magma_minproductFloatComplex *dx = da + blockIdx.x * ldda;

    __shared__ float sum[ BLOCK_SIZE ];
    float re, lsum;

    if (blockIdx.x == 0 && i==0)
       *dlsticc = 0;

    // get norm of dx only if lsticc[blockIdx] != 0
    if( dlsticcs[blockIdx.x] == 0 ) 
        return;
    else
        dlsticcs[blockIdx.x] = 0;

    lsum = 0;
    for( int j = i; j < m; j += BLOCK_SIZE ) {

#if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
#else
        re = MAGMA_minproduct_C_REAL( dx[j] );
        float im = MAGMA_minproduct_C_IMAG( dx[j] );
        lsum += re*re + im*im;
#endif

    }
    sum[i] = lsum;
    magma_minproduct_sum_reduce< BLOCK_SIZE >( i, sum );

    if (i==0){
      dxnorm[blockIdx.x]  = sqrt(sum[0]);
      dxnorm2[blockIdx.x] = sqrt(sum[0]);
    }
}


/* --------------------------------------------------------------------------- */



/**
    Purpose
    -------
    CLAQPS computes a step of QR factorization with column pivoting
    of a complex M-by-N matrix A by using Blas-3.  It tries to factorize
    NB columns from A starting from the row OFFSET+1, and updates all
    of the matrix with Blas-3 xGEMM.

    In some cases, due to catastrophic cancellations, it cannot
    factorize NB columns.  Hence, the actual number of factorized
    columns is returned in KB.

    Block A(1:OFFSET,1:N) is accordingly pivoted, but not factorized.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A. M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A. N >= 0

    @param[in]
    offset  INTEGER
            The number of rows of A that have been factorized in
            previous steps.

    @param[in]
    NB      INTEGER
            The number of columns to factorize.

    @param[out]
    kb      INTEGER
            The number of columns actually factorized.

    @param[in,out]
    A       COMPLEX array, dimension (LDDA,N)
            On entry, the M-by-N matrix A.
            On exit, block A(OFFSET+1:M,1:KB) is the triangular
            factor obtained and block A(1:OFFSET,1:N) has been
            accordingly pivoted, but no factorized.
            The rest of the matrix, block A(OFFSET+1:M,KB+1:N) has
            been updated.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A. LDDA >= max(1,M).

    @param[in,out]
    jpvt    INTEGER array, dimension (N)
            JPVT(I) = K <==> Column K of the full matrix A has been
            permuted into position I in AP.

    @param[out]
    dtau    COMPLEX array, dimension (KB)
            The scalar factors of the elementary reflectors.

    @param[in,out]
    dvn1    REAL array, dimension (N)
            The vector with the partial column norms.

    @param[in,out]
    dvn2    REAL array, dimension (N)
            The vector with the exact column norms.

    @param[in,out]
    dauxv   COMPLEX array, dimension (NB)
            Auxiliar vector.

    @param[in,out]
    dF       COMPLEX array, dimension (LDDF,NB)
            Matrix F**H = L * Y**H * A.

    @param[in]
    lddf    INTEGER
            The leading dimension of the array F. LDDF >= max(1,N).

    @ingroup magma_minproduct_cgeqp3_aux
    ********************************************************************/
extern "C" magma_minproduct_int_t
magma_minproduct_claqps3_gpu(
    magma_minproduct_int_t m, magma_minproduct_int_t n, magma_minproduct_int_t offset,
    magma_minproduct_int_t nb, magma_minproduct_int_t *kb,
    magma_minproductFloatComplex_ptr dA,  magma_minproduct_int_t ldda,
    magma_minproduct_int_t *jpvt,
    magma_minproductFloatComplex_ptr dtau, 
    magma_minproductFloat_ptr dvn1, magma_minproductFloat_ptr dvn2,
    magma_minproductFloatComplex_ptr dauxv,
    magma_minproductFloatComplex_ptr dF,  magma_minproduct_int_t lddf)
{
#define dA(i_, j_) (dA + (i_) + (j_)*(ldda))
#define dF(i_, j_) (dF + (i_) + (j_)*(lddf))

    magma_minproductFloatComplex c_zero    = MAGMA_minproduct_C_MAKE( 0.,0.);
    magma_minproductFloatComplex c_one     = MAGMA_minproduct_C_MAKE( 1.,0.);
    magma_minproductFloatComplex c_neg_one = MAGMA_minproduct_C_MAKE(-1.,0.);
    magma_minproduct_int_t ione = 1;
    
    magma_minproduct_int_t i__1, i__2;
    
    magma_minproduct_int_t k, rk;
    magma_minproductFloatComplex tauk;
    magma_minproduct_int_t pvt, itemp;

    magma_minproductFloatComplex_ptr dAkk = dauxv;
    dauxv += 1;

    int lsticc, *dlsticc, *dlsticcs;
    magma_minproduct_malloc( (void**) &dlsticcs, (n+1)*sizeof(int) );
    hipMemset( dlsticcs, 0, (n+1)*sizeof(int) );
    dlsticc = dlsticcs + n;
 
    // float tol3z = magma_minproduct_ssqrt( lapackf77_slamch("Epsilon"));

    lsticc = 0;
    k = 0;
    while( k < nb && lsticc == 0 ) {
        rk = offset + k;
        
        /* Determine ith pivot column and swap if necessary */
        pvt = k - 1 + magma_minproduct_isamax( n-k, &dvn1[k], ione );

        if (pvt != k) {
            magma_minproductblas_cswap( k, dF(pvt,0), lddf, dF(k,0), lddf);
            itemp     = jpvt[pvt];
            jpvt[pvt] = jpvt[k];
            jpvt[k]   = itemp;
            #if (defined(PRECISION_d) || defined(PRECISION_z))
                //magma_minproduct_dswap( 1, &dvn1[pvt], 1, &dvn1[k], 1 );
                //magma_minproduct_dswap( 1, &dvn2[pvt], 1, &dvn2[k], 1 );
                magma_minproduct_dswap( 2, &dvn1[pvt], n+offset, &dvn1[k], n+offset);
            #else
                //magma_minproduct_sswap( 1, &dvn1[pvt], 1, &dvn1[k], 1 );
                //magma_minproduct_sswap( 1, &dvn2[pvt], 1, &dvn2[k], 1 );
                magma_minproduct_sswap(2, &dvn1[pvt], n+offset, &dvn1[k], n+offset);
            #endif
        }

        /* Apply previous Householder reflectors to column K:
           A(RK:M,K) := A(RK:M,K) - A(RK:M,1:K-1)*F(K,1:K-1)'  */
        magma_minproduct_cswap_gemv_kernel<<< (m + BLOCK_SIZE1-1) / BLOCK_SIZE1, BLOCK_SIZE1, 0, magma_minproduct_stream >>> 
                              ( m, rk, k, dA(0, 0), ldda, dF(k,  0), lddf, dA(0, k), dA(0,pvt));
                                 
        /*  Generate elementary reflector H(k). */
        magma_minproduct_cscale_kernel<<< 1, BLOCK_SIZE, 0, magma_minproduct_stream >>>
               (m-rk, dA(rk, k),   &dtau[k], &dvn1[k], dAkk);
        // printf("m-rk = %d\n", m-rk);

        /* Compute Kth column of F:
           Compute  F(K+1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) on the GPU */
        if (k < n-1) {
            magma_minproduct_cgetvector( 1, &dtau[k], 1, &tauk, 1 );
            magma_minproduct_cgemv( Magma_minproductConjTrans, m-rk, n,
                         tauk,   dA( rk,  0 ), ldda,
                                 dA( rk,  k   ), 1,
                         c_zero, dauxv, 1 );
            if (k==0) 
               magma_minproductblas_clacpy(Magma_minproductUpperLower, n-k-1, 1, dauxv+k+1, n-k-1, dF( k+1, k   ), n-k-1);
        }
        
        /* Incremental updating of F:
           F(1:N,K) := F(1:N,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K). 
           F(1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K)
                    := tau(K)(A(RK:M,K+1:N)' - F(1:N,1:K-1)*A(RK:M,1:K-1)') A(RK:M,K)  
           so, F is (updated A)*V */
        if (k > 0) {
            /* I think we only need stricly lower-triangular part */
            magma_minproduct_cgemv_kernel<<< (n-k-1 + BLOCK_SIZE1 -1)/BLOCK_SIZE1, BLOCK_SIZE1, 0, magma_minproduct_stream >>>
                       (n-k-1, k, dF(k+1,0), lddf, dauxv, dauxv+k+1, dF(k+1,k));
        }
        
        /* Update the current row of A:
           A(RK,K+1:N) := A(RK,K+1:N) - A(RK,1:K)*F(K+1:N,1:K)'.               */
        if (k < n-1) {
            i__1 = n - k - 1;
            i__2 = k + 1;
            /* left-looking update of rows,                     *
             * since F=A**H v with original A, so no right-looking */
            magma_minproduct_cgemv_kernel_adjust<<<(n-k-1 + BLOCK_SIZE2-1)/BLOCK_SIZE2, BLOCK_SIZE2, 0, magma_minproduct_stream>>>
                           (n-k-1, k+1, dA(rk, 0  ), ldda, dF(k+1,0  ), lddf, dA(rk, k+1),
                           &dvn1[k+1], &dvn2[k+1], dAkk, dlsticc, dlsticcs);
            magma_minproduct_getmatrix(1,1, sizeof(int), dlsticc, 1, &lsticc, 1); 
 
            // TTT: force not to recompute; has to be finally commented 
            if ( nb<3 )
            lsticc = 0; 

            // printf("k=%d n-k = %d\n", k, n-k);
            // forcing recompute works! - forcing it requires changing dlsticcs as well, e.g.,
            // can be done in the kernel directly (magma_minproductblas_scnrm2_check_kernel)
            // if (k==16) lsticc = 1;
        }
        
        /* Update partial column norms. */
/*
        if (rk < min(m, n+offset)-1){
           magma_minproductblas_scnrm2_row_check_adjust(n-k-1, tol3z, &dvn1[k+1], 
                                             &dvn2[k+1], dA(rk,k+1), ldda, lsticcs); 
        }

        #if defined(PRECISION_d) || defined(PRECISION_z)
            magma_minproduct_sgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
        #else
            magma_minproduct_sgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
        #endif
*/

        if (k>=n-1)
           magma_minproductblas_clacpy(Magma_minproductUpperLower, 1, 1, dAkk, 1, dA(rk, k), 1);

        ++k;
    }
    // leave k as the last column done
    --k;
    *kb = k + 1;
    rk = offset + *kb - 1;

    //printf("actually factored = %d",*kb);

    /* Apply the block reflector to the rest of the matrix:
       A(OFFSET+KB+1:M,KB+1:N) := A(OFFSET+KB+1:M,KB+1:N) - 
                                  A(OFFSET+KB+1:M,1:KB)*F(KB+1:N,1:KB)'  */
    if (*kb < min(n, m - offset)-1) {
        i__1 = m - rk - 1;
        i__2 = n - *kb;
        
        magma_minproduct_cgemm( Magma_minproductNoTrans, Magma_minproductConjTrans, i__1, i__2, *kb,
                     c_neg_one, dA(rk+1, 0  ), ldda,
                                dF(*kb,  0  ), lddf,
                     c_one,     dA(rk+1, *kb), ldda );
    }

    /* Recomputation of difficult columns. */
    if( lsticc > 0 ) {
        // printf( " -- recompute dnorms --\n" );
        //magma_minproductblas_scnrm2_check(m-rk-1, n-*kb, A(rk+1,rk+1), lda,
        //                       &dvn1[rk+1], &dvn2[rk+1], dlsticcs);
       
        // There is a bug when we get to recompute  
        magma_minproductblas_scnrm2_check_kernel<<< n-*kb, BLOCK_SIZE >>>
                     ( m-rk-1, dA(rk+1,rk+1), ldda, &dvn1[rk+1], &dvn2[rk+1], dlsticc, dlsticcs);
    }
    magma_minproduct_free(dlsticcs);
    
    return MAGMA_minproduct_SUCCESS;
} /* magma_minproduct_claqps */
