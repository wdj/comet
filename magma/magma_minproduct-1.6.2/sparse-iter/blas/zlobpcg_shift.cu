#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_minproduct.h"

__global__ void
magma_minproduct_zlobpcg_shift_kernel( 
    magma_minproduct_int_t num_rows, 
    magma_minproduct_int_t num_vecs, 
    magma_minproduct_int_t shift, 
    magma_minproductDoubleComplex * x )
{

    int idx = threadIdx.x ;     // thread in row
    int row = blockIdx.y * gridDim.x + blockIdx.x; // global block index

    if( row<num_rows){
        magma_minproductDoubleComplex tmp = x[idx];
        __syncthreads();

        if( idx > shift-1 ){
            idx-=shift;
            x[idx] = tmp;
            __syncthreads();
        }

    }
}




/**
    Purpose
    -------
    
    For a Block-LOBPCG, the set of residuals (entries consecutive in memory)  
    shrinks and the vectors are shifted in case shift residuals drop below 
    threshold. The memory layout of x is:

        / x1[0] x2[0] x3[0] \
        | x1[1] x2[1] x3[1] |
    x = | x1[2] x2[2] x3[2] | = x1[0] x2[0] x3[0] x1[1] x2[1] x3[1] x1[2] .
        | x1[3] x2[3] x3[3] |
        \ x1[4] x2[4] x3[4] /
    
    Arguments
    ---------

    @param[in]
    num_rows    magma_minproduct_int_t
                number of rows

    @param[in]
    num_vecs    magma_minproduct_int_t
                number of vectors

    @param[in]
    shift       magma_minproduct_int_t
                shift number

    @param[in/out]
    x           magma_minproductDoubleComplex_ptr 
                input/output vector x

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_zaux
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_zlobpcg_shift(
    magma_minproduct_int_t num_rows,
    magma_minproduct_int_t num_vecs, 
    magma_minproduct_int_t shift,
    magma_minproductDoubleComplex_ptr x,
    magma_minproduct_queue_t queue )
{
    magma_minproduct_int_t num_threads = num_vecs;
    // every thread handles one row containing the 
    if (  num_threads > 1024 )
        printf("error: too many threads requested.\n");

    int Ms = num_threads * sizeof( magma_minproductDoubleComplex );
    if (  Ms > 1024*8 )
        printf("error: too much shared memory requested.\n");

    dim3 block( num_threads, 1, 1 );

    int dimgrid1 = (int) sqrt( (double) num_rows);
    int dimgrid2 = magma_minproduct_ceildiv( num_rows, dimgrid1 );

    dim3 grid( dimgrid1, dimgrid2, 1);

    magma_minproduct_zlobpcg_shift_kernel<<< grid, block, Ms, queue >>>
            ( num_rows, num_vecs, shift, x );


    return MAGMA_minproduct_SUCCESS;
}



