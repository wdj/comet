#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarfbx.cu normal z -> c, Fri Jan 30 19:00:08 2015

*/
#include "common_magma_tally2.h"
#include "commonblas_c.h"
#include "magma_tally2_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512




//==============================================================================
extern "C"
__global__ void 
magma_tally2_cgemv_kernel1(int m, const magma_tally2FloatComplex * __restrict__ V, int ldv, 
                    const magma_tally2FloatComplex * __restrict__ c, 
                    magma_tally2FloatComplex *dwork)
{
    const int i = threadIdx.x;
    const magma_tally2FloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magma_tally2FloatComplex sum[ BLOCK_SIZE ];
    magma_tally2FloatComplex lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally2_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally2_C_MUL( MAGMA_tally2_C_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_tally2_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_tally2_cgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        CGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
    ----------------------------------------------------------------------------- */
extern "C"
__global__ void
magma_tally2_cgemv_kernel3(int m, const magma_tally2FloatComplex * __restrict__ V, int ldv, magma_tally2FloatComplex *c,
                    magma_tally2FloatComplex *dwork, magma_tally2FloatComplex *tau)
{
    const int i = threadIdx.x;
    const magma_tally2FloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magma_tally2FloatComplex sum[ BLOCK_SIZE ];
    magma_tally2FloatComplex lsum;

    if (i==0)
       c[0] = MAGMA_tally2_C_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally2_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally2_C_MUL( MAGMA_tally2_C_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_tally2_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================
extern "C"
__global__ void
magma_tally2_cgemv_kernel2(int m, int n, const magma_tally2FloatComplex * __restrict__ V, int ldv, 
                    const magma_tally2FloatComplex * __restrict__ x, magma_tally2FloatComplex *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    magma_tally2FloatComplex lsum;

    V += j;

    lsum = MAGMA_tally2_C_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_tally2_C_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a complex block reflector H to a complex vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the complex k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a complex block of
    k elementary reflectors. 
*/
extern "C" void
magma_tally2_clarfbx_gpu(
    magma_tally2_int_t m, magma_tally2_int_t k,
    magma_tally2FloatComplex_ptr V,  magma_tally2_int_t ldv,
    magma_tally2FloatComplex_ptr dT, magma_tally2_int_t ldt,
    magma_tally2FloatComplex_ptr c,
    magma_tally2FloatComplex_ptr dwork)
{
    /* dwork = V**H c     */
    magma_tally2_cgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_tally2_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_tally2_ctrmv_tkernel<<< k, k, 0, magma_tally2_stream >>>( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_tally2_cgemv_kernel2<<< blocks3, threads3, 0, magma_tally2_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
