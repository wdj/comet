#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c
       @author Azzam Haidar

*/
#include "common_magma_tally3.h"
#include "magma_tally3_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================
//==============================================================================

__global__
void magma_tally3_zlarf_kernel( int m, const magma_tally3DoubleComplex *dv, const magma_tally3DoubleComplex *dtau,
                         magma_tally3DoubleComplex *dc, int lddc )
{
    if ( !MAGMA_tally3_Z_EQUAL(*dtau, MAGMA_tally3_Z_ZERO) ) {
        const int tx = threadIdx.x;
        dc = dc + blockIdx.x * lddc;

        __shared__ magma_tally3DoubleComplex sum[ BLOCK_SIZE ];
        magma_tally3DoubleComplex tmp;

        /* perform  w := v**H * C  */
        if (tx==0)
            tmp = dc[0]; //since V[0] should be one
        else
            tmp = MAGMA_tally3_Z_ZERO;
        for( int j = tx+1; j < m; j += BLOCK_SIZE ){
            tmp += MAGMA_tally3_Z_MUL( MAGMA_tally3_Z_CNJG( dv[j] ), dc[j] );
        }
        sum[tx] = tmp;
        magma_tally3_sum_reduce< BLOCK_SIZE >( tx, sum );

        /*  C := C - v * w  */
        __syncthreads();
        tmp = - MAGMA_tally3_Z_CNJG(*dtau) * sum[0];
        for( int j = m-tx-1; j>0 ; j -= BLOCK_SIZE )
             dc[j] += tmp * dv[j];

        if(tx==0) dc[0] += tmp;
    }
}

//==============================================================================
//==============================================================================

__global__
void magma_tally3_zlarf_smkernel( int m, int n, magma_tally3DoubleComplex *dv, magma_tally3DoubleComplex *dtau,
                           magma_tally3DoubleComplex *dc, int lddc )
{
    if ( ! MAGMA_tally3_Z_EQUAL(*dtau, MAGMA_tally3_Z_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            dc = dc + k * lddc;
    
            __shared__ magma_tally3DoubleComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            magma_tally3DoubleComplex lsum;
    
            /*  w := v**H * C  */
            lsum = MAGMA_tally3_Z_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_tally3_Z_MUL( MAGMA_tally3_Z_ONE, dc[j] );
                else
                   lsum += MAGMA_tally3_Z_MUL( MAGMA_tally3_Z_CNJG( dv[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_tally3_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            magma_tally3DoubleComplex z__1 = - MAGMA_tally3_Z_CNJG(*dtau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * dv[j];
            }
        }
    }
}

//==============================================================================

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v**H
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H**H (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_tally3_zlarf_sm(magma_tally3_int_t m, magma_tally3_int_t n, magma_tally3DoubleComplex *dv, magma_tally3DoubleComplex *dtau,
               magma_tally3DoubleComplex *dc, magma_tally3_int_t lddc)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_tally3_zlarf_smkernel<<< blocks, threads, 0, magma_tally3_stream >>>( m, n, dv, dtau, dc, lddc );
}
//==============================================================================
/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v**H
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H**H (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

 */

extern "C" magma_tally3_int_t
magma_tally3_zlarf_gpu(
    magma_tally3_int_t m,  magma_tally3_int_t n,
    magma_tally3DoubleComplex_const_ptr dv,
    magma_tally3DoubleComplex_const_ptr dtau,
    magma_tally3DoubleComplex_ptr dC,  magma_tally3_int_t lddc)
{
    dim3 grid( n, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    if ( n > 0 ) {
        magma_tally3_zlarf_kernel<<< grid, threads, 0, magma_tally3_stream >>>( m, dv, dtau, dC, lddc);
    }

    // The computation can be done on 1 SM with the following routine.
    // magma_tally3_zlarf_sm(m, n, dv, dtau, dc, lddc);

    return MAGMA_tally3_SUCCESS;
}

//==============================================================================
