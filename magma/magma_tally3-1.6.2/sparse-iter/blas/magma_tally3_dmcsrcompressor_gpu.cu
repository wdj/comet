#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from magma_tally3_zmcsrcompressor_gpu.cu normal z -> d, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magma_tally3sparse.h"

#define BLOCK_SIZE1 256
#define BLOCK_SIZE2 1


// copy nonzeros into new structure
__global__ void
magma_tally3_dmcsrgpu_kernel1( int num_rows,
                 double *A_val,
                 magma_tally3_index_t *A_rowptr,
                 magma_tally3_index_t *A_colind,
                 double *B_val,
                 magma_tally3_index_t *B_rowptr,
                 magma_tally3_index_t *B_colind ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        double zero = MAGMA_tally3_D_ZERO;
        int start = A_rowptr[ row ];
        int new_location = start;
        int end = A_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
       //         B_val[new_location] = A_val[j];
       //         B_colind[new_location] = A_colind[j];
                new_location++;
            }
        }
        // this is not a correctr rowpointer! this is nn_z in this row!
        B_rowptr[ row ] = new_location-start;
    }
}


// generate a valid rowpointer
__global__ void
magma_tally3_dmcsrgpu_kernel2( int num_rows,
                 magma_tally3_index_t *B_rowptr,
                 magma_tally3_index_t *A_rowptr ){

    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int j, nnz = 0;

    if( idx == 0 ){
    A_rowptr[ 0 ] = nnz;
        for( j=0; j<num_rows; j++ ){
            nnz+=B_rowptr[ j ];
            A_rowptr[ j+1 ] = nnz;
        }
    }
}



// copy new structure into original matrix
__global__ void
magma_tally3_dmcsrgpu_kernel3( int num_rows,
                 double *B_val,
                 magma_tally3_index_t *B_rowptr,
                 magma_tally3_index_t *B_colind,
                 magma_tally3_index_t *B2_rowptr,
                 double *A_val,
                 magma_tally3_index_t *A_rowptr,
                 magma_tally3_index_t *A_colind
                                            ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j, new_location;
    
    if(row<num_rows){
    new_location = A_rowptr[ row ];
        int start = B2_rowptr[ row ];
        int end = B2_rowptr[ row+1 ];
        double zero = MAGMA_tally3_D_ZERO;
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
                B_val[new_location] = A_val[j];
                B_colind[new_location] = A_colind[j];
                new_location++;
            }
               // A_val[ j ] = B_val[ j ];
               // A_colind[ j ] = B_colind[ j ];
        }
    }
}


/**
    Purpose
    -------

    Removes zeros in a CSR matrix. This is a GPU implementation of the
    CSR compressor.

    Arguments
    ---------

    @param
    A           magma_tally3_d_matrix*
                input/output matrix
    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_daux
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_dmcsrcompressor_gpu(
    magma_tally3_d_matrix *A,
    magma_tally3_queue_t queue )
{
    magma_tally3_int_t info = 0;
    magma_tally3_d_matrix B={Magma_tally3_CSR}, B2={Magma_tally3_CSR};
    magma_tally3_d_matrix dA={Magma_tally3_CSR}, CSRA={Magma_tally3_CSR};
    magma_tally3_index_t *cputmp = NULL;
    
    if ( A->memory_location == Magma_tally3_DEV && A->storage_type == Magma_tally3_CSR ) {

        CHECK( magma_tally3_index_malloc( &B.drow, A->num_rows + 1 ));
        CHECK( magma_tally3_index_malloc( &B2.drow, A->num_rows + 1 ));
        
        magma_tally3_index_copyvector( (A->num_rows+1), A->drow, 1, B2.drow, 1 );

        dim3 grid1( magma_tally3_ceildiv( A->num_rows, BLOCK_SIZE1 ) );

        // copying the nonzeros into B and write in B.drow how many there are
        magma_tally3_dmcsrgpu_kernel1<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, A->dval, A->drow, A->dcol, B.dval, B.drow, B.dcol );

        // correct the row pointer
        dim3 grid2( 1, 1, 1);
        magma_tally3_dmcsrgpu_kernel2<<< grid2, BLOCK_SIZE2, 0, queue >>>
                ( A->num_rows, B.drow, A->drow );
        // access the true number of nonzeros

        CHECK( magma_tally3_index_malloc_cpu( &cputmp, 1 ));

        magma_tally3_index_getvector( 1, A->row+(A->num_rows), 1, cputmp, 1 );
        A->nnz = (magma_tally3_int_t) cputmp[0];

        // reallocate with right size
        CHECK( magma_tally3_dmalloc( &B.dval, A->nnz ));
        CHECK( magma_tally3_index_malloc( &B.dcol, A->nnz ));
        
        // copy correct values back
        magma_tally3_dmcsrgpu_kernel3<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, B.dval, B.drow, B.dcol, B2.drow, A->dval, A->drow, A->dcol );

        magma_tally3_free( A->dcol );
        magma_tally3_free( A->dval );

        A->dcol = B.dcol;
        A->dval = B.dval;


    }
    else {
        magma_tally3_storage_t A_storage = A->storage_type;
        magma_tally3_location_t A_location = A->memory_location;
        CHECK( magma_tally3_dmconvert( *A, &CSRA, A->storage_type, Magma_tally3_CSR, queue ));
        CHECK( magma_tally3_dmtransfer( *A, &dA, A->memory_location, Magma_tally3_DEV, queue ));

        CHECK( magma_tally3_dmcsrcompressor_gpu( &dA, queue ));

        magma_tally3_dmfree( &dA, queue );
        magma_tally3_dmfree( A, queue );
        CHECK( magma_tally3_dmtransfer( dA, &CSRA, Magma_tally3_DEV, A_location, queue ));
        CHECK( magma_tally3_dmconvert( CSRA, A, Magma_tally3_CSR, A_storage, queue ));
        magma_tally3_dmfree( &dA, queue );
        magma_tally3_dmfree( &CSRA, queue );

    }
    
cleanup:
    magma_tally3_dmfree( &dA, queue );
    magma_tally3_dmfree( &CSRA, queue );
    magma_tally3_free( B2.drow );
    magma_tally3_free( B.drow );
    return info;
}


