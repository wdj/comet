#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_minproduct.h"

#define BLOCK_SIZE 512


__global__ void 
zmgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        magma_minproductDoubleComplex alpha, 
        magma_minproductDoubleComplex * dval, 
        magma_minproduct_index_t * dcolind,
        magma_minproductDoubleComplex * dx,
        magma_minproductDoubleComplex beta, 
        magma_minproductDoubleComplex * dy)
{
    extern __shared__ magma_minproductDoubleComplex dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_minproduct_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_minproductDoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_minproduct_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_minproductDoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductDoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_minproductDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductDoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_minproductDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_zblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_zmgeelltmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t num_vecs,
    magma_minproduct_int_t nnz_per_row,
    magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductDoubleComplex_ptr dx,
    magma_minproductDoubleComplex beta,
    magma_minproductDoubleComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    dim3 grid( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( magma_minproductDoubleComplex ); // num_vecs vectors 
    zmgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_minproduct_SUCCESS;
}



