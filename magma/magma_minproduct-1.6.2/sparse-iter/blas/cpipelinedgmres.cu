#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zpipelinedgmres.cu normal z -> c, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magma_minproduct.h"

#define COMPLEX

#define BLOCK_SIZE 512


template< int n >
__device__ void sum_reduce( /*int n,*/ int i, float* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  
        __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  
        __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  
        __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  
        __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  
        __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  
        __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  
        __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  
        __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  
        __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  
        __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  
        __syncthreads(); }
}

__global__ void
magma_minproduct_cpipelined_correction( 
    int n,  
    int k,
    magma_minproductFloatComplex * skp, 
    magma_minproductFloatComplex * r,
    magma_minproductFloatComplex * v )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float zz= 0.0, tmp= 0.0;

    extern __shared__ magma_minproductFloatComplex temp[];    
    
    temp[ i ] = ( i < k ) ? skp[ i ] * skp[ i ] : MAGMA_minproduct_C_MAKE( 0.0, 0.0);
    __syncthreads();
     if (i < 64) { temp[ i ] += temp[ i + 64 ]; } __syncthreads(); 
     if( i < 32 ){
        temp[ i ] += temp[ i + 32 ];__syncthreads();    
        temp[ i ] += temp[ i + 16 ];__syncthreads(); 
        temp[ i ] += temp[ i +  8 ];__syncthreads(); 
        temp[ i ] += temp[ i +  4 ];__syncthreads(); 
        temp[ i ] += temp[ i +  2 ];__syncthreads(); 
        temp[ i ] += temp[ i +  1 ];__syncthreads();      
    }
    if( i == 0 ){
        tmp = MAGMA_minproduct_C_REAL( temp[ i ] );
        zz = MAGMA_minproduct_C_REAL( skp[(k)] );
        skp[k] = MAGMA_minproduct_C_MAKE( sqrt(zz-tmp),0.0 );
    }
}

__global__ void
magma_minproduct_cpipelined_copyscale( 
    int n,  
    int k,
    magma_minproductFloatComplex * skp, 
    magma_minproductFloatComplex * r,
    magma_minproductFloatComplex * v )
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    magma_minproductFloatComplex rr=skp[k];

    if( i<n ){
        v[i] =  r[i] * 1.0 / rr;

    }
}

//----------------------------------------------------------------------------//

__global__ void
magma_minproduct_cpipelinedscnrm2_kernel( 
    int m, 
    magma_minproductFloatComplex * da, 
    int ldda, 
    magma_minproductFloatComplex * dxnorm )
{
    const int i = threadIdx.x;
    magma_minproductFloatComplex_ptr dx = da + blockIdx.x * ldda;

    __shared__ float sum[ 512 ];
    float re, lsum;

    // get norm of dx
    lsum = 0;
    for( int j = i; j < m; j += 512 ) {
        #ifdef REAL
            re = dx[j];
            lsum += re*re;
        #else
            re = MAGMA_minproduct_C_REAL( dx[j] );
            float im = MAGMA_minproduct_C_IMAG( dx[j] );
            lsum += re*re + im*im;
        #endif
    }
    sum[i] = lsum;
    sum_reduce< 512 >( i, sum );

    if (i==0)
        dxnorm[blockIdx.x] = MAGMA_minproduct_C_MAKE( sqrt(sum[0]), 0.0 );
}

//----------------------------------------------------------------------------//

__global__ void
magma_minproduct_cpipelinesscale( 
    int n, 
    magma_minproductFloatComplex * r, 
    magma_minproductFloatComplex * drnorm )
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i<n ){
        r[i] =  r[i] * 1.0 / drnorm[0];
    }
}

/**
    Purpose
    -------

    Computes the correction term of the pipelined GMRES according to P. Ghysels 
    and scales and copies the new search direction
    
    Returns the vector v = r/ ( skp[k] - (sum_i=1^k skp[i]^2) ) .

    Arguments
    ---------

    @param[in]
    n           int
                length of v_i

    @param[in]
    k           int
                # skp entries v_i^T * r ( without r )

    @param[in]
    r           magma_minproductFloatComplex_ptr 
                vector of length n

    @param[in]
    v           magma_minproductFloatComplex_ptr 
                vector of length n
                
    @param[in]  
    skp         magma_minproductFloatComplex_ptr 
                array of parameters

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_caux
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_ccopyscale(
    int n, 
    int k,
    magma_minproductFloatComplex_ptr r, 
    magma_minproductFloatComplex_ptr v,
    magma_minproductFloatComplex_ptr skp,
    magma_minproduct_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_minproduct_ceildiv( k, BLOCK_SIZE ) );
    unsigned int Ms =   Bs.x * sizeof( magma_minproductFloatComplex ); 

    dim3 Gs2( magma_minproduct_ceildiv( n, BLOCK_SIZE ) );


    magma_minproduct_cpipelined_correction<<<Gs, Bs, Ms, queue >>>
                                            ( n, k, skp, r, v );
    magma_minproduct_cpipelined_copyscale<<<Gs2, Bs, 0, queue >>>
                                            ( n, k, skp, r, v );

    return MAGMA_minproduct_SUCCESS;
}


extern "C" magma_minproduct_int_t
magma_minproduct_scnrm2scale(
    int m, 
    magma_minproductFloatComplex_ptr r, 
    int lddr, 
    magma_minproductFloatComplex_ptr drnorm,
    magma_minproduct_queue_t queue )
{
    dim3  blocks( 1 );
    dim3 threads( 512 );
    magma_minproduct_cpipelinedscnrm2_kernel<<< blocks, threads, 0, queue >>>
                                ( m, r, lddr, drnorm );

    dim3 Bs( BLOCK_SIZE );
    dim3 Gs2( magma_minproduct_ceildiv( m, BLOCK_SIZE ) );
    magma_minproduct_cpipelinesscale<<<Gs2, Bs, 0, queue >>>( m, r, drnorm );

    return MAGMA_minproduct_SUCCESS;
}

