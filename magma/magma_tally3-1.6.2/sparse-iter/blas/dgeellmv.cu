#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeellmv.cu normal z -> d, Sun May  3 11:22:58 2015

*/

#include "common_magma_tally3.h"

#define BLOCK_SIZE 512


// ELLPACK SpMV kernel
//Michael Garland
__global__ void 
dgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double * dval, 
    magma_tally3_index_t * dcolind,
    double * dx,
    double beta, 
    double * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_tally3_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            double val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELLPACK SpMV kernel
//Michael Garland
__global__ void 
dgeellmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double lambda, 
    double * dval, 
    magma_tally3_index_t * dcolind,
    double * dx,
    double beta, 
    int offset,
    int blocksize,
    magma_tally3_index_t * addrows,
    double * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_tally3_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            double val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}





/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLPACK.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally3_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally3_int_t
                number of rows in A

    @param[in]
    n           magma_tally3_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally3_int_t
                number of elements in the longest row 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    dval        magma_tally3Double_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally3Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally3Double_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[out]
    dy          magma_tally3Double_ptr
                input/output vector y

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_dblas
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_dgeellmv(
    magma_tally3_trans_t transA,
    magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3_int_t nnz_per_row,
    double alpha,
    magma_tally3Double_ptr dval,
    magma_tally3Index_ptr dcolind,
    magma_tally3Double_ptr dx,
    double beta,
    magma_tally3Double_ptr dy,
    magma_tally3_queue_t queue )
{
    dim3 grid( magma_tally3_ceildiv( m, BLOCK_SIZE ) );
    magma_tally3_int_t threads = BLOCK_SIZE;
   dgeellmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally3_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELLPACK.
    It is the shifted version of the ELLPACK SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally3_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally3_int_t
                number of rows in A

    @param[in]
    n           magma_tally3_int_t
                number of columns in A 
    @param[in]
    nnz_per_row magma_tally3_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       double
                scalar multiplier
                
    @param[in]
    lambda      double
                scalar multiplier

    @param[in]
    dval        magma_tally3Double_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally3Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally3Double_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier
                
    @param[in]
    offset      magma_tally3_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally3_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally3Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally3Double_ptr
                input/output vector y

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_dblas
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_dgeellmv_shift(
    magma_tally3_trans_t transA,
    magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3_int_t nnz_per_row,
    double alpha,
    double lambda,
    magma_tally3Double_ptr dval,
    magma_tally3Index_ptr dcolind,
    magma_tally3Double_ptr dx,
    double beta,
    int offset,
    int blocksize,
    magma_tally3Index_ptr addrows,
    magma_tally3Double_ptr dy,
    magma_tally3_queue_t queue )
{
    dim3 grid( magma_tally3_ceildiv( m, BLOCK_SIZE ) );
    magma_tally3_int_t threads = BLOCK_SIZE;
   dgeellmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, lambda, dval, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy );


   return MAGMA_tally3_SUCCESS;
}



