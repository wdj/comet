#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeellmv.cu normal z -> s, Sun May  3 11:22:58 2015

*/

#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


// ELLPACK SpMV kernel
//Michael Garland
__global__ void 
sgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    float alpha, 
    float * dval, 
    magma_tally2_index_t * dcolind,
    float * dx,
    float beta, 
    float * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        float dot = MAGMA_tally2_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            float val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELLPACK SpMV kernel
//Michael Garland
__global__ void 
sgeellmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    float alpha, 
    float lambda, 
    float * dval, 
    magma_tally2_index_t * dcolind,
    float * dx,
    float beta, 
    int offset,
    int blocksize,
    magma_tally2_index_t * addrows,
    float * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        float dot = MAGMA_tally2_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            float val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}





/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLPACK.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 

    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    dval        magma_tally2Float_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally2Float_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[out]
    dy          magma_tally2Float_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_sblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_sgeellmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    float alpha,
    magma_tally2Float_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2Float_ptr dx,
    float beta,
    magma_tally2Float_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
   sgeellmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally2_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELLPACK.
    It is the shifted version of the ELLPACK SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       float
                scalar multiplier
                
    @param[in]
    lambda      float
                scalar multiplier

    @param[in]
    dval        magma_tally2Float_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally2Float_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier
                
    @param[in]
    offset      magma_tally2_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally2_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally2Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally2Float_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_sblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_sgeellmv_shift(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    float alpha,
    float lambda,
    magma_tally2Float_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2Float_ptr dx,
    float beta,
    int offset,
    int blocksize,
    magma_tally2Index_ptr addrows,
    magma_tally2Float_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
   sgeellmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, lambda, dval, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy );


   return MAGMA_tally2_SUCCESS;
}



