#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/
#include "common_magma_minproductsparse.h"

#define BLOCK_SIZE 512


#define PRECISION_z


// SELLC SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
__global__ void 
zgesellcmv_kernel(   
    int num_rows, 
    int num_cols,
    int blocksize,
    magma_minproductDoubleComplex alpha, 
    magma_minproductDoubleComplex * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproduct_index_t * drowptr,
    magma_minproductDoubleComplex * dx,
    magma_minproductDoubleComplex beta, 
    magma_minproductDoubleComplex * dy)
{
    // threads assigned to rows
    int Idx = blockDim.x * blockIdx.x + threadIdx.x ;
    int offset = drowptr[ blockIdx.x ];
    int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;
    if(Idx < num_rows ){
        magma_minproductDoubleComplex dot = MAGMA_minproduct_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n++){ 
            int col = dcolind [offset+ blocksize * n + threadIdx.x ];
            magma_minproductDoubleComplex val = dval[offset+ blocksize * n + threadIdx.x];
            if( val != 0){
                  dot=dot+val*dx[col];
            }
        }

        dy[ Idx ] = dot * alpha + beta * dy [ Idx ];
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is SELLC/SELLP.
    
    Arguments
    ---------

    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A

    @param[in]
    m           magma_minproduct_int_t
                number of rows in A

    @param[in]
    n           magma_minproduct_int_t
                number of columns in A 

    @param[in]
    blocksize   magma_minproduct_int_t
                number of rows in one ELL-slice

    @param[in]
    slices      magma_minproduct_int_t
                number of slices in matrix

    @param[in]
    alignment   magma_minproduct_int_t
                number of threads assigned to one row (=1)

    @param[in]
    alpha       magma_minproductDoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_minproductDoubleComplex_ptr
                array containing values of A in SELLC/P

    @param[in]
    dcolind     magma_minproductIndex_ptr
                columnindices of A in SELLC/P

    @param[in]
    drowptr     magma_minproductIndex_ptr
                rowpointer of SELLP

    @param[in]
    dx          magma_minproductDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_minproductDoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_minproductDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_zblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_zgesellcmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t blocksize,
    magma_minproduct_int_t slices,
    magma_minproduct_int_t alignment,
    magma_minproductDoubleComplex alpha,
    magma_minproductDoubleComplex_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductIndex_ptr drowptr,
    magma_minproductDoubleComplex_ptr dx,
    magma_minproductDoubleComplex beta,
    magma_minproductDoubleComplex_ptr dy,
    magma_minproduct_queue_t queue )
{
    // the kernel can only handle up to 65535 slices 
   // (~2M rows for blocksize 32)
   dim3 grid( slices, 1, 1);
   magma_minproduct_int_t threads = blocksize;
   zgesellcmv_kernel<<< grid, threads, 0, queue >>>
   ( m, n, blocksize, alpha,
        dval, dcolind, drowptr, dx, beta, dy );

   return MAGMA_minproduct_SUCCESS;
}

