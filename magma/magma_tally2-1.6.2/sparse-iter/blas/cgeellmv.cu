#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeellmv.cu normal z -> c, Sun May  3 11:22:58 2015

*/

#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


// ELLPACK SpMV kernel
//Michael Garland
__global__ void 
cgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally2FloatComplex alpha, 
    magma_tally2FloatComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2FloatComplex * dx,
    magma_tally2FloatComplex beta, 
    magma_tally2FloatComplex * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally2FloatComplex dot = MAGMA_tally2_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            magma_tally2FloatComplex val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELLPACK SpMV kernel
//Michael Garland
__global__ void 
cgeellmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally2FloatComplex alpha, 
    magma_tally2FloatComplex lambda, 
    magma_tally2FloatComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2FloatComplex * dx,
    magma_tally2FloatComplex beta, 
    int offset,
    int blocksize,
    magma_tally2_index_t * addrows,
    magma_tally2FloatComplex * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally2FloatComplex dot = MAGMA_tally2_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            magma_tally2FloatComplex val = dval [ num_cols_per_row * row + n ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}





/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLPACK.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally2FloatComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2FloatComplex_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally2FloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2FloatComplex
                scalar multiplier

    @param[out]
    dy          magma_tally2FloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_cblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_cgeellmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    magma_tally2FloatComplex alpha,
    magma_tally2FloatComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2FloatComplex_ptr dx,
    magma_tally2FloatComplex beta,
    magma_tally2FloatComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
   cgeellmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally2_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELLPACK.
    It is the shifted version of the ELLPACK SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_tally2FloatComplex
                scalar multiplier
                
    @param[in]
    lambda      magma_tally2FloatComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2FloatComplex_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally2FloatComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2FloatComplex
                scalar multiplier
                
    @param[in]
    offset      magma_tally2_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally2_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally2Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally2FloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_cblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_cgeellmv_shift(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    magma_tally2FloatComplex alpha,
    magma_tally2FloatComplex lambda,
    magma_tally2FloatComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2FloatComplex_ptr dx,
    magma_tally2FloatComplex beta,
    int offset,
    int blocksize,
    magma_tally2Index_ptr addrows,
    magma_tally2FloatComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
   cgeellmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, lambda, dval, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy );


   return MAGMA_tally2_SUCCESS;
}



