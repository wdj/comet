#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally4.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512



__global__ void
magma_tally4_zlobpcg_maxpy_kernel( 
    magma_tally4_int_t num_rows, 
    magma_tally4_int_t num_vecs, 
    magma_tally4DoubleComplex * X, 
    magma_tally4DoubleComplex * Y)
{

    int row = blockIdx.x * blockDim.x + threadIdx.x; // global row index

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ ){ 

            Y[ row + i*num_rows ] += X[ row + i*num_rows ];
        }
    }
}




/**
    Purpose
    -------
    
    This routine computes a axpy for a mxn matrix:
        
        Y = X + Y
        
    It replaces:
            magma_tally4_zaxpy(m*n, c_one, Y, 1, X, 1);


        / x1[0] x2[0] x3[0] \
        | x1[1] x2[1] x3[1] |
    X = | x1[2] x2[2] x3[2] | = x1[0] x1[1] x1[2] x1[3] x1[4] x2[0] x2[1] .
        | x1[3] x2[3] x3[3] |
        \ x1[4] x2[4] x3[4] /
    
    Arguments
    ---------

    @param[in]
    num_rows    magma_tally4_int_t
                number of rows

    @param[in]
    num_vecs    magma_tally4_int_t
                number of vectors

    @param[in]
    X           magma_tally4DoubleComplex_ptr 
                input vector X

    @param[in/out]
    Y           magma_tally4DoubleComplex_ptr 
                input/output vector Y

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zgegpuk
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zlobpcg_maxpy(
    magma_tally4_int_t num_rows,
    magma_tally4_int_t num_vecs, 
    magma_tally4DoubleComplex_ptr X,
    magma_tally4DoubleComplex_ptr Y,
    magma_tally4_queue_t queue )
{
    // every thread handles one row

    magma_tally4_int_t block_size = BLOCK_SIZE;
     magma_tally4_int_t threads = BLOCK_SIZE;
    dim3 block( block_size );
    dim3 grid( magma_tally4_ceildiv( num_rows, block_size ) );

    magma_tally4_zlobpcg_maxpy_kernel<<< grid, threads, 0, queue >>>
                                ( num_rows, num_vecs, X, Y );


    return MAGMA_tally4_SUCCESS;
}



