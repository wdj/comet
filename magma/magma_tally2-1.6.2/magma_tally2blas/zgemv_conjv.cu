#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Stan Tomov

       @precisions normal z -> s d c
*/
#include "common_magma_tally2.h"
#include "commonblas_z.h"

#define PRECISION_z

#define num_threads 256


__global__ void
zgemv_conjv_kernel(
    int m, int n, magma_tally2DoubleComplex alpha,
    const magma_tally2DoubleComplex * __restrict__ A, int lda,
    const magma_tally2DoubleComplex * __restrict__ x, int incx, magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex *       __restrict__ y, int incy)
{
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;

    if ( ind < m ) {
        magma_tally2DoubleComplex res = MAGMA_tally2_Z_ZERO;
        
        #pragma unroll
        for( int i=0; i < n; i ++ ) {
            res += A[0] * MAGMA_tally2_Z_CNJG(x[0]);
            A += lda;
            x += incx;
        }
        
        y[ind*incy] = alpha * res + beta * y[ind*incy];
    }
}


/**
    Purpose
    -------
    ZGEMV_CONJV performs the matrix-vector operation
    
        y := alpha*A*conj(x)    + beta*y, 
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX_16 array of dimension n

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension m

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_tally2_zblas2
    ********************************************************************/
extern "C" void
magma_tally2blas_zgemv_conjv(
    magma_tally2_int_t m, magma_tally2_int_t n, magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex_const_ptr dA, magma_tally2_int_t ldda,
    magma_tally2DoubleComplex_const_ptr dx, magma_tally2_int_t incx,
    magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex_ptr dy, magma_tally2_int_t incy)
{
    magma_tally2_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < m )
        info = -5;
    else if ( incx == 0 )
        info = -7;
    else if ( incy == 0 )
        info = -10;
    
    if (info != 0) {
        magma_tally2_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_tally2_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    zgemv_conjv_kernel<<< grid, threads, 0, magma_tally2_stream >>>
            (m, n, alpha, dA, ldda, dx, incx, beta, dy, incy);

}

#undef num_threads
