#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally3.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    magma_tally3DoubleComplex alpha, 
    magma_tally3DoubleComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3DoubleComplex * dx,
    magma_tally3DoubleComplex beta, 
    magma_tally3DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally3DoubleComplex dot = MAGMA_tally3_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    magma_tally3DoubleComplex alpha, 
    magma_tally3DoubleComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3DoubleComplex * dx,
    magma_tally3DoubleComplex beta, 
    magma_tally3DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally3DoubleComplex dot = MAGMA_tally3_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    magma_tally3DoubleComplex alpha, 
    magma_tally3DoubleComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3DoubleComplex * dx,
    magma_tally3DoubleComplex beta, 
    magma_tally3DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally3DoubleComplex dot = MAGMA_tally3_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_tally3_trans_t
                transposition parameter for A
    @param[in]
    m           magma_tally3_int_t
                number of rows 

    @param[in]
    n           magma_tally3_int_t
                number of columns

    @param[in]
    nnz_per_row magma_tally3_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       magma_tally3DoubleComplex
                scalar alpha

    @param[in]
    dval        magma_tally3DoubleComplex_ptr
                val array

    @param[in]
    dcolind     magma_tally3Index_ptr
                col indices  

    @param[in]
    drowlength  magma_tally3Index_ptr
                number of elements in each row

    @param[in]
    dx          magma_tally3DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally3DoubleComplex
                scalar beta

    @param[out]
    dy          magma_tally3DoubleComplex_ptr
                output vector y

    @param[in]
    blocksize   magma_tally3_int_t
                threads per block

    @param[in]
    alignment   magma_tally3_int_t
                threads assigned to each row

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_zblas
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_zgeellrtmv(
    magma_tally3_trans_t transA,
    magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3_int_t nnz_per_row,
    magma_tally3DoubleComplex alpha,
    magma_tally3DoubleComplex_ptr dval,
    magma_tally3Index_ptr dcolind,
    magma_tally3Index_ptr drowlength,
    magma_tally3DoubleComplex_ptr dx,
    magma_tally3DoubleComplex beta,
    magma_tally3DoubleComplex_ptr dy,
    magma_tally3_int_t alignment,
    magma_tally3_int_t blocksize,
    magma_tally3_queue_t queue )
{
    int num_blocks = magma_tally3_ceildiv( m, blocksize );

    magma_tally3_int_t num_threads = alignment*blocksize;
    magma_tally3_int_t threads = alignment*blocksize;

    int real_row_length = magma_tally3_roundup( nnz_per_row, alignment );

    magma_tally3_int_t arch = magma_tally3_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = (int) sqrt( (double) num_blocks );
    int dimgrid2 = magma_tally3_ceildiv( num_blocks, dimgrid1 );
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( magma_tally3DoubleComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( alignment == 32 ) {
        zgeellrtmv_kernel_32<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        zgeellrtmv_kernel_16<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        zgeellrtmv_kernel_8<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", alignment);
        return MAGMA_tally3_ERR_NOT_SUPPORTED;
    }



   return MAGMA_tally3_SUCCESS;
}


