#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @generated from zgeqr2_kernels.cu normal z -> c, Fri Jan 30 19:00:10 2015
*/



#include "common_magma_tally2.h"
#include "batched_kernel_param.h"



static    magma_tally2FloatComplex neg_one = MAGMA_tally2_C_NEG_ONE;
static    magma_tally2FloatComplex one  = MAGMA_tally2_C_ONE;
static    magma_tally2FloatComplex zero  = MAGMA_tally2_C_ZERO;

__global__ void
cgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magma_tally2FloatComplex **dV_array,    int ldv,
                  magma_tally2FloatComplex **dR_array,    int ldr)
{

    magma_tally2FloatComplex *dV = dV_array[blockIdx.x];
    magma_tally2FloatComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if( tid < n && column < n) 
    {
       for(int i=column; i<n; i++)
       {
          dR[tid + i * ldr]  =  dV[tid + i * ldv];  
       }
    }
}

void cgeqrf_copy_upper_batched(                
                  magma_tally2_int_t n, magma_tally2_int_t nb,
                  magma_tally2FloatComplex **dV_array,    magma_tally2_int_t ldv,
                  magma_tally2FloatComplex **dR_array,    magma_tally2_int_t ldr,
          magma_tally2_int_t batchCount, magma_tally2_queue_t queue)
{
   /* 
        copy some data in dV to dR
   */

      if( nb >= n) return ;

      cgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);

}



extern "C" magma_tally2_int_t
magma_tally2_clarfb_cgemm_batched(
                  hipblasHandle_t myhandle,
                  magma_tally2_int_t m, magma_tally2_int_t n, magma_tally2_int_t k,
                  magma_tally2FloatComplex **dV_array,    magma_tally2_int_t ldv,
                  magma_tally2FloatComplex **dT_array,    magma_tally2_int_t ldt,
                  magma_tally2FloatComplex **dA_array,    magma_tally2_int_t lda,
                  magma_tally2FloatComplex **W_array,     magma_tally2_int_t ldw,
                  magma_tally2FloatComplex **W2_array,    magma_tally2_int_t ldw2,
                  magma_tally2_int_t batchCount, magma_tally2_queue_t queue)

{

    // W is workspace size of W is nb * n 
    // W = V^H * A. V is stored in A(i:m, i:ib)

    
    if( m <=0 || n <= 0 || k <=0 ) return 1;

#if 1  // CUBLAS is faster than MAGMA_tally2BLAS by 17GFLOP/S at size 512 batchCount = 2000
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, m,
                             &one, (const magma_tally2FloatComplex**) dV_array, ldv,
                                    (const magma_tally2FloatComplex**) dA_array, lda,
                             &zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, k,
                             &one, (const magma_tally2FloatComplex**) dT_array, ldt,
                                    (const magma_tally2FloatComplex**) W_array, ldw,
                             &zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
                             &neg_one, (const magma_tally2FloatComplex**) dV_array, ldv,
                                    (const magma_tally2FloatComplex**) W2_array, ldw2,
                             &one,  dA_array, lda, batchCount );

#else 

    magma_tally2blas_cgemm_batched(Magma_tally2ConjTrans, Magma_tally2NoTrans, k, n, m,
                             one, (const magma_tally2FloatComplex**) dV_array, ldv,
                                    (const magma_tally2FloatComplex**) dA_array, lda,
                             zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    magma_tally2blas_cgemm_batched(Magma_tally2ConjTrans, Magma_tally2NoTrans, k, n, k,
                             one, (const magma_tally2FloatComplex**) dT_array, ldt,
                                    (const magma_tally2FloatComplex**) W_array, ldw,
                             zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    magma_tally2blas_cgemm_batched(Magma_tally2NoTrans, Magma_tally2NoTrans, m, n, k,
                             neg_one, (const magma_tally2FloatComplex**) dV_array, ldv,
                                    (const magma_tally2FloatComplex**) W2_array, ldw2,
                             one,  dA_array, lda, batchCount );
          
#endif       
    return 0;

}



