#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlacpy_cnjg.cu normal z -> c, Fri Jan 30 19:00:08 2015

*/
#include "common_magma_tally3.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    magma_tally3FloatComplex *A1;
    magma_tally3FloatComplex *A2;
    int n, lda1, lda2;
} magma_tally3gpu_clacpy_cnjg_params_t;

__global__ void magma_tally3gpu_clacpy_cnjg( magma_tally3gpu_clacpy_cnjg_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = x*params.lda1;
    unsigned int offset2 = x*params.lda2;
    if( x < params.n )
    {
        magma_tally3FloatComplex *A1  = params.A1 + offset1;
        magma_tally3FloatComplex *A2  = params.A2 + offset2;
        *A2 = MAGMA_tally3_C_CNJG(*A1);
    }
}


extern "C" void 
magma_tally3blas_clacpy_cnjg_q(
    magma_tally3_int_t n, magma_tally3FloatComplex *dA1, magma_tally3_int_t lda1, 
    magma_tally3FloatComplex *dA2, magma_tally3_int_t lda2,
    magma_tally3_queue_t queue )
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_tally3gpu_clacpy_cnjg_params_t params = { dA1, dA2, n, lda1, lda2 };
    magma_tally3gpu_clacpy_cnjg<<< blocks, blocksize, 0, queue >>>( params );
}


extern "C" void 
magma_tally3blas_clacpy_cnjg(
    magma_tally3_int_t n, magma_tally3FloatComplex *dA1, magma_tally3_int_t lda1, 
    magma_tally3FloatComplex *dA2, magma_tally3_int_t lda2)
{
    magma_tally3blas_clacpy_cnjg_q( n, dA1, lda1, dA2, lda2, magma_tally3_stream );
}
