#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> s d c
       @author Hartwig Anzt

*/
#include "common_magma_tally4sparse.h"

#define BLOCK_SIZE1 256
#define BLOCK_SIZE2 1


// copy nonzeros into new structure
__global__ void
magma_tally4_zmcsrgpu_kernel1( int num_rows,
                 magma_tally4DoubleComplex *A_val,
                 magma_tally4_index_t *A_rowptr,
                 magma_tally4_index_t *A_colind,
                 magma_tally4DoubleComplex *B_val,
                 magma_tally4_index_t *B_rowptr,
                 magma_tally4_index_t *B_colind ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magma_tally4DoubleComplex zero = MAGMA_tally4_Z_ZERO;
        int start = A_rowptr[ row ];
        int new_location = start;
        int end = A_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
       //         B_val[new_location] = A_val[j];
       //         B_colind[new_location] = A_colind[j];
                new_location++;
            }
        }
        // this is not a correctr rowpointer! this is nn_z in this row!
        B_rowptr[ row ] = new_location-start;
    }
}


// generate a valid rowpointer
__global__ void
magma_tally4_zmcsrgpu_kernel2( int num_rows,
                 magma_tally4_index_t *B_rowptr,
                 magma_tally4_index_t *A_rowptr ){

    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int j, nnz = 0;

    if( idx == 0 ){
    A_rowptr[ 0 ] = nnz;
        for( j=0; j<num_rows; j++ ){
            nnz+=B_rowptr[ j ];
            A_rowptr[ j+1 ] = nnz;
        }
    }
}



// copy new structure into original matrix
__global__ void
magma_tally4_zmcsrgpu_kernel3( int num_rows,
                 magma_tally4DoubleComplex *B_val,
                 magma_tally4_index_t *B_rowptr,
                 magma_tally4_index_t *B_colind,
                 magma_tally4_index_t *B2_rowptr,
                 magma_tally4DoubleComplex *A_val,
                 magma_tally4_index_t *A_rowptr,
                 magma_tally4_index_t *A_colind
                                            ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j, new_location;
    
    if(row<num_rows){
    new_location = A_rowptr[ row ];
        int start = B2_rowptr[ row ];
        int end = B2_rowptr[ row+1 ];
        magma_tally4DoubleComplex zero = MAGMA_tally4_Z_ZERO;
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
                B_val[new_location] = A_val[j];
                B_colind[new_location] = A_colind[j];
                new_location++;
            }
               // A_val[ j ] = B_val[ j ];
               // A_colind[ j ] = B_colind[ j ];
        }
    }
}


/**
    Purpose
    -------

    Removes zeros in a CSR matrix. This is a GPU implementation of the
    CSR compressor.

    Arguments
    ---------

    @param
    A           magma_tally4_z_matrix*
                input/output matrix
    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zaux
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zmcsrcompressor_gpu(
    magma_tally4_z_matrix *A,
    magma_tally4_queue_t queue )
{
    magma_tally4_int_t info = 0;
    magma_tally4_z_matrix B={Magma_tally4_CSR}, B2={Magma_tally4_CSR};
    magma_tally4_z_matrix dA={Magma_tally4_CSR}, CSRA={Magma_tally4_CSR};
    magma_tally4_index_t *cputmp = NULL;
    
    if ( A->memory_location == Magma_tally4_DEV && A->storage_type == Magma_tally4_CSR ) {

        CHECK( magma_tally4_index_malloc( &B.drow, A->num_rows + 1 ));
        CHECK( magma_tally4_index_malloc( &B2.drow, A->num_rows + 1 ));
        
        magma_tally4_index_copyvector( (A->num_rows+1), A->drow, 1, B2.drow, 1 );

        dim3 grid1( magma_tally4_ceildiv( A->num_rows, BLOCK_SIZE1 ) );

        // copying the nonzeros into B and write in B.drow how many there are
        magma_tally4_zmcsrgpu_kernel1<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, A->dval, A->drow, A->dcol, B.dval, B.drow, B.dcol );

        // correct the row pointer
        dim3 grid2( 1, 1, 1);
        magma_tally4_zmcsrgpu_kernel2<<< grid2, BLOCK_SIZE2, 0, queue >>>
                ( A->num_rows, B.drow, A->drow );
        // access the true number of nonzeros

        CHECK( magma_tally4_index_malloc_cpu( &cputmp, 1 ));

        magma_tally4_index_getvector( 1, A->row+(A->num_rows), 1, cputmp, 1 );
        A->nnz = (magma_tally4_int_t) cputmp[0];

        // reallocate with right size
        CHECK( magma_tally4_zmalloc( &B.dval, A->nnz ));
        CHECK( magma_tally4_index_malloc( &B.dcol, A->nnz ));
        
        // copy correct values back
        magma_tally4_zmcsrgpu_kernel3<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, B.dval, B.drow, B.dcol, B2.drow, A->dval, A->drow, A->dcol );

        magma_tally4_free( A->dcol );
        magma_tally4_free( A->dval );

        A->dcol = B.dcol;
        A->dval = B.dval;


    }
    else {
        magma_tally4_storage_t A_storage = A->storage_type;
        magma_tally4_location_t A_location = A->memory_location;
        CHECK( magma_tally4_zmconvert( *A, &CSRA, A->storage_type, Magma_tally4_CSR, queue ));
        CHECK( magma_tally4_zmtransfer( *A, &dA, A->memory_location, Magma_tally4_DEV, queue ));

        CHECK( magma_tally4_zmcsrcompressor_gpu( &dA, queue ));

        magma_tally4_zmfree( &dA, queue );
        magma_tally4_zmfree( A, queue );
        CHECK( magma_tally4_zmtransfer( dA, &CSRA, Magma_tally4_DEV, A_location, queue ));
        CHECK( magma_tally4_zmconvert( CSRA, A, Magma_tally4_CSR, A_storage, queue ));
        magma_tally4_zmfree( &dA, queue );
        magma_tally4_zmfree( &CSRA, queue );

    }
    
cleanup:
    magma_tally4_zmfree( &dA, queue );
    magma_tally4_zmfree( &CSRA, queue );
    magma_tally4_free( B2.drow );
    magma_tally4_free( B.drow );
    return info;
}


