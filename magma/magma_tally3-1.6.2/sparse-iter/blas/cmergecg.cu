#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zmergecg.cu normal z -> c, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magma_tally3sparse.h"

#define BLOCK_SIZE 512

#define PRECISION_c


// These routines merge multiple kernels from cmergecg into one
// for a description see 
// "Reformulated Conjugate Gradient for the Energy-Aware 
// Solution of Linear Systems on GPUs (ICPP '13)

// accelerated reduction for one vector
__global__ void
magma_tally3_ccgreduce_kernel_spmv1( 
    int Gs,
    int n, 
    magma_tally3FloatComplex * vtmp,
    magma_tally3FloatComplex * vtmp2 )
{

    extern __shared__ magma_tally3FloatComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_tally3_C_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_tally3_C_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using CSR and the first step of the reduction
__global__ void
magma_tally3_ccgmerge_spmvcsr_kernel(  
    int n,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * drowptr, 
    magma_tally3_index_t * dcolind,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp )
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    temp[ Idx ] = MAGMA_tally3_C_MAKE( 0.0, 0.0);

    if( i<n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_ZERO;
        int start = drowptr[ i ];
        int end = drowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * d[ dcolind[j] ];
        z[ i ] =  dot;
        temp[ Idx ] =  d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using ELL and the first step of the reduction
__global__ void
magma_tally3_ccgmerge_spmvell_kernel(  
    int n,
    int num_cols_per_row,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp )
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_tally3_C_MAKE( 0.0, 0.0);

    if(i < n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row ; k ++){
            int col = dcolind [ n * k + i ];
            magma_tally3FloatComplex val = dval [ n * k + i ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using ELLPACK and the first step of the reduction
__global__ void
magma_tally3_ccgmerge_spmvellpack_kernel(  
    int n,
    int num_cols_per_row,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp )
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = MAGMA_tally3_C_MAKE( 0.0, 0.0);

    if(i < n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        for ( int k = 0; k < num_cols_per_row ; k ++){
            int col = dcolind [ num_cols_per_row * i + k ];
            magma_tally3FloatComplex val = dval [ num_cols_per_row * i + k ];
            if( val != 0)
                dot += val * d[ col ];
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }

    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_tally3_ccgmerge_spmvellpackrt_kernel_8(  
    int n,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp,
    magma_tally3_int_t T, 
    magma_tally3_int_t alignment  )
{

int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3FloatComplex shared[];

    if(i < n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3FloatComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3FloatComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }

        }
    }

}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_tally3_ccgmerge_spmvellpackrt_kernel_16(  
    int n,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp,
    magma_tally3_int_t T, 
    magma_tally3_int_t alignment  )
{

int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3FloatComplex shared[];

    if(i < n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3FloatComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3FloatComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }

        }
    }

}

// computes the SpMV using ELLRT 8 threads per row
__global__ void
magma_tally3_ccgmerge_spmvellpackrt_kernel_32(  
    int n,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowlength,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp,
    magma_tally3_int_t T, 
    magma_tally3_int_t alignment  )
{

int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally3FloatComplex shared[];

    if(i < n ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int max_ = magma_tally3_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally3FloatComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally3FloatComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * d[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                z[i] = (shared[idb]+shared[idb+1]);
            }

        }
    }

}





// additional kernel necessary to compute first reduction step
__global__ void
magma_tally3_ccgmerge_spmvellpackrt_kernel2(  
    int n,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * vtmp2 )
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    

    temp[ Idx ] = ( i < n ) ? z[i]*d[i] : MAGMA_tally3_C_MAKE(0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}



// computes the SpMV using SELLC
__global__ void
magma_tally3_ccgmerge_spmvsellc_kernel(   
    int num_rows, 
    int blocksize,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowptr,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * vtmp)
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int offset = drowptr[ blockIdx.x ];
    int border = (drowptr[ blockIdx.x+1 ]-offset)/blocksize;

 temp[ Idx ] = MAGMA_tally3_C_MAKE( 0.0, 0.0);


    if(i < num_rows ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n ++){
            int col = dcolind [offset+ blocksize * n + Idx ];
            magma_tally3FloatComplex val = dval[offset+ blocksize * n + Idx];
            if( val != 0){
                  dot=dot+val*d[col];
            }
        }
        z[ i ] =  dot;
        temp[ Idx ] = d[ i ] * dot;
    }
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_tally3_ccgmerge_spmvsellpt_kernel_8( 
    int num_rows, 
    int blocksize,
    int T,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowptr,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y ;     // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ magma_tally3FloatComplex shared[];

    if(row < num_rows ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_ ; k++ ){
            magma_tally3FloatComplex val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 4 ){
            shared[ldx]+=shared[ldx+blocksize*4];              
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }

        }

    }
}
// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_tally3_ccgmerge_spmvsellpt_kernel_16( 
    int num_rows, 
    int blocksize,
    int T,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowptr,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y ;     // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ magma_tally3FloatComplex shared[];

    if(row < num_rows ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_ ; k++ ){
            magma_tally3FloatComplex val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 8 ){
            shared[ldx]+=shared[ldx+blocksize*8];              
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }

        }

    }
}


// SELLP SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
// SELLC SpMV kernel modified assigning multiple threads to each row - 2D kernel
__global__ void
magma_tally3_ccgmerge_spmvsellpt_kernel_32( 
    int num_rows, 
    int blocksize,
    int T,
    magma_tally3FloatComplex * dval, 
    magma_tally3_index_t * dcolind,
    magma_tally3_index_t * drowptr,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z)
{
   // T threads assigned to each row
    int idx = threadIdx.y ;     // thread in row
    int idy = threadIdx.x;      // local row
    int ldx = idx * blocksize + idy;
    int bdx = blockIdx.y * gridDim.x + blockIdx.x; // global block index
    int row = bdx * blocksize + idy;  // global row index

    extern __shared__ magma_tally3FloatComplex shared[];

    if(row < num_rows ){
        magma_tally3FloatComplex dot = MAGMA_tally3_C_MAKE(0.0, 0.0);
        int offset = drowptr[ bdx ];
        int block = blocksize * T; // total number of threads

        int max_ = (drowptr[ bdx+1 ]-offset)/block;  
            // number of elements each thread handles
        for ( int k = 0; k < max_ ; k++ ){
            magma_tally3FloatComplex val = 
                        dval[ offset + ldx + block*k ];
            int col = 
                    dcolind[ offset + ldx + block*k ];
            dot += val * d[ col ];
        }
        shared[ldx]  = dot;

        __syncthreads();
        if( idx < 16 ){
            shared[ldx]+=shared[ldx+blocksize*16];              
            __syncthreads();
            if( idx < 8 ) shared[ldx]+=shared[ldx+blocksize*8];  
            __syncthreads();
            if( idx < 4 ) shared[ldx]+=shared[ldx+blocksize*4];   
            __syncthreads();
            if( idx < 2 ) shared[ldx]+=shared[ldx+blocksize*2];   
            __syncthreads();
            if( idx == 0 ) {
                z[row] = 
                (shared[ldx]+shared[ldx+blocksize*1]);
            }

        }

    }
}


// kernel to handle scalars
__global__ void // rho = beta/tmp; gamma = beta;
magma_tally3_ccg_rhokernel(  
    magma_tally3FloatComplex * skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magma_tally3FloatComplex tmp = skp[1];
        skp[3] = tmp/skp[4];
        skp[2] = tmp;
    }
}

/**
    Purpose
    -------

    Merges the first SpmV using different formats with the dot product 
    and the computation of rho

    Arguments
    ---------

    @param[in]
    A           magma_tally3_c_matrix
                input matrix 

    @param[in]
    d1          magma_tally3FloatComplex_ptr 
                temporary vector

    @param[in]
    d2          magma_tally3FloatComplex_ptr 
                temporary vector

    @param[in]
    dd          magma_tally3FloatComplex_ptr 
                input vector d

    @param[out]
    dz          magma_tally3FloatComplex_ptr 
                input vector z

    @param[out]
    skp         magma_tally3FloatComplex_ptr 
                array for parameters ( skp[3]=rho )

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_cgegpuk
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_ccgmerge_spmv1(
    magma_tally3_c_matrix A,
    magma_tally3FloatComplex_ptr d1,
    magma_tally3FloatComplex_ptr d2,
    magma_tally3FloatComplex_ptr dd,
    magma_tally3FloatComplex_ptr dz,
    magma_tally3FloatComplex_ptr skp,
    magma_tally3_queue_t queue )
{
    // set queue for old dense routines
    magma_tally3_queue_t orig_queue;
    magma_tally3blasGetKernelStream( &orig_queue );

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_tally3_ceildiv( A.num_rows, local_block_size ) );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( magma_tally3FloatComplex ); 
    magma_tally3FloatComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        

    if ( A.storage_type == Magma_tally3_CSR )
        magma_tally3_ccgmerge_spmvcsr_kernel<<<Gs, Bs, Ms, queue >>>
        ( A.num_rows, A.dval, A.drow, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_tally3_ELLPACKT )
        magma_tally3_ccgmerge_spmvellpack_kernel<<<Gs, Bs, Ms, queue >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_tally3_ELL )
        magma_tally3_ccgmerge_spmvell_kernel<<<Gs, Bs, Ms, queue >>>
        ( A.num_rows, A.max_nnz_row, A.dval, A.dcol, dd, dz, d1 );
    else if ( A.storage_type == Magma_tally3_SELLP ) {
            int num_threadssellp = A.blocksize*A.alignment;
            magma_tally3_int_t arch = magma_tally3_getdevice_arch();
            if ( arch < 200 && num_threadssellp > 256 )
                printf("error: too much shared memory requested.\n");

            dim3 block( A.blocksize, A.alignment, 1);
            int dimgrid1 = sqrt(A.numblocks);
            int dimgrid2 = magma_tally3_ceildiv( A.numblocks, dimgrid1 );

            dim3 gridsellp( dimgrid1, dimgrid2, 1);
            int Mssellp = num_threadssellp * sizeof( magma_tally3FloatComplex );

            if ( A.alignment == 8)
                magma_tally3_ccgmerge_spmvsellpt_kernel_8
                <<< gridsellp, block, Mssellp, queue >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 16)
                magma_tally3_ccgmerge_spmvsellpt_kernel_16
                <<< gridsellp, block, Mssellp, queue >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else if ( A.alignment == 32)
                magma_tally3_ccgmerge_spmvsellpt_kernel_32
                <<< gridsellp, block, Mssellp, queue >>>
                ( A.num_rows, A.blocksize, A.alignment, 
                    A.dval, A.dcol, A.drow, dd, dz);

            else
                printf("error: alignment not supported.\n");

        // in case of using SELLP, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.
        magma_tally3_ccgmerge_spmvellpackrt_kernel2<<<Gs, Bs, Ms, queue >>>
                              ( A.num_rows, dz, dd, d1 );

    }
    else if ( A.storage_type == Magma_tally3_ELLRT ) {
        // in case of using ELLRT, we need a different grid, assigning
        // threads_per_row processors to each row
        // the block size is num_threads
        // fixed values


    int num_blocks = ( (A.num_rows+A.blocksize-1)/A.blocksize);

    int num_threads = A.alignment*A.blocksize;

    int real_row_length = ((int)(A.max_nnz_row+A.alignment-1)/A.alignment)
                            *A.alignment;

    magma_tally3_int_t arch = magma_tally3_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = sqrt(num_blocks);
    int dimgrid2 = magma_tally3_ceildiv( num_blocks, dimgrid1 );
    dim3 gridellrt( dimgrid1, dimgrid2, 1);

    int Mellrt = A.alignment * A.blocksize * sizeof( magma_tally3FloatComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( A.alignment == 32 ) {
        magma_tally3_ccgmerge_spmvellpackrt_kernel_32
                <<< gridellrt, num_threads , Mellrt, queue >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 16 ) {
        magma_tally3_ccgmerge_spmvellpackrt_kernel_16
                <<< gridellrt, num_threads , Mellrt, queue >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else if ( A.alignment == 8 ) {
        magma_tally3_ccgmerge_spmvellpackrt_kernel_8
                <<< gridellrt, num_threads , Mellrt, queue >>>
                 ( A.num_rows, A.dval, A.dcol, A.drow, dd, dz, d1, 
                                                 A.alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", A.alignment);
        return MAGMA_tally3_ERR_NOT_SUPPORTED;
    }
        // in case of using ELLRT, we can't efficiently merge the 
        // dot product and the first reduction loop into the SpMV kernel
        // as the SpMV grid would result in low occupancy.

        magma_tally3_ccgmerge_spmvellpackrt_kernel2<<<Gs, Bs, Ms, queue >>>
                              ( A.num_rows, dz, dd, d1 );
    }

    while( Gs.x > 1 ) {
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_tally3_ccgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                                        ( Gs.x,  A.num_rows, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_tally3_ccopyvector( 1, aux1, 1, skp+4, 1 );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_tally3_ccg_rhokernel<<<Gs2, Bs2, 0>>>( skp );

   magma_tally3blasSetKernelStream( orig_queue );
   return MAGMA_tally3_SUCCESS;
}


/* -------------------------------------------------------------------------- */

// updates x and r and computes the first part of the dot product r*r
__global__ void
magma_tally3_ccgmerge_xrbeta_kernel(  
    int n, 
    magma_tally3FloatComplex * x, 
    magma_tally3FloatComplex * r,
    magma_tally3FloatComplex * d,
    magma_tally3FloatComplex * z,
    magma_tally3FloatComplex * skp,
    magma_tally3FloatComplex * vtmp )
{

    extern __shared__ magma_tally3FloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    magma_tally3FloatComplex rho = skp[3];
    magma_tally3FloatComplex mrho = MAGMA_tally3_C_MAKE( -1.0, 0.0)*rho;

    temp[ Idx ] = MAGMA_tally3_C_MAKE( 0.0, 0.0);

    if( i<n ){
        x[i] += rho * d[i] ;
        r[i] += mrho  * z[i];
        temp[ Idx ] = r[i] * r[i];
    }
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }

}

// kernel to handle scalars
__global__ void //alpha = beta / gamma
magma_tally3_ccg_alphabetakernel(  
    magma_tally3FloatComplex * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magma_tally3FloatComplex tmp1 = skp[1];
        skp[0] =  tmp1/skp[2];
        //printf("beta=%e\n", MAGMA_tally3_C_REAL(tmp1));
    }
}

// update search Krylov vector d
__global__ void
magma_tally3_ccg_d_kernel(  
    int n, 
    magma_tally3FloatComplex * skp,
    magma_tally3FloatComplex * r,
    magma_tally3FloatComplex * d )
{
  
    int i   = blockIdx.x * blockDim.x + threadIdx.x;

    magma_tally3FloatComplex alpha = skp[0];

    if( i<n ){
        d[i] = r[i] + alpha * d[i];
    }

}



/**
    Purpose
    -------

    Merges the update of r and x with the dot product and performs then 
    the update for the Krylov vector d

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magma_tally3FloatComplex_ptr 
                temporary vector

    @param[in]
    d2          magma_tally3FloatComplex_ptr 
                temporary vector

    @param[in/out]
    dx          magma_tally3FloatComplex_ptr
                input vector x

    @param[in/out]
    dr          magma_tally3FloatComplex_ptr 
                input/output vector r

    @param[in]
    dd          magma_tally3FloatComplex_ptr 
                input vector d

    @param[in]
    dz          magma_tally3FloatComplex_ptr 
                input vector z
    @param[in]
    skp         magma_tally3FloatComplex_ptr 
                array for parameters

    @param[in]
    queue       magma_tally3_queue_t
                Queue to execute in.

    @ingroup magma_tally3sparse_csygpuk
    ********************************************************************/

extern "C" magma_tally3_int_t
magma_tally3_ccgmerge_xrbeta(
    int n,
    magma_tally3FloatComplex_ptr d1,
    magma_tally3FloatComplex_ptr d2,
    magma_tally3FloatComplex_ptr dx,
    magma_tally3FloatComplex_ptr dr,
    magma_tally3FloatComplex_ptr dd,
    magma_tally3FloatComplex_ptr dz, 
    magma_tally3FloatComplex_ptr skp,
    magma_tally3_queue_t queue )
{
    // set queue for old dense routines
    magma_tally3_queue_t orig_queue;
    magma_tally3blasGetKernelStream( &orig_queue );

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_tally3_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( magma_tally3FloatComplex ); 
    magma_tally3FloatComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
    magma_tally3_ccgmerge_xrbeta_kernel<<<Gs, Bs, Ms>>>
                                    ( n, dx, dr, dd, dz, skp, d1);  



    while( Gs.x > 1 ) {
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_tally3_ccgreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_tally3_ccopyvector( 1, aux1, 1, skp+1, 1 );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_tally3_ccg_alphabetakernel<<<Gs2, Bs2, 0>>>( skp );

    dim3 Bs3( local_block_size );
    dim3 Gs3( magma_tally3_ceildiv( n, local_block_size ) );
    magma_tally3_ccg_d_kernel<<<Gs3, Bs3, 0>>>( n, skp, dr, dd );  

   magma_tally3blasSetKernelStream( orig_queue );
   return MAGMA_tally3_SUCCESS;
}

/* -------------------------------------------------------------------------- */

