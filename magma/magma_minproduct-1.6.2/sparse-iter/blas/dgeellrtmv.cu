#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgeellrtmv.cu normal z -> d, Sun May  3 11:22:58 2015

*/

#include "common_magma_minproduct.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproduct_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_minproduct_D_MAKE(0.0, 0.0);
        int max_ = magma_minproduct_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproduct_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_minproduct_D_MAKE(0.0, 0.0);
        int max_ = magma_minproduct_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_minproduct_index_t * dcolind,
    magma_minproduct_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_minproduct_D_MAKE(0.0, 0.0);
        int max_ = magma_minproduct_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_minproduct_trans_t
                transposition parameter for A
    @param[in]
    m           magma_minproduct_int_t
                number of rows 

    @param[in]
    n           magma_minproduct_int_t
                number of columns

    @param[in]
    nnz_per_row magma_minproduct_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       double
                scalar alpha

    @param[in]
    dval        magma_minproductDouble_ptr
                val array

    @param[in]
    dcolind     magma_minproductIndex_ptr
                col indices  

    @param[in]
    drowlength  magma_minproductIndex_ptr
                number of elements in each row

    @param[in]
    dx          magma_minproductDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar beta

    @param[out]
    dy          magma_minproductDouble_ptr
                output vector y

    @param[in]
    blocksize   magma_minproduct_int_t
                threads per block

    @param[in]
    alignment   magma_minproduct_int_t
                threads assigned to each row

    @param[in]
    queue       magma_minproduct_queue_t
                Queue to execute in.

    @ingroup magma_minproductsparse_dblas
    ********************************************************************/

extern "C" magma_minproduct_int_t
magma_minproduct_dgeellrtmv(
    magma_minproduct_trans_t transA,
    magma_minproduct_int_t m, magma_minproduct_int_t n,
    magma_minproduct_int_t nnz_per_row,
    double alpha,
    magma_minproductDouble_ptr dval,
    magma_minproductIndex_ptr dcolind,
    magma_minproductIndex_ptr drowlength,
    magma_minproductDouble_ptr dx,
    double beta,
    magma_minproductDouble_ptr dy,
    magma_minproduct_int_t alignment,
    magma_minproduct_int_t blocksize,
    magma_minproduct_queue_t queue )
{
    int num_blocks = magma_minproduct_ceildiv( m, blocksize );

    magma_minproduct_int_t num_threads = alignment*blocksize;
    magma_minproduct_int_t threads = alignment*blocksize;

    int real_row_length = magma_minproduct_roundup( nnz_per_row, alignment );

    magma_minproduct_int_t arch = magma_minproduct_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = (int) sqrt( (double) num_blocks );
    int dimgrid2 = magma_minproduct_ceildiv( num_blocks, dimgrid1 );
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( double );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( alignment == 32 ) {
        dgeellrtmv_kernel_32<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        dgeellrtmv_kernel_16<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        dgeellrtmv_kernel_8<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", alignment);
        return MAGMA_minproduct_ERR_NOT_SUPPORTED;
    }



   return MAGMA_minproduct_SUCCESS;
}


