#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarfbx.cu normal z -> d, Fri Jan 30 19:00:08 2015

*/
#include "common_magma_minproduct.h"
#include "commonblas_d.h"
#include "magma_minproduct_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512




//==============================================================================
extern "C"
__global__ void 
magma_minproduct_dgemv_kernel1(int m, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ c, 
                    double *dwork)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_minproduct_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_minproduct_D_MUL( MAGMA_minproduct_D_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_minproduct_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_minproduct_dgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        DGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
    ----------------------------------------------------------------------------- */
extern "C"
__global__ void
magma_minproduct_dgemv_kernel3(int m, const double * __restrict__ V, int ldv, double *c,
                    double *dwork, double *tau)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    if (i==0)
       c[0] = MAGMA_minproduct_D_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_minproduct_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_minproduct_D_MUL( MAGMA_minproduct_D_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_minproduct_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================
extern "C"
__global__ void
magma_minproduct_dgemv_kernel2(int m, int n, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ x, double *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    double lsum;

    V += j;

    lsum = MAGMA_minproduct_D_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_minproduct_D_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a real block reflector H to a real vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the real k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a real block of
    k elementary reflectors. 
*/
extern "C" void
magma_minproduct_dlarfbx_gpu(
    magma_minproduct_int_t m, magma_minproduct_int_t k,
    magma_minproductDouble_ptr V,  magma_minproduct_int_t ldv,
    magma_minproductDouble_ptr dT, magma_minproduct_int_t ldt,
    magma_minproductDouble_ptr c,
    magma_minproductDouble_ptr dwork)
{
    /* dwork = V**H c     */
    magma_minproduct_dgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_minproduct_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_minproduct_dtrmv_tkernel<<< k, k, 0, magma_minproduct_stream >>>( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_minproduct_dgemv_kernel2<<< blocks3, threads3, 0, magma_minproduct_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
