#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c


       @author Adrien REMY
*/
#include "common_magma_tally4.h"
#include "zgerbt.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_tally4_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_tally4blas_zprbt_mtv_q(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *du, magma_tally4DoubleComplex *db,
    magma_tally4_queue_t queue)
{
    /*

     */
    magma_tally4_int_t threads = block_length;
    magma_tally4_int_t grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_tally4blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n, db, 0);
    magma_tally4blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n+n/2, db, n/2);

    threads = block_length;
    grid = n/(2*block_length) + ((n%(2*block_length))!=0);
    magma_tally4blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n, du, 0, db, 0);
}

/**
    @see magma_tally4blas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_tally4blas_zprbt_mtv(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *du, magma_tally4DoubleComplex *db)
{
    magma_tally4blas_zprbt_mtv_q(n, du, db, magma_tally4_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    ZPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_tally4_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_tally4blas_zprbt_mv_q(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *dv, magma_tally4DoubleComplex *db,
    magma_tally4_queue_t queue)
{

    magma_tally4_int_t threads = block_length;
    magma_tally4_int_t grid = n/(2*block_length) + ((n%(2*block_length))!=0);

    magma_tally4blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n, dv, 0, db, 0);


    threads = block_length;
    grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_tally4blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n, db, 0);
    magma_tally4blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n+n/2, db, n/2);
}

/**
    @see magma_tally4blas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_tally4blas_zprbt_mv(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *dv, magma_tally4DoubleComplex *db)
{
    magma_tally4blas_zprbt_mv_q(n, dv, db, magma_tally4_stream);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_tally4_queue_t
            Queue to execute in.

    ********************************************************************/
extern "C" void 
magma_tally4blas_zprbt_q(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *dA, magma_tally4_int_t ldda, 
    magma_tally4DoubleComplex *du, magma_tally4DoubleComplex *dv,
    magma_tally4_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid(n/(4*block_height) + ((n%(4*block_height))!=0), 
            n/(4*block_width)  + ((n%(4*block_width))!=0));

    magma_tally4blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,            0, ldda, du,   0, dv,   0);
    magma_tally4blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,     ldda*n/2, ldda, du,   0, dv, n/2);
    magma_tally4blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,          n/2, ldda, du, n/2, dv,   0);
    magma_tally4blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2(n/(2*block_height) + ((n%(2*block_height))!=0), 
            n/(2*block_width)  + ((n%(2*block_width))!=0));
    magma_tally4blas_zelementary_multiplication_kernel<<< grid2, threads2, 0, queue >>>(n, dA, 0, ldda, du, -ldda, dv, -ldda);
}


/**
    @see magma_tally4blas_zprbt_q
    ********************************************************************/
extern "C" void 
magma_tally4blas_zprbt(
    magma_tally4_int_t n, 
    magma_tally4DoubleComplex *dA, magma_tally4_int_t ldda, 
    magma_tally4DoubleComplex *du, magma_tally4DoubleComplex *dv)
{
    magma_tally4blas_zprbt_q(n, dA, ldda, du, dv, magma_tally4_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
zaxpycp2_kernel(
    int m, magma_tally4DoubleComplex *r, magma_tally4DoubleComplex *x,
    const magma_tally4DoubleComplex *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_tally4_Z_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magma_tally4blas_zaxpycp2_q(
    magma_tally4_int_t m, magma_tally4DoubleComplex *r, magma_tally4DoubleComplex *x,
    const magma_tally4DoubleComplex *b,
    magma_tally4_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    zaxpycp2_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b );
}


extern "C" void
magma_tally4blas_zaxpycp2(
    magma_tally4_int_t m, magma_tally4DoubleComplex *r, magma_tally4DoubleComplex *x,
    const magma_tally4DoubleComplex *b)
{
    magma_tally4blas_zaxpycp2_q( m, r, x, b, magma_tally4_stream );
}
