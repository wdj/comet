#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @generated from zgeqr2_kernels.cu normal z -> c, Fri Jan 30 19:00:10 2015
*/



#include "common_magma_tally4.h"
#include "batched_kernel_param.h"



static    magma_tally4FloatComplex neg_one = MAGMA_tally4_C_NEG_ONE;
static    magma_tally4FloatComplex one  = MAGMA_tally4_C_ONE;
static    magma_tally4FloatComplex zero  = MAGMA_tally4_C_ZERO;

__global__ void
cgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magma_tally4FloatComplex **dV_array,    int ldv,
                  magma_tally4FloatComplex **dR_array,    int ldr)
{

    magma_tally4FloatComplex *dV = dV_array[blockIdx.x];
    magma_tally4FloatComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if( tid < n && column < n) 
    {
       for(int i=column; i<n; i++)
       {
          dR[tid + i * ldr]  =  dV[tid + i * ldv];  
       }
    }
}

void cgeqrf_copy_upper_batched(                
                  magma_tally4_int_t n, magma_tally4_int_t nb,
                  magma_tally4FloatComplex **dV_array,    magma_tally4_int_t ldv,
                  magma_tally4FloatComplex **dR_array,    magma_tally4_int_t ldr,
          magma_tally4_int_t batchCount, magma_tally4_queue_t queue)
{
   /* 
        copy some data in dV to dR
   */

      if( nb >= n) return ;

      cgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);

}



extern "C" magma_tally4_int_t
magma_tally4_clarfb_cgemm_batched(
                  hipblasHandle_t myhandle,
                  magma_tally4_int_t m, magma_tally4_int_t n, magma_tally4_int_t k,
                  magma_tally4FloatComplex **dV_array,    magma_tally4_int_t ldv,
                  magma_tally4FloatComplex **dT_array,    magma_tally4_int_t ldt,
                  magma_tally4FloatComplex **dA_array,    magma_tally4_int_t lda,
                  magma_tally4FloatComplex **W_array,     magma_tally4_int_t ldw,
                  magma_tally4FloatComplex **W2_array,    magma_tally4_int_t ldw2,
                  magma_tally4_int_t batchCount, magma_tally4_queue_t queue)

{

    // W is workspace size of W is nb * n 
    // W = V^H * A. V is stored in A(i:m, i:ib)

    
    if( m <=0 || n <= 0 || k <=0 ) return 1;

#if 1  // CUBLAS is faster than MAGMA_tally4BLAS by 17GFLOP/S at size 512 batchCount = 2000
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, m,
                             &one, (const magma_tally4FloatComplex**) dV_array, ldv,
                                    (const magma_tally4FloatComplex**) dA_array, lda,
                             &zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, k,
                             &one, (const magma_tally4FloatComplex**) dT_array, ldt,
                                    (const magma_tally4FloatComplex**) W_array, ldw,
                             &zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
                             &neg_one, (const magma_tally4FloatComplex**) dV_array, ldv,
                                    (const magma_tally4FloatComplex**) W2_array, ldw2,
                             &one,  dA_array, lda, batchCount );

#else 

    magma_tally4blas_cgemm_batched(Magma_tally4ConjTrans, Magma_tally4NoTrans, k, n, m,
                             one, (const magma_tally4FloatComplex**) dV_array, ldv,
                                    (const magma_tally4FloatComplex**) dA_array, lda,
                             zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    magma_tally4blas_cgemm_batched(Magma_tally4ConjTrans, Magma_tally4NoTrans, k, n, k,
                             one, (const magma_tally4FloatComplex**) dT_array, ldt,
                                    (const magma_tally4FloatComplex**) W_array, ldw,
                             zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    magma_tally4blas_cgemm_batched(Magma_tally4NoTrans, Magma_tally4NoTrans, m, n, k,
                             neg_one, (const magma_tally4FloatComplex**) dV_array, ldv,
                                    (const magma_tally4FloatComplex**) W2_array, ldw2,
                             one,  dA_array, lda, batchCount );
          
#endif       
    return 0;

}



