#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
__global__ void 
zgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally2DoubleComplex dot = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally2DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
zgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex lambda, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    int offset,
    int blocksize,
    magma_tally2_index_t * addrows,
    magma_tally2DoubleComplex * dy)
{

    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        magma_tally2DoubleComplex dot = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magma_tally2DoubleComplex val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally2DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally2DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally2DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_z
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_zgeelltmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2DoubleComplex_ptr dx,
    magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
    zgeelltmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally2_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magma_tally2DoubleComplex
                scalar multiplier

    @param[in]
    lambda      magma_tally2DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2DoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELL

    @param[in]
    dx          magma_tally2DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2DoubleComplex
                scalar multiplier
                
    @param[in]
    offset      magma_tally2_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_tally2_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magma_tally2Index_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magma_tally2DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_zblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_zgeelltmv_shift(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex lambda,
    magma_tally2DoubleComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2DoubleComplex_ptr dx,
    magma_tally2DoubleComplex beta,
    int offset,
    int blocksize,
    magma_tally2Index_ptr addrows,
    magma_tally2DoubleComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
    magma_tally2DoubleComplex tmp_shift;
    //magma_tally2_zsetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    zgeelltmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


   return MAGMA_tally2_SUCCESS;
}



