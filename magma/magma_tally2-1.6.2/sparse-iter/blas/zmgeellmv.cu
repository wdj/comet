#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/
#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


__global__ void 
zmgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_vecs,
    int num_cols_per_row,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;

    extern __shared__ magma_tally2DoubleComplex dot[];

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
                dot[ threadIdx.x + i*blockDim.x ] = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            magma_tally2DoubleComplex val = dval [ num_cols_per_row * row + n ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++)
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++)
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i * num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELLPACK. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                              
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_tally2_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magma_tally2DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2DoubleComplex_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magma_tally2DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally2DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_zblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_zmgeellmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t num_vecs,
    magma_tally2_int_t nnz_per_row,
    magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2DoubleComplex_ptr dx,
    magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ) );
    magma_tally2_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                            * sizeof( magma_tally2DoubleComplex ); // num_vecs vectors 
    zmgeellmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_tally2_SUCCESS;
}



