#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Mark Gates

       @precisions normal z -> s d c

*/
#include "common_magma_tally2.h"

#define NB 64


/* Vector is divided into ceil(n/nb) blocks.
   Each thread swaps one element, x[tid] <---> y[tid].
*/
__global__ void zswap_kernel(
    int n,
    magma_tally2DoubleComplex *x, int incx,
    magma_tally2DoubleComplex *y, int incy )
{
    magma_tally2DoubleComplex tmp;
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    if ( ind < n ) {
        x += ind*incx;
        y += ind*incy;
        tmp = *x;
        *x  = *y;
        *y  = tmp;
    }
}


/**
    Purpose:
    =============
    Swap vector x and y; \f$ x <-> y \f$.

    @param[in]
    n       Number of elements in vector x and y. n >= 0.

    @param[in,out]
    dx      COMPLEX_16 array on GPU device.
            The n element vector x of dimension (1 + (n-1)*incx).

    @param[in]
    incx    Stride between consecutive elements of dx. incx != 0.

    @param[in,out]
    dy      COMPLEX_16 array on GPU device.
            The n element vector y of dimension (1 + (n-1)*incy).

    @param[in]
    incy    Stride between consecutive elements of dy. incy != 0.

    @ingroup magma_tally2_zblas1
    ********************************************************************/
extern "C" void 
magma_tally2blas_zswap_q(
    magma_tally2_int_t n,
    magma_tally2DoubleComplex_ptr dx, magma_tally2_int_t incx, 
    magma_tally2DoubleComplex_ptr dy, magma_tally2_int_t incy,
    magma_tally2_queue_t queue )
{
    dim3 grid( (n+NB-1) / NB );
    dim3 threads( NB );
    zswap_kernel<<< grid, threads, 0, queue >>>( n, dx, incx, dy, incy );
}


/**
    @see magma_tally2blas_zswap_q
    @ingroup magma_tally2_zblas1
    ********************************************************************/
extern "C" void 
magma_tally2blas_zswap(
    magma_tally2_int_t n,
    magma_tally2DoubleComplex_ptr dx, magma_tally2_int_t incx, 
    magma_tally2DoubleComplex_ptr dy, magma_tally2_int_t incy)
{
    magma_tally2blas_zswap_q( n, dx, incx, dy, incy, magma_tally2_stream );
}
