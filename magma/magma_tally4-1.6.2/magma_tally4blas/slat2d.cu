#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from clat2z.cu mixed zc -> ds, Fri Jan 30 19:00:08 2015
       @author Mark Gates
*/
#include "common_magma_tally4.h"

#define PRECISION_d

#define BLK_X 64
#define BLK_Y 32


/*
    Divides matrix into ceil( n/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.
    Updates only the diagonal and below.
    Blocks that are fully above the diagonal exit immediately.
    
    Code similar to dlag2s and zlaset.
*/
__global__
void slat2d_lower(
    int n,
    const float *SA, int ldsa,
    double      *A,  int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (below diag) */
    bool full = (iby + BLK_Y <= n && (ind >= iby + BLK_Y));
    /* do only rows inside matrix, and blocks not above diag */
    if ( ind < n && ind + BLK_X > iby ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = (double)( SA[j*ldsa] );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n && ind >= iby+j; ++j ) {
                A[j*lda] = (double)( SA[j*ldsa] );
            }
        }
    }
}


/*
    Similar to slat2d_full, but updates only the diagonal and above.
    Blocks that are fully below the diagonal exit immediately.
    
    Code similar to dlag2s and zlaset.
*/
__global__
void slat2d_upper(
    int n,
    const float *SA, int ldsa,
    double      *A,  int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (above diag) */
    bool full = (iby + BLK_Y <= n && (ind + BLK_X <= iby));
    /* do only rows inside matrix, and blocks not below diag */
    if ( ind < n && ind < iby + BLK_Y ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = (double)( SA[j*ldsa] );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                if ( ind <= iby+j ) {
                    A[j*lda] = (double)( SA[j*ldsa] );
                }
            }
        }
    }
}


/**
    Purpose
    -------
    SLAT2D_STREAM converts a single-real matrix, SA,
                        to a double-real matrix, A.

    Note that while it is possible to overflow while converting
    from double to single, it is not possible to overflow when
    converting from single to double.

    Arguments
    ---------
    @param[in]
    uplo    magma_tally4_uplo_t
            Specifies the part of the matrix A to be converted.
      -     = Magma_tally4Upper:      Upper triangular part
      -     = Magma_tally4Lower:      Lower triangular part
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  n >= 0.
    
    @param[in]
    A       DOUBLE PRECISION array, dimension (LDA,n)
            On entry, the n-by-n coefficient matrix A.
    
    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,n).
    
    @param[out]
    SA      SINGLE PRECISION array, dimension (LDSA,n)
            On exit, if INFO=0, the n-by-n coefficient matrix SA;
            if INFO > 0, the content of SA is unspecified.
    
    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,n).
    
    @param[out]
    info    INTEGER
      -     = 0:  successful exit.
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_tally4_queue_t
            Queue to execute in.
    
    @ingroup magma_tally4_daux2
    ********************************************************************/
extern "C" void
magma_tally4blas_slat2d_q(
    magma_tally4_uplo_t uplo, magma_tally4_int_t n,
    magma_tally4Float_const_ptr SA, magma_tally4_int_t ldsa,
    magma_tally4Double_ptr      A,  magma_tally4_int_t lda,
    magma_tally4_int_t *info,
    magma_tally4_queue_t queue )
{
    *info = 0;
    if ( uplo != Magma_tally4Lower && uplo != Magma_tally4Upper )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( lda < max(1,n) )
        *info = -4;
    else if ( ldsa < max(1,n) )
        *info = -6;
    
    if (*info != 0) {
        magma_tally4_xerbla( __func__, -(*info) );
        return; //*info;
    }

    /* quick return */
    if ( n == 0 ) {
        return;
    }
    
    dim3 threads( BLK_X );
    dim3 grid( (n+BLK_X-1)/BLK_X, (n+BLK_Y-1)/BLK_Y );
    
    if (uplo == Magma_tally4Lower)
        slat2d_lower<<< grid, threads, 0, queue >>> (n, SA, ldsa, A, lda);
    else if (uplo == Magma_tally4Upper)                                         
        slat2d_upper<<< grid, threads, 0, queue >>> (n, SA, ldsa, A, lda);
}


/**
    @see magma_tally4blas_slat2d_q
    @ingroup magma_tally4_daux2
    ********************************************************************/
extern "C" void
magma_tally4blas_slat2d(
    magma_tally4_uplo_t uplo, magma_tally4_int_t n,
    magma_tally4Float_const_ptr SA, magma_tally4_int_t ldsa,
    magma_tally4Double_ptr      A,  magma_tally4_int_t lda,
    magma_tally4_int_t *info )
{
    magma_tally4blas_slat2d_q( uplo, n, SA, ldsa, A, lda, info, magma_tally4_stream );
}
