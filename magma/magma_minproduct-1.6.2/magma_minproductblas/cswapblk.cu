#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zswapblk.cu normal z -> c, Fri Jan 30 19:00:09 2015

*/
#include "common_magma_minproduct.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
 *  Blocked version: swap several pairs of lines
 */
typedef struct {
    magma_minproductFloatComplex *A;
    magma_minproductFloatComplex *B;
    int n, ldda, lddb, npivots;
    short ipiv[BLOCK_SIZE];
} magma_minproductgpu_cswapblk_params_t;

__global__ void magma_minproductgpu_cswapblkrm( magma_minproductgpu_cswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        magma_minproductFloatComplex *A = params.A + y - params.ldda;
        magma_minproductFloatComplex *B = params.B + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A += params.ldda;
            if ( params.ipiv[i] == -1 )
                continue;
            magma_minproductFloatComplex  tmp1 = *A;
            magma_minproductFloatComplex *tmp2 = B + params.ipiv[i]*params.lddb;
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
}

__global__ void magma_minproductgpu_cswapblkcm( magma_minproductgpu_cswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.ldda;
    unsigned int offset2 = y*params.lddb;
    if( y < params.n )
    {
        magma_minproductFloatComplex *A = params.A + offset1 - 1;
        magma_minproductFloatComplex *B = params.B + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A++;
            if ( params.ipiv[i] == -1 )
                continue;
            magma_minproductFloatComplex  tmp1 = *A;
            magma_minproductFloatComplex *tmp2 = B + params.ipiv[i];
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
    __syncthreads();
}


/**
    @ingroup magma_minproduct_cblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_cswapblk_q(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductFloatComplex_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductFloatComplex_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset,
    magma_minproduct_queue_t queue )
{
    magma_minproduct_int_t  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_minproduct_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == Magma_minproductColMajor ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_cswapblk_params_t params = { dA+k, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_cswapblkcm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
    else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_cswapblk_params_t params = { dA+k*ldda, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_cswapblkrm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
}


/**
    @see magma_minproductblas_cswapblk_q
    @ingroup magma_minproduct_cblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_cswapblk(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductFloatComplex_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductFloatComplex_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset )
{
    magma_minproductblas_cswapblk_q(
        order, n, dA, ldda, dB, lddb, i1, i2, ipiv, inci, offset, magma_minproduct_stream );
}
