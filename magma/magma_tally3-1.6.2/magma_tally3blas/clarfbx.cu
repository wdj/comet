#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarfbx.cu normal z -> c, Fri Jan 30 19:00:08 2015

*/
#include "common_magma_tally3.h"
#include "commonblas_c.h"
#include "magma_tally3_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512




//==============================================================================
extern "C"
__global__ void 
magma_tally3_cgemv_kernel1(int m, const magma_tally3FloatComplex * __restrict__ V, int ldv, 
                    const magma_tally3FloatComplex * __restrict__ c, 
                    magma_tally3FloatComplex *dwork)
{
    const int i = threadIdx.x;
    const magma_tally3FloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magma_tally3FloatComplex sum[ BLOCK_SIZE ];
    magma_tally3FloatComplex lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally3_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally3_C_MUL( MAGMA_tally3_C_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_tally3_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_tally3_cgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        CGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
    ----------------------------------------------------------------------------- */
extern "C"
__global__ void
magma_tally3_cgemv_kernel3(int m, const magma_tally3FloatComplex * __restrict__ V, int ldv, magma_tally3FloatComplex *c,
                    magma_tally3FloatComplex *dwork, magma_tally3FloatComplex *tau)
{
    const int i = threadIdx.x;
    const magma_tally3FloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magma_tally3FloatComplex sum[ BLOCK_SIZE ];
    magma_tally3FloatComplex lsum;

    if (i==0)
       c[0] = MAGMA_tally3_C_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_tally3_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_tally3_C_MUL( MAGMA_tally3_C_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_tally3_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================
extern "C"
__global__ void
magma_tally3_cgemv_kernel2(int m, int n, const magma_tally3FloatComplex * __restrict__ V, int ldv, 
                    const magma_tally3FloatComplex * __restrict__ x, magma_tally3FloatComplex *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    magma_tally3FloatComplex lsum;

    V += j;

    lsum = MAGMA_tally3_C_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_tally3_C_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a complex block reflector H to a complex vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the complex k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a complex block of
    k elementary reflectors. 
*/
extern "C" void
magma_tally3_clarfbx_gpu(
    magma_tally3_int_t m, magma_tally3_int_t k,
    magma_tally3FloatComplex_ptr V,  magma_tally3_int_t ldv,
    magma_tally3FloatComplex_ptr dT, magma_tally3_int_t ldt,
    magma_tally3FloatComplex_ptr c,
    magma_tally3FloatComplex_ptr dwork)
{
    /* dwork = V**H c     */
    magma_tally3_cgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_tally3_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_tally3_ctrmv_tkernel<<< k, k, 0, magma_tally3_stream >>>( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_tally3_cgemv_kernel2<<< blocks3, threads3, 0, magma_tally3_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
