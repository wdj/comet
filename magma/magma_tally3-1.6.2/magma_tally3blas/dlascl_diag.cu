#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlascl_diag.cu normal z -> d, Fri Jan 30 19:00:09 2015
*/
#include "common_magma_tally3.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
dlascl_diag_lower(int m, int n, magma_tally3Double_const_ptr D, int ldd, 
                                      magma_tally3Double_ptr A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] /= D[j+j*ldd];
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
dlascl_diag_upper(int m, int n, magma_tally3Double_const_ptr D, int ldd, 
                                      magma_tally3Double_ptr A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] /= D[ind+ind*ldd];
    }
}


/**
    Purpose
    -------
    DLASCL2 scales the M by N real matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_tally3_type_t
            TYPE indices the storage type of the input matrix A.
            = Magma_tally3Full:   full matrix.
            = Magma_tally3Lower:  lower triangular matrix.
            = Magma_tally3Upper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA_tally3 does not currently support.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in]
    dD      DOUBLE PRECISION vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    \param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @ingroup magma_tally3_daux2
    ********************************************************************/
extern "C" void
magma_tally3blas_dlascl_diag_q(
    magma_tally3_type_t type, magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3Double_const_ptr dD, magma_tally3_int_t lddd, 
          magma_tally3Double_ptr dA, magma_tally3_int_t ldda, 
    magma_tally3_int_t *info, magma_tally3_queue_t queue )
{
    *info = 0;
    if ( type != Magma_tally3Lower && type != Magma_tally3Upper && type != Magma_tally3Full )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -5;
    
    if (*info != 0) {
        magma_tally3_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    if (type == Magma_tally3Lower) {
        dlascl_diag_lower <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
    else if (type == Magma_tally3Upper) {
        dlascl_diag_upper <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
}


/**
    @see magma_tally3blas_dlascl2_q
    @ingroup magma_tally3_daux2
    ********************************************************************/
extern "C" void
magma_tally3blas_dlascl_diag(
    magma_tally3_type_t type, magma_tally3_int_t m, magma_tally3_int_t n,
    magma_tally3Double_const_ptr dD, magma_tally3_int_t lddd, 
          magma_tally3Double_ptr dA, magma_tally3_int_t ldda, 
    magma_tally3_int_t *info )
{
    magma_tally3blas_dlascl_diag_q( type, m, n, dD, lddd, dA, ldda, info, magma_tally3_stream );
}
