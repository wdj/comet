#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c

*/
#include "common_magma_tally3.h"


/*********************************************************/
/*
 *  Swap diagonal blocks of two matrices.
 *  Each thread block swaps one diagonal block.
 *  Each thread iterates across one row of the block.
 */

__global__ void 
zswapdblk_kernel( int nb,
                  magma_tally3DoubleComplex *dA, int ldda, int inca,
                  magma_tally3DoubleComplex *dB, int lddb, int incb )
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    dA += tx + bx * nb * (ldda + inca);
    dB += tx + bx * nb * (lddb + incb);

    magma_tally3DoubleComplex tmp;

    #pragma unroll
    for( int i = 0; i < nb; i++ ){
        tmp        = dA[i*ldda];
        dA[i*ldda] = dB[i*lddb];
        dB[i*lddb] = tmp;
    }
}


/**
    Purpose
    -------
    zswapdblk swaps diagonal blocks of size nb x nb between matrices
    dA and dB on the GPU. It swaps nblocks = n/nb blocks.
    For i = 1 .. nblocks, submatrices
    dA( i*nb*inca, i*nb ) and
    dB( i*nb*incb, i*nb ) are swapped.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns of the matrices dA and dB.  N >= 0.

    @param[in]
    nb      INTEGER
            The size of diagonal blocks.
            NB > 0 and NB <= maximum threads per CUDA block (512 or 1024).

    @param[in,out]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            The matrix dA.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.
            LDDA >= (nblocks - 1)*nb*inca + nb.

    @param[in]
    inca    INTEGER
            The row increment between diagonal blocks of dA. inca >= 0. For example,
            inca = 1 means blocks are stored on the diagonal at dA(i*nb, i*nb),
            inca = 0 means blocks are stored side-by-side    at dA(0,    i*nb).

    @param[in,out]
    dB      COMPLEX_16 array, dimension (LDDB,N)
            The matrix dB.

    @param[in]
    lddb    INTEGER
            The leading dimension of the array db.
            LDDB >= (nblocks - 1)*nb*incb + nb.

    @param[in]
    incb    INTEGER
            The row increment between diagonal blocks of dB. incb >= 0. See inca.
    
    @param[in]
    queue   magma_tally3_queue_t
            Queue to execute in.

    @ingroup magma_tally3_zaux2
    ********************************************************************/
extern "C" void 
magma_tally3blas_zswapdblk_q(
    magma_tally3_int_t n, magma_tally3_int_t nb,
    magma_tally3DoubleComplex_ptr dA, magma_tally3_int_t ldda, magma_tally3_int_t inca,
    magma_tally3DoubleComplex_ptr dB, magma_tally3_int_t lddb, magma_tally3_int_t incb,
    magma_tally3_queue_t queue )
{
    magma_tally3_int_t nblocks = n / nb;
    
    magma_tally3_int_t info = 0;
    if (n < 0) {
        info = -1;
    } else if (nb < 1 || nb > 1024) {
        info = -2;
    } else if (ldda < (nblocks-1)*nb*inca + nb) {
        info = -4;
    } else if (inca < 0) {
        info = -5;
    } else if (lddb < (nblocks-1)*nb*incb + nb) {
        info = -7;
    } else if (incb < 0) {
        info = -8;
    }

    if (info != 0) {
        magma_tally3_xerbla( __func__, -(info) );
        return;  //info;
    }

    if ( nblocks > 0 ) {
        zswapdblk_kernel<<< nblocks, nb, 0, queue >>>
            ( nb, dA, ldda, inca,
                  dB, lddb, incb );
    }
}


/**
    @see magma_tally3blas_zswapdblk_q
    @ingroup magma_tally3_zaux2
    ********************************************************************/
extern "C" void 
magma_tally3blas_zswapdblk(
    magma_tally3_int_t n, magma_tally3_int_t nb,
    magma_tally3DoubleComplex_ptr dA, magma_tally3_int_t ldda, magma_tally3_int_t inca,
    magma_tally3DoubleComplex_ptr dB, magma_tally3_int_t lddb, magma_tally3_int_t incb )
{
    magma_tally3blas_zswapdblk_q( n, nb, dA, ldda, inca, dB, lddb, incb, magma_tally3_stream );
}
