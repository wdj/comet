#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zswapblk.cu normal z -> s, Fri Jan 30 19:00:09 2015

*/
#include "common_magma_minproduct.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
 *  Blocked version: swap several pairs of lines
 */
typedef struct {
    float *A;
    float *B;
    int n, ldda, lddb, npivots;
    short ipiv[BLOCK_SIZE];
} magma_minproductgpu_sswapblk_params_t;

__global__ void magma_minproductgpu_sswapblkrm( magma_minproductgpu_sswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        float *A = params.A + y - params.ldda;
        float *B = params.B + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A += params.ldda;
            if ( params.ipiv[i] == -1 )
                continue;
            float  tmp1 = *A;
            float *tmp2 = B + params.ipiv[i]*params.lddb;
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
}

__global__ void magma_minproductgpu_sswapblkcm( magma_minproductgpu_sswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.ldda;
    unsigned int offset2 = y*params.lddb;
    if( y < params.n )
    {
        float *A = params.A + offset1 - 1;
        float *B = params.B + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A++;
            if ( params.ipiv[i] == -1 )
                continue;
            float  tmp1 = *A;
            float *tmp2 = B + params.ipiv[i];
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
    __syncthreads();
}


/**
    @ingroup magma_minproduct_sblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_sswapblk_q(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductFloat_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductFloat_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset,
    magma_minproduct_queue_t queue )
{
    magma_minproduct_int_t  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_minproduct_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == Magma_minproductColMajor ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_sswapblk_params_t params = { dA+k, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_sswapblkcm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
    else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_minproduct_int_t sb = min(BLOCK_SIZE, i2-k);
            magma_minproductgpu_sswapblk_params_t params = { dA+k*ldda, dB, n, ldda, lddb, sb };
            for( magma_minproduct_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magma_minproductgpu_sswapblkrm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
}


/**
    @see magma_minproductblas_sswapblk_q
    @ingroup magma_minproduct_sblas2
    ********************************************************************/
extern "C" void 
magma_minproductblas_sswapblk(
    magma_minproduct_order_t order, magma_minproduct_int_t n, 
    magma_minproductFloat_ptr dA, magma_minproduct_int_t ldda,
    magma_minproductFloat_ptr dB, magma_minproduct_int_t lddb,
    magma_minproduct_int_t i1, magma_minproduct_int_t i2,
    const magma_minproduct_int_t *ipiv, magma_minproduct_int_t inci, magma_minproduct_int_t offset )
{
    magma_minproductblas_sswapblk_q(
        order, n, dA, ldda, dB, lddb, i1, i2, ipiv, inci, offset, magma_minproduct_stream );
}
