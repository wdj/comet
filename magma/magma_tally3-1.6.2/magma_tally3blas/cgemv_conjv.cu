#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally3 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Stan Tomov

       @generated from zgemv_conjv.cu normal z -> c, Fri Jan 30 19:00:08 2015
*/
#include "common_magma_tally3.h"
#include "commonblas_c.h"

#define PRECISION_c

#define num_threads 256


__global__ void
cgemv_conjv_kernel(
    int m, int n, magma_tally3FloatComplex alpha,
    const magma_tally3FloatComplex * __restrict__ A, int lda,
    const magma_tally3FloatComplex * __restrict__ x, int incx, magma_tally3FloatComplex beta,
    magma_tally3FloatComplex *       __restrict__ y, int incy)
{
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;

    if ( ind < m ) {
        magma_tally3FloatComplex res = MAGMA_tally3_C_ZERO;
        
        #pragma unroll
        for( int i=0; i < n; i ++ ) {
            res += A[0] * MAGMA_tally3_C_CNJG(x[0]);
            A += lda;
            x += incx;
        }
        
        y[ind*incy] = alpha * res + beta * y[ind*incy];
    }
}


/**
    Purpose
    -------
    CGEMV_CONJV performs the matrix-vector operation
    
        y := alpha*A*conj(x)    + beta*y, 
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX array of dimension n

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      REAL array of dimension m

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_tally3_cblas2
    ********************************************************************/
extern "C" void
magma_tally3blas_cgemv_conjv(
    magma_tally3_int_t m, magma_tally3_int_t n, magma_tally3FloatComplex alpha,
    magma_tally3FloatComplex_const_ptr dA, magma_tally3_int_t ldda,
    magma_tally3FloatComplex_const_ptr dx, magma_tally3_int_t incx,
    magma_tally3FloatComplex beta,
    magma_tally3FloatComplex_ptr dy, magma_tally3_int_t incy)
{
    magma_tally3_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < m )
        info = -5;
    else if ( incx == 0 )
        info = -7;
    else if ( incy == 0 )
        info = -10;
    
    if (info != 0) {
        magma_tally3_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_tally3_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    cgemv_conjv_kernel<<< grid, threads, 0, magma_tally3_stream >>>
            (m, n, alpha, dA, ldda, dx, incx, beta, dy, incy);

}

#undef num_threads
