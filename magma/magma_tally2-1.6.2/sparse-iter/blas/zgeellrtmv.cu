#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally2.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2_index_t * drowlength,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally2DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally2DoubleComplex dot = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally2_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally2DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally2DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2_index_t * drowlength,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally2DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally2DoubleComplex dot = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally2_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally2DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally2DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * dcolind,
    magma_tally2_index_t * drowlength,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally2DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally2DoubleComplex dot = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally2_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally2DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally2DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A
    @param[in]
    m           magma_tally2_int_t
                number of rows 

    @param[in]
    n           magma_tally2_int_t
                number of columns

    @param[in]
    nnz_per_row magma_tally2_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       magma_tally2DoubleComplex
                scalar alpha

    @param[in]
    dval        magma_tally2DoubleComplex_ptr
                val array

    @param[in]
    dcolind     magma_tally2Index_ptr
                col indices  

    @param[in]
    drowlength  magma_tally2Index_ptr
                number of elements in each row

    @param[in]
    dx          magma_tally2DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2DoubleComplex
                scalar beta

    @param[out]
    dy          magma_tally2DoubleComplex_ptr
                output vector y

    @param[in]
    blocksize   magma_tally2_int_t
                threads per block

    @param[in]
    alignment   magma_tally2_int_t
                threads assigned to each row

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_zblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_zgeellrtmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t nnz_per_row,
    magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex_ptr dval,
    magma_tally2Index_ptr dcolind,
    magma_tally2Index_ptr drowlength,
    magma_tally2DoubleComplex_ptr dx,
    magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex_ptr dy,
    magma_tally2_int_t alignment,
    magma_tally2_int_t blocksize,
    magma_tally2_queue_t queue )
{
    int num_blocks = magma_tally2_ceildiv( m, blocksize );

    magma_tally2_int_t num_threads = alignment*blocksize;
    magma_tally2_int_t threads = alignment*blocksize;

    int real_row_length = magma_tally2_roundup( nnz_per_row, alignment );

    magma_tally2_int_t arch = magma_tally2_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = (int) sqrt( (double) num_blocks );
    int dimgrid2 = magma_tally2_ceildiv( num_blocks, dimgrid1 );
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( magma_tally2DoubleComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( alignment == 32 ) {
        zgeellrtmv_kernel_32<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        zgeellrtmv_kernel_16<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        zgeellrtmv_kernel_8<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", alignment);
        return MAGMA_tally2_ERR_NOT_SUPPORTED;
    }



   return MAGMA_tally2_SUCCESS;
}


