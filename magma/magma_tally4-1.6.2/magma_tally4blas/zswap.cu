#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Mark Gates

       @precisions normal z -> s d c

*/
#include "common_magma_tally4.h"

#define NB 64


/* Vector is divided into ceil(n/nb) blocks.
   Each thread swaps one element, x[tid] <---> y[tid].
*/
__global__ void zswap_kernel(
    int n,
    magma_tally4DoubleComplex *x, int incx,
    magma_tally4DoubleComplex *y, int incy )
{
    magma_tally4DoubleComplex tmp;
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    if ( ind < n ) {
        x += ind*incx;
        y += ind*incy;
        tmp = *x;
        *x  = *y;
        *y  = tmp;
    }
}


/**
    Purpose:
    =============
    Swap vector x and y; \f$ x <-> y \f$.

    @param[in]
    n       Number of elements in vector x and y. n >= 0.

    @param[in,out]
    dx      COMPLEX_16 array on GPU device.
            The n element vector x of dimension (1 + (n-1)*incx).

    @param[in]
    incx    Stride between consecutive elements of dx. incx != 0.

    @param[in,out]
    dy      COMPLEX_16 array on GPU device.
            The n element vector y of dimension (1 + (n-1)*incy).

    @param[in]
    incy    Stride between consecutive elements of dy. incy != 0.

    @ingroup magma_tally4_zblas1
    ********************************************************************/
extern "C" void 
magma_tally4blas_zswap_q(
    magma_tally4_int_t n,
    magma_tally4DoubleComplex_ptr dx, magma_tally4_int_t incx, 
    magma_tally4DoubleComplex_ptr dy, magma_tally4_int_t incy,
    magma_tally4_queue_t queue )
{
    dim3 grid( (n+NB-1) / NB );
    dim3 threads( NB );
    zswap_kernel<<< grid, threads, 0, queue >>>( n, dx, incx, dy, incy );
}


/**
    @see magma_tally4blas_zswap_q
    @ingroup magma_tally4_zblas1
    ********************************************************************/
extern "C" void 
magma_tally4blas_zswap(
    magma_tally4_int_t n,
    magma_tally4DoubleComplex_ptr dx, magma_tally4_int_t incx, 
    magma_tally4DoubleComplex_ptr dy, magma_tally4_int_t incy)
{
    magma_tally4blas_zswap_q( n, dx, incx, dy, incy, magma_tally4_stream );
}
