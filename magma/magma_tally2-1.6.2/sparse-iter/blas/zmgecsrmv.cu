#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/
#include "common_magma_tally2.h"

#define BLOCK_SIZE 512


__global__ void 
zmgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    magma_tally2DoubleComplex alpha, 
    magma_tally2DoubleComplex * dval, 
    magma_tally2_index_t * drowptr, 
    magma_tally2_index_t * dcolind,
    magma_tally2DoubleComplex * dx,
    magma_tally2DoubleComplex beta, 
    magma_tally2DoubleComplex * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magma_tally2DoubleComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_tally2_Z_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            magma_tally2DoubleComplex val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_tally2_trans_t
                transposition parameter for A

    @param[in]
    m           magma_tally2_int_t
                number of rows in A

    @param[in]
    n           magma_tally2_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       magma_tally2DoubleComplex
                scalar multiplier

    @param[in]
    dval        magma_tally2DoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magma_tally2Index_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magma_tally2Index_ptr
                columnindices of A in CSR

    @param[in]
    dx          magma_tally2DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally2DoubleComplex
                scalar multiplier

    @param[out]
    dy          magma_tally2DoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_tally2_queue_t
                Queue to execute in.

    @ingroup magma_tally2sparse_zblas
    ********************************************************************/

extern "C" magma_tally2_int_t
magma_tally2_zmgecsrmv(
    magma_tally2_trans_t transA,
    magma_tally2_int_t m, magma_tally2_int_t n,
    magma_tally2_int_t num_vecs, 
    magma_tally2DoubleComplex alpha,
    magma_tally2DoubleComplex_ptr dval,
    magma_tally2Index_ptr drowptr,
    magma_tally2Index_ptr dcolind,
    magma_tally2DoubleComplex_ptr dx,
    magma_tally2DoubleComplex beta,
    magma_tally2DoubleComplex_ptr dy,
    magma_tally2_queue_t queue )
{
    dim3 grid( magma_tally2_ceildiv( m, BLOCK_SIZE ), 1, 1);
    magma_tally2_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magma_tally2DoubleComplex ); // num_vecs vectors 
    zmgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_tally2_SUCCESS;
}



