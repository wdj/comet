#include "hip/hip_runtime.h"
/*
    -- MAGMA_minproduct (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c


       @author Adrien REMY
*/
#include "common_magma_minproduct.h"
#include "zgerbt.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_minproduct_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_minproductblas_zprbt_mtv_q(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *du, magma_minproductDoubleComplex *db,
    magma_minproduct_queue_t queue)
{
    /*

     */
    magma_minproduct_int_t threads = block_length;
    magma_minproduct_int_t grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_minproductblas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n, db, 0);
    magma_minproductblas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n+n/2, db, n/2);

    threads = block_length;
    grid = n/(2*block_length) + ((n%(2*block_length))!=0);
    magma_minproductblas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n, du, 0, db, 0);
}

/**
    @see magma_minproductblas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_minproductblas_zprbt_mtv(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *du, magma_minproductDoubleComplex *db)
{
    magma_minproductblas_zprbt_mtv_q(n, du, db, magma_minproduct_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    ZPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_minproduct_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_minproductblas_zprbt_mv_q(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *dv, magma_minproductDoubleComplex *db,
    magma_minproduct_queue_t queue)
{

    magma_minproduct_int_t threads = block_length;
    magma_minproduct_int_t grid = n/(2*block_length) + ((n%(2*block_length))!=0);

    magma_minproductblas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n, dv, 0, db, 0);


    threads = block_length;
    grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_minproductblas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n, db, 0);
    magma_minproductblas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n+n/2, db, n/2);
}

/**
    @see magma_minproductblas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_minproductblas_zprbt_mv(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *dv, magma_minproductDoubleComplex *db)
{
    magma_minproductblas_zprbt_mv_q(n, dv, db, magma_minproduct_stream);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_minproduct_queue_t
            Queue to execute in.

    ********************************************************************/
extern "C" void 
magma_minproductblas_zprbt_q(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *dA, magma_minproduct_int_t ldda, 
    magma_minproductDoubleComplex *du, magma_minproductDoubleComplex *dv,
    magma_minproduct_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid(n/(4*block_height) + ((n%(4*block_height))!=0), 
            n/(4*block_width)  + ((n%(4*block_width))!=0));

    magma_minproductblas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,            0, ldda, du,   0, dv,   0);
    magma_minproductblas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,     ldda*n/2, ldda, du,   0, dv, n/2);
    magma_minproductblas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,          n/2, ldda, du, n/2, dv,   0);
    magma_minproductblas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2(n/(2*block_height) + ((n%(2*block_height))!=0), 
            n/(2*block_width)  + ((n%(2*block_width))!=0));
    magma_minproductblas_zelementary_multiplication_kernel<<< grid2, threads2, 0, queue >>>(n, dA, 0, ldda, du, -ldda, dv, -ldda);
}


/**
    @see magma_minproductblas_zprbt_q
    ********************************************************************/
extern "C" void 
magma_minproductblas_zprbt(
    magma_minproduct_int_t n, 
    magma_minproductDoubleComplex *dA, magma_minproduct_int_t ldda, 
    magma_minproductDoubleComplex *du, magma_minproductDoubleComplex *dv)
{
    magma_minproductblas_zprbt_q(n, dA, ldda, du, dv, magma_minproduct_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
zaxpycp2_kernel(
    int m, magma_minproductDoubleComplex *r, magma_minproductDoubleComplex *x,
    const magma_minproductDoubleComplex *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_minproduct_Z_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magma_minproductblas_zaxpycp2_q(
    magma_minproduct_int_t m, magma_minproductDoubleComplex *r, magma_minproductDoubleComplex *x,
    const magma_minproductDoubleComplex *b,
    magma_minproduct_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    zaxpycp2_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b );
}


extern "C" void
magma_minproductblas_zaxpycp2(
    magma_minproduct_int_t m, magma_minproductDoubleComplex *r, magma_minproductDoubleComplex *x,
    const magma_minproductDoubleComplex *b)
{
    magma_minproductblas_zaxpycp2_q( m, r, x, b, magma_minproduct_stream );
}
