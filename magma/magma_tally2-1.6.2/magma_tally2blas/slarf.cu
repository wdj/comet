#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarf.cu normal z -> s, Fri Jan 30 19:00:08 2015
       @author Azzam Haidar

*/
#include "common_magma_tally2.h"
#include "magma_tally2_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================
//==============================================================================

__global__
void magma_tally2_slarf_kernel( int m, const float *dv, const float *dtau,
                         float *dc, int lddc )
{
    if ( !MAGMA_tally2_S_EQUAL(*dtau, MAGMA_tally2_S_ZERO) ) {
        const int tx = threadIdx.x;
        dc = dc + blockIdx.x * lddc;

        __shared__ float sum[ BLOCK_SIZE ];
        float tmp;

        /* perform  w := v**H * C  */
        if (tx==0)
            tmp = dc[0]; //since V[0] should be one
        else
            tmp = MAGMA_tally2_S_ZERO;
        for( int j = tx+1; j < m; j += BLOCK_SIZE ){
            tmp += MAGMA_tally2_S_MUL( MAGMA_tally2_S_CNJG( dv[j] ), dc[j] );
        }
        sum[tx] = tmp;
        magma_tally2_sum_reduce< BLOCK_SIZE >( tx, sum );

        /*  C := C - v * w  */
        __syncthreads();
        tmp = - MAGMA_tally2_S_CNJG(*dtau) * sum[0];
        for( int j = m-tx-1; j>0 ; j -= BLOCK_SIZE )
             dc[j] += tmp * dv[j];

        if(tx==0) dc[0] += tmp;
    }
}

//==============================================================================
//==============================================================================

__global__
void magma_tally2_slarf_smkernel( int m, int n, float *dv, float *dtau,
                           float *dc, int lddc )
{
    if ( ! MAGMA_tally2_S_EQUAL(*dtau, MAGMA_tally2_S_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            dc = dc + k * lddc;
    
            __shared__ float sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            float lsum;
    
            /*  w := v**H * C  */
            lsum = MAGMA_tally2_S_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_tally2_S_MUL( MAGMA_tally2_S_ONE, dc[j] );
                else
                   lsum += MAGMA_tally2_S_MUL( MAGMA_tally2_S_CNJG( dv[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_tally2_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            float z__1 = - MAGMA_tally2_S_CNJG(*dtau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * dv[j];
            }
        }
    }
}

//==============================================================================

/*
    Apply a real elementary reflector H to a real M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v**H
    where tau is a real scalar and v is a real vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H**H (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_tally2_slarf_sm(magma_tally2_int_t m, magma_tally2_int_t n, float *dv, float *dtau,
               float *dc, magma_tally2_int_t lddc)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_tally2_slarf_smkernel<<< blocks, threads, 0, magma_tally2_stream >>>( m, n, dv, dtau, dc, lddc );
}
//==============================================================================
/*
    Apply a real elementary reflector H to a real M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v**H
    where tau is a real scalar and v is a real vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H**H (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

 */

extern "C" magma_tally2_int_t
magma_tally2_slarf_gpu(
    magma_tally2_int_t m,  magma_tally2_int_t n,
    magma_tally2Float_const_ptr dv,
    magma_tally2Float_const_ptr dtau,
    magma_tally2Float_ptr dC,  magma_tally2_int_t lddc)
{
    dim3 grid( n, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    if ( n > 0 ) {
        magma_tally2_slarf_kernel<<< grid, threads, 0, magma_tally2_stream >>>( m, dv, dtau, dC, lddc);
    }

    // The computation can be done on 1 SM with the following routine.
    // magma_tally2_slarf_sm(m, n, dv, dtau, dc, lddc);

    return MAGMA_tally2_SUCCESS;
}

//==============================================================================
