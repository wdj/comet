#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally4 (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @precisions normal z -> c d s

*/

#include "common_magma_tally4.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4_index_t * drowlength,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    magma_tally4DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally4DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally4DoubleComplex dot = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally4_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally4DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally4DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4_index_t * drowlength,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    magma_tally4DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally4DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally4DoubleComplex dot = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally4_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally4DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally4DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    magma_tally4DoubleComplex alpha, 
    magma_tally4DoubleComplex * dval, 
    magma_tally4_index_t * dcolind,
    magma_tally4_index_t * drowlength,
    magma_tally4DoubleComplex * dx,
    magma_tally4DoubleComplex beta, 
    magma_tally4DoubleComplex * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magma_tally4DoubleComplex shared[];

    if(i < num_rows ){
        magma_tally4DoubleComplex dot = MAGMA_tally4_Z_MAKE(0.0, 0.0);
        int max_ = magma_tally4_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magma_tally4DoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magma_tally4DoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_tally4_trans_t
                transposition parameter for A
    @param[in]
    m           magma_tally4_int_t
                number of rows 

    @param[in]
    n           magma_tally4_int_t
                number of columns

    @param[in]
    nnz_per_row magma_tally4_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       magma_tally4DoubleComplex
                scalar alpha

    @param[in]
    dval        magma_tally4DoubleComplex_ptr
                val array

    @param[in]
    dcolind     magma_tally4Index_ptr
                col indices  

    @param[in]
    drowlength  magma_tally4Index_ptr
                number of elements in each row

    @param[in]
    dx          magma_tally4DoubleComplex_ptr
                input vector x

    @param[in]
    beta        magma_tally4DoubleComplex
                scalar beta

    @param[out]
    dy          magma_tally4DoubleComplex_ptr
                output vector y

    @param[in]
    blocksize   magma_tally4_int_t
                threads per block

    @param[in]
    alignment   magma_tally4_int_t
                threads assigned to each row

    @param[in]
    queue       magma_tally4_queue_t
                Queue to execute in.

    @ingroup magma_tally4sparse_zblas
    ********************************************************************/

extern "C" magma_tally4_int_t
magma_tally4_zgeellrtmv(
    magma_tally4_trans_t transA,
    magma_tally4_int_t m, magma_tally4_int_t n,
    magma_tally4_int_t nnz_per_row,
    magma_tally4DoubleComplex alpha,
    magma_tally4DoubleComplex_ptr dval,
    magma_tally4Index_ptr dcolind,
    magma_tally4Index_ptr drowlength,
    magma_tally4DoubleComplex_ptr dx,
    magma_tally4DoubleComplex beta,
    magma_tally4DoubleComplex_ptr dy,
    magma_tally4_int_t alignment,
    magma_tally4_int_t blocksize,
    magma_tally4_queue_t queue )
{
    int num_blocks = magma_tally4_ceildiv( m, blocksize );

    magma_tally4_int_t num_threads = alignment*blocksize;
    magma_tally4_int_t threads = alignment*blocksize;

    int real_row_length = magma_tally4_roundup( nnz_per_row, alignment );

    magma_tally4_int_t arch = magma_tally4_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = (int) sqrt( (double) num_blocks );
    int dimgrid2 = magma_tally4_ceildiv( num_blocks, dimgrid1 );
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( magma_tally4DoubleComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( alignment == 32 ) {
        zgeellrtmv_kernel_32<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        zgeellrtmv_kernel_16<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        zgeellrtmv_kernel_8<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", alignment);
        return MAGMA_tally4_ERR_NOT_SUPPORTED;
    }



   return MAGMA_tally4_SUCCESS;
}


