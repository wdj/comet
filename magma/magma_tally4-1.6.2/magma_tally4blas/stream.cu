/*
    -- MAGMA_tally4 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Mark Gates
*/

#include "common_magma_tally4.h"

magma_tally4_queue_t magma_tally4_stream = 0;


/**
    Purpose
    -------
    magma_tally4blasSetKernelStream sets the CUDA stream that MAGMA_tally4 BLAS and
    CUBLAS (v1) routines use (unless explicitly given a stream).
    
    In a multi-threaded application, be careful to avoid race conditions
    when using this. For instance, if calls are executed in this order:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  magma_tally4blasSetKernelStream( s1 )         
    2.                                      magma_tally4blasSetKernelStream( s2 )
    3.  magma_tally4_dgemm( ... )
    4.                                      magma_tally4_dgemm( ... )
    @endverbatim
    
    both magma_tally4_dgemm would occur on stream s2. A lock should be used to prevent
    this, so the dgemm in thread 1 uses stream s1, and the dgemm in thread 2
    uses s2:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  lock()                                  
    2.  magma_tally4blasSetKernelStream( s1 )          
    3.  magma_tally4_dgemm( ... )                      
    4.  unlock()                                
    5.                                      lock()
    6.                                      magma_tally4blasSetKernelStream( s2 )
    7.                                      magma_tally4_dgemm( ... )
    8.                                      unlock()
    @endverbatim
    
    Most BLAS calls in MAGMA_tally4, such as magma_tally4_dgemm, are asynchronous, so the lock
    will only have to wait until dgemm is queued, not until it is finished.
    
    Arguments
    ---------
    @param[in]
    stream  magma_tally4_queue_t
            The CUDA stream.

    @ingroup magma_tally4_util
    ********************************************************************/
extern "C"
hipblasStatus_t magma_tally4blasSetKernelStream( magma_tally4_queue_t stream )
{
    magma_tally4_stream = stream;
    return cublasSetKernelStream( stream );
}


/**
    Purpose
    -------
    magma_tally4blasGetKernelStream gets the CUDA stream that MAGMA_tally4 BLAS
    routines use.

    Arguments
    ---------
    @param[out]
    stream  magma_tally4_queue_t
            The CUDA stream.

    @ingroup magma_tally4_util
    ********************************************************************/
extern "C"
hipblasStatus_t magma_tally4blasGetKernelStream( magma_tally4_queue_t *stream )
{
    *stream = magma_tally4_stream;
    return HIPBLAS_STATUS_SUCCESS;
}
