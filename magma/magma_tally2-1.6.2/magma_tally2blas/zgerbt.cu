#include "hip/hip_runtime.h"
/*
    -- MAGMA_tally2 (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @precisions normal z -> s d c


       @author Adrien REMY
*/
#include "common_magma_tally2.h"
#include "zgerbt.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_tally2_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_tally2blas_zprbt_mtv_q(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *du, magma_tally2DoubleComplex *db,
    magma_tally2_queue_t queue)
{
    /*

     */
    magma_tally2_int_t threads = block_length;
    magma_tally2_int_t grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_tally2blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n, db, 0);
    magma_tally2blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n+n/2, db, n/2);

    threads = block_length;
    grid = n/(2*block_length) + ((n%(2*block_length))!=0);
    magma_tally2blas_zapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n, du, 0, db, 0);
}

/**
    @see magma_tally2blas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_tally2blas_zprbt_mtv(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *du, magma_tally2DoubleComplex *db)
{
    magma_tally2blas_zprbt_mtv_q(n, du, db, magma_tally2_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    ZPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_tally2_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magma_tally2blas_zprbt_mv_q(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *dv, magma_tally2DoubleComplex *db,
    magma_tally2_queue_t queue)
{

    magma_tally2_int_t threads = block_length;
    magma_tally2_int_t grid = n/(2*block_length) + ((n%(2*block_length))!=0);

    magma_tally2blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n, dv, 0, db, 0);


    threads = block_length;
    grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magma_tally2blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n, db, 0);
    magma_tally2blas_zapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n+n/2, db, n/2);
}

/**
    @see magma_tally2blas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magma_tally2blas_zprbt_mv(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *dv, magma_tally2DoubleComplex *db)
{
    magma_tally2blas_zprbt_mv_q(n, dv, db, magma_tally2_stream);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_tally2_queue_t
            Queue to execute in.

    ********************************************************************/
extern "C" void 
magma_tally2blas_zprbt_q(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *dA, magma_tally2_int_t ldda, 
    magma_tally2DoubleComplex *du, magma_tally2DoubleComplex *dv,
    magma_tally2_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid(n/(4*block_height) + ((n%(4*block_height))!=0), 
            n/(4*block_width)  + ((n%(4*block_width))!=0));

    magma_tally2blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,            0, ldda, du,   0, dv,   0);
    magma_tally2blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,     ldda*n/2, ldda, du,   0, dv, n/2);
    magma_tally2blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,          n/2, ldda, du, n/2, dv,   0);
    magma_tally2blas_zelementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2(n/(2*block_height) + ((n%(2*block_height))!=0), 
            n/(2*block_width)  + ((n%(2*block_width))!=0));
    magma_tally2blas_zelementary_multiplication_kernel<<< grid2, threads2, 0, queue >>>(n, dA, 0, ldda, du, -ldda, dv, -ldda);
}


/**
    @see magma_tally2blas_zprbt_q
    ********************************************************************/
extern "C" void 
magma_tally2blas_zprbt(
    magma_tally2_int_t n, 
    magma_tally2DoubleComplex *dA, magma_tally2_int_t ldda, 
    magma_tally2DoubleComplex *du, magma_tally2DoubleComplex *dv)
{
    magma_tally2blas_zprbt_q(n, dA, ldda, du, dv, magma_tally2_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
zaxpycp2_kernel(
    int m, magma_tally2DoubleComplex *r, magma_tally2DoubleComplex *x,
    const magma_tally2DoubleComplex *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_tally2_Z_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magma_tally2blas_zaxpycp2_q(
    magma_tally2_int_t m, magma_tally2DoubleComplex *r, magma_tally2DoubleComplex *x,
    const magma_tally2DoubleComplex *b,
    magma_tally2_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    zaxpycp2_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b );
}


extern "C" void
magma_tally2blas_zaxpycp2(
    magma_tally2_int_t m, magma_tally2DoubleComplex *r, magma_tally2DoubleComplex *x,
    const magma_tally2DoubleComplex *b)
{
    magma_tally2blas_zaxpycp2_q( m, r, x, b, magma_tally2_stream );
}
