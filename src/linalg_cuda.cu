#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
/*!
 * \file   linalg_cuda.cu
 * \author Wayne Joubert
 * \date   Tue May 15 12:03:55 EDT 2018
 * \brief  CUDA code to support linear algebra operations.
 * \note   Copyright (C) 2018 Oak Ridge National Laboratory, UT-Battelle, LLC.
 */
//-----------------------------------------------------------------------------

#include "stdint.h"

#include "hipblas.h"
#include "hip/hip_fp16.h"

#include "env.hh"
#include "linalg_cuda.cuh"

//=============================================================================
// HELPERS
//=============================================================================

//-----------------------------------------------------------------------------
/// \brief Specialized class to provide selected constants of GemmIn_t type.
///
///        Provide constants "0", "1" and "2" in the datatype
///        required to input to the cuBLAS reduced precision GEMM.

// Note: we could use __half here instead of GMUInt16.  The intent here
// was to use a type based on standard C/C++.  No actual computations
// are done in this code based on the specifics of the type, so it doesn't
// matter.  Important thing is that sizeof(GMUInt16) == sizeof(__half) == 2.

template<typename GemmIn_t> struct TCBufTypes;

template<> struct TCBufTypes<GMUInt16> {
  static __device__ GMUInt16 zero() {return (GMUInt16)0x0000;}
                                        // = *(GMUInt16*)&__float2half(0.);
  static __device__ GMUInt16 one() {return (GMUInt16)0x3c00;}
                                        // = *(GMUInt16*)&__float2half(1.);
  static __device__ GMUInt16 two() {return (GMUInt16)0x4000;}
                                        // = *(GMUInt16*)&__float2half(2.);
};

template<> struct TCBufTypes<GMUInt8> {
  static __device__ GMUInt8 zero() {return (GMUInt8)0;}
  static __device__ GMUInt8 one() {return (GMUInt8)1;}
  static __device__ GMUInt8 two() {return (GMUInt8)2;}
};

//-----------------------------------------------------------------------------
/// \brief Seclector for types etc. for tc methods.

template<int TC_METHOD> struct TCSelector;

template<> struct TCSelector<GM_TC_METHOD_INT8> {
  typedef GMUInt8 GemmIn_t;
  typedef int32_t GemmOut_t;
  static bool __host__ __device__ is_gemm_out_type_int32() {return true;}
  static hipDataType __host__ __device__ cuda_type_in() {return HIP_R_8I;}
  static hipDataType __host__ __device__ cuda_type_out() {return HIP_R_32I;}
  enum { COUNT = 1 };
};

template<> struct TCSelector<GM_TC_METHOD_FLOAT16> {
  typedef GMUInt16 GemmIn_t;
  typedef float GemmOut_t;
  static bool __host__ __device__ is_gemm_out_type_int32() {return false;}
  static hipDataType __host__ __device__ cuda_type_in() {return HIP_R_16F;}
  static hipDataType __host__ __device__ cuda_type_out() {return HIP_R_32F;}
  enum { COUNT = 1 };
};

//=============================================================================
// FILE_LOCAL (STATIC) FUNCTIONS
//=============================================================================

//-----------------------------------------------------------------------------
/// \brief GPU kernel to support gm_tc_buf_write_.

template<typename GemmIn_t>
__global__ static void gm_tc_buf_write_kernel_(
  GemmIn_t* vo,
  GMUInt32* vi32,
  int vi32_dim0,
  int num_way,
  bool is_sparse,
  bool is_right,
  int nvlea,
  int nvle,
  int nvleD2,
  int nvleX2,
  int nfl,
  int nflD2,
  int nflD2_step,
  int flD2_min) {

  // Two fields (seminibbles) map to two halves of (2*sizeof(GemmIn_t))-bit word

  const int vlX2 = threadIdx.x + blockIdx.x * blockDim.x;
  const int flD2_step = blockIdx.y + gridDim.y * blockIdx.z;

  if (vlX2 >= nvleX2 || flD2_step >= nflD2_step) {
    return;
  }

  const int i01 = vlX2 % 2; // count either 0 bits or 1 bits.
  const int vl = vlX2 / 2;

  const int flD2 = flD2_min + flD2_step;

  // Output array interpreted as having GemmIn_t scalars has nfl rows.

  const GMUInt32* const vi32_col = vi32 + vl * (size_t)vi32_dim0;

  // Pick up two consecutive field values:
  // first field seminibble0, second field seminibble1
  // Set to zero if outside of active range.

  const int nibble = vl<nvlea ? (vi32_col[flD2/8] >> (4*(flD2%8))) & 15 : 0;

  const int seminibble0 = nibble & 3;
  const int seminibble1 = (nibble>>2) & 3;

  // Count number of 0 (or 1) bits in respective seminibble.
  // Determine whether to skip (1,0) null indicator value.

  const bool skip_10 = is_sparse || (num_way == 3 && ! is_right);

  // Possible counts, represented in target type.
  const GemmIn_t zero = TCBufTypes<GemmIn_t>::zero();
  const GemmIn_t one = TCBufTypes<GemmIn_t>::one();
  const GemmIn_t two = TCBufTypes<GemmIn_t>::two();

  const GemmIn_t out0 = seminibble0 == 3*i01     ? two :
                        seminibble0 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble0 == 1         ? one :
                                                   zero;

  const GemmIn_t out1 = seminibble1 == 3*i01     ? two :
                        seminibble1 == 3*(1-i01) ? zero :
                                       !skip_10  ? one :
                        seminibble1 == 1         ? one :
                                                   zero;
  // Always keep pair of cols together, corresponding to the two i01 values.
  // Right case: straight copy of cols to cols in sequence.
  // Left case: interleave to make later swizzling of metrics array work:
  // [ A A B B C C D D E E F F ] -> [ A A D D B B E E C C F F]

  const int vl_index = is_right ? vl : vl < nvleD2 ? 2*vl : 2*vl - nvle + 1;
  const int vlX2_index = i01 + 2*vl_index;

  const int flD2_index = flD2_step;

  const int fl_index_0 = 0 + 2 * flD2_index;
  const int fl_index_1 = 1 + 2 * flD2_index;

  const int vlX2_dim = nvleX2;

  vo[vlX2_index + vlX2_dim * (size_t)fl_index_0] = out0;
  vo[vlX2_index + vlX2_dim * (size_t)fl_index_1] = out1;
}

//-----------------------------------------------------------------------------
/// \brief Convert bitwise matrix to required format for GEMM.

template<int TC_METHOD>
static void gm_tc_buf_write_(
  bool is_right,
  int I_max,
  int I_max_dim,
  int nvl,
  int npvfl,
  int npvfl_step,
  int pvfl_min,
  void* vi,
  TCBufs& tc_bufs,
  GMEnv* env) {

  GMInsist(env && vi);
  GMInsist(I_max_dim >= 0 && I_max_dim <= nvl);
  GMInsist(I_max >= 0 && I_max <= I_max_dim);
  GMInsist(nvl >= 0);
  GMInsist(npvfl >= 0);
  GMInsist(tc_bufs.tc_buf_left);
  GMInsist(tc_bufs.tc_buf_right);
  GMInsist(npvfl >= 0);
  GMInsist(npvfl_step >= 0 && npvfl_step <= npvfl);
  GMInsist(pvfl_min >= 0 && pvfl_min + npvfl_step <= npvfl);

  // num_vector-related dimensions.

  const int nvle = is_right ? nvl : I_max_dim; // effective nvl dimension
  const int nvleD2 = nvle / 2;
  const int nvleX2 = nvle * 2;
  const int nvlea = is_right ? nvl : I_max; // num active nvle; others zeroed
  // NOTE: we are ignoring the issue from decomp_mgr that
  // num_vector_active_local may be strictly less than num_vector_local;
  // doesn't matter: just compute on fake values that will later be ignored.

  GMInsist(nvle % 2 == 0 && nvl % 2 == 0 &&
           "tc method here requires num_vector_local multiple of 2.");

  // num_field-related dimensions.

  const int nfl = npvfl * 64;
  const int nflD2 = nfl / 2;
  const int nfl_step = npvfl_step * 64;
  const int nflD2_step = nfl_step / 2;
  const int fl_min = pvfl_min * 64;
  const int flD2_min = fl_min / 2;
  // Remember: end padding is set to zero; will correct zero counts later.

  // CUDA thread dims.

  const int threadblocksize = 256;
  const int blockdim_y = 32768;
  const int num_threadblocks_0 = gm_ceil_i8(nvleX2, threadblocksize);
  const int num_threadblocks_1 = gm_min_i8(nflD2_step, blockdim_y);
  const int num_threadblocks_2 = gm_ceil_i8(nflD2_step, blockdim_y);

  // Arrays.

  typedef typename TCSelector<TC_METHOD>::GemmIn_t GemmIn_t;
  GMUInt32* vi32 = (GMUInt32*)vi;
  const int vi32_dim0 = npvfl * 4; // 4 = sizeof(doublecomplex) / sizeof(int32)
  GemmIn_t* const tc_buf = is_right ? (GemmIn_t*)tc_bufs.tc_buf_right :
                                      (GemmIn_t*)tc_bufs.tc_buf_left;
  GMInsist(nvleX2 * (2*nflD2_step) *
           sizeof(typename TCSelector<TC_METHOD>::GemmIn_t)
           <= tc_bufs.tc_buf_size &&
           "Subscriptrange error on tc buf.");

  // Kernel call.

  gm_tc_buf_write_kernel_<GemmIn_t><<<
      dim3(num_threadblocks_0, num_threadblocks_1, num_threadblocks_2),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(
    tc_buf, vi32, vi32_dim0,
    GMEnv_num_way(env), env->sparse, is_right,
    nvlea, nvle, nvleD2, nvleX2, nfl, nflD2, nflD2_step, flD2_min);

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
/// \brief Call cublas to perform required GEMM.

template<int TC_METHOD>
static void gm_tc_solve_(
  bool is_first,
  int nvll,
  int nvl,
  int npvfl_step,
  void* dA,
  void* dB,
  void* dC,
  TCBufs& tc_bufs,
  GMEnv* env) {

  GMInsist(env && dA && dB && dC);
  GMInsist(nvll >= 0);
  GMInsist(nvl >= 0);
  GMInsist(nvll <= nvl);
  GMInsist(npvfl_step >= 0);
  GMInsist(env->tc >= 1 && env->tc < GM_NUM_TC_METHOD);

#if __CUDACC_VER_MAJOR__ >= 9

  const int nfl_step = npvfl_step * 64;

  const int m = 2 * nvll; // metrics array dim
  const int n = 2 * nvl; // metrics array dim
  const int k = nfl_step; // vectors array (as GemmIn_t) dim

  const typename TCSelector<TC_METHOD>::GemmOut_t alpha = 1;
  const typename TCSelector<TC_METHOD>::GemmOut_t beta = is_first ? 0 : 1;

  // See https://devblogs.nvidia.com/programming-tensor-cores-cuda-9/
  // "Invoke the GEMM, ensuring k, lda, ldb, and ldc are all multiples of 8, 
  // and m is a multiple of 4"
  // "GEMMs that do not satisfy the above rules will fall back
  // to a non-Tensor Core implementation"
  // See also https://docs.nvidia.com/cuda/cublas/index.html#cublas-gemmEx

  GMInsist(k % 8 == 0); // nfl is derived from padded-up npvfl, so always ok.
  GMInsist(m % 8 == 0); // need I_max_dim % 4 == 0; see gm_gemm_size_required()
  GMInsist(n % 8 == 0); // need nvl % 4 == 0

  // Make BLAS call.

  hipblasStatus_t status = hipblasGemmEx(
    tc_bufs.cublas_handle,
    HIPBLAS_OP_N, HIPBLAS_OP_T,
    m, n, k,
    (void*)&alpha,
    tc_bufs.tc_buf_left,
    TCSelector<TC_METHOD>::cuda_type_in(),
    m,
    tc_bufs.tc_buf_right,
    TCSelector<TC_METHOD>::cuda_type_in(),
    n,
    (void*)&beta,
    dC,
    TCSelector<TC_METHOD>::cuda_type_out(),
    m,
    TCSelector<TC_METHOD>::cuda_type_out(),
    //CUBLAS_GEMM_ALGO3_TENSOR_OP // best timing, for cuda 9.1.85, transpose
    //CUBLAS_GEMM_DFALT_TENSOR_OP // good timing, for cuda 9.2.88, transpose
    CUBLAS_GEMM_ALGO4_TENSOR_OP // best timing, for cuda 9.2.88, transpose
  );
  // TODO: use CUDA 10 autotuning here (later).

  if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
    printf("Error: HIPBLAS_STATUS_NOT_INITIALIZED\n");
  } else if (status == HIPBLAS_STATUS_ARCH_MISMATCH) {
    printf("Error: HIPBLAS_STATUS_ARCH_MISMATCH\n");
  } else if (status == HIPBLAS_STATUS_NOT_SUPPORTED) {
    printf("Error: HIPBLAS_STATUS_NOT_SUPPORTED\n");
  } else if (status == HIPBLAS_STATUS_INVALID_VALUE) {
    printf("Error: HIPBLAS_STATUS_INVALID_VALUE\n");
  } else if (status == HIPBLAS_STATUS_EXECUTION_FAILED) {
    printf("Error: HIPBLAS_STATUS_EXECUTION_FAILED\n");
  }

  GMInsist(status == HIPBLAS_STATUS_SUCCESS);

  env->ops_local += 2 * m * (double)n * (double)k;

#endif // __CUDACC_VER_MAJOR__
}

//-----------------------------------------------------------------------------
/// \brief GPU kernel to support gm_tc_repair_metrics_.
///
///        This function has two purposes:
///        1. Convert the 2X2 table from each pair of compared vectors
///        from 4 32-bit (int32 or float32) values to the required
///        16-byte double complex packed format.
///        2. Permute the table elements to the required places.
///
///        The reason for the permutation is as follows.
///        For the output matrix of this function, each single 2X2 matrix
///        is arranged contiguously in memory as a double complex value.
///        However, the input matrices to the GEMM do not give a result
///        matrix that is consistent with this ordering.
///        Thus there needs to be a copy to rearrange.  Furthermore,
///        we want to make this an in-place rearrangement to save
///        space, and additionally we want to assign work to threads
///        with no race conditions and with coalesced memory accesses.
///
///        The method can be explained as follows.
///        1. The input "left" and "right" matrices to the modified GEMM
///        can be thought of each as a group of column vectors.
///        2. Each column (of 2-bit entries) is converted into two columns,
///        with entries being the counts of 0 bits and 1 bits of the
///        original vectors.  Each pair of vectors is kept together
///        side-by-side in these new left and right matrices L and R.
///        3. The columns of L are permuted, to give L' = L P
///        Example:
///          R  = [ G, G, H, H, I, I, J, J, K, K, L, L ]
///          L  = [ A, A, B, B, C, C, D, D, E, E, F, F ]
///          L' = [ A, A, D, D, B, B, E, E, C, C, F, F ]
///        (note L is used in 2 different senses here)
///        4. The GEMM is computed, M = (L')^T R = P^T L^T R.  Because of
///        the permutation of L, the rows of M are permuted.
///        Here, for brevity we drop the transpose, writing A^T G as AG, etc.
///          M = [ AG, AG, AH, AH, . . . ]
///              [ AG, AG, AH, AH, . . . ]
///              [ DG, DG, DH, DH, . . . ]
///              [ DG, DG, DH, DH, . . . ]
///              [ BG, BG, BH, BH, . . . ]
///              [ BG, BG, BH, BH, . . . ]
///              [ EG, EG, EH, EH, . . . ]
///              [ EG, EG, EH, EH, . . . ]
///              [ CG, CG, CH, CH, . . . ]
///              [ CG, CG, CH, CH, . . . ]
///              [ FG, FG, FH, FH, . . . ]
///              [ FG, FG, FH, FH, . . . ]
///        Here we are considering M to be stored in column-major order.
///        5. Next we consider this as composed of size 4X2 blocks,
///        assign a CUDA thread to each block and do an in-block
///        permutation. Note each thread loads 2 16-byte (double) words,
///        with stride between threads of 16 bytes.
///        (need to check on efficiency of this w.r.t. coalescing etc.)
///          [ AG, AG ] -> [ AG, DG ]
///          [ AG, AG ] -> [ AG, DG ]
///          [ DG, DG ] -> [ AG, DG ]
///          [ DG, DG ] -> [ AG, DG ]
///        As can be seen, all four entries AG of the table are now
///        contiguous in memory.

template<typename GemmOut_t>
__global__ static void gm_tc_repair_metrics_kernel_(
  int nvl, int nvll, int nvll2, void* vo) { 
  // Row and column of metrics array.

  const int thread_r = threadIdx.x + blockIdx.x * blockDim.x;
  const int thread_c = blockIdx.y;

  if (thread_r >= nvll2 || thread_c >= nvl) {
    return;
  }

  // Considered as an array of floats, array is 2*nvl rows X 2*nvl cols.
  // Each thread manipulates a block of 4 rows and 2 cols.
  // Thus the dimensions of the metrics array in blocks is nvll2 X nvl.
  // Each block viewed as an array of doubles is 2 X 2.

  // Two col numbers being processed of this (float) array.

  // ISSUE: does the compiler need to / understand that the pointers are aliased

//  const size_t fc_offset0 = thread_c * (size_t)(4*nvll);
//  const size_t fc_offset1 = thread_c * (size_t)(4*nvll) + 2*nvll;
//
//  const size_t fcr_offset0 = fc_offset0 + 4*thread_r;
//  const size_t fcr_offset1 = fc_offset1 + 4*thread_r;

  const size_t fcr_offset0 = 4*thread_r + thread_c * (size_t)(4*nvll);
  const size_t fcr_offset1 = 4*thread_r + thread_c * (size_t)(4*nvll) + 2*nvll;

  // Read the 8 floats.

  GemmOut_t* fvo = (GemmOut_t*)vo;

  const GemmOut_t f00 = fvo[fcr_offset0+0];
  const GemmOut_t f01 = fvo[fcr_offset0+1];
  const GemmOut_t f02 = fvo[fcr_offset0+2];
  const GemmOut_t f03 = fvo[fcr_offset0+3];

  const GemmOut_t f10 = fvo[fcr_offset1+0];
  const GemmOut_t f11 = fvo[fcr_offset1+1];
  const GemmOut_t f12 = fvo[fcr_offset1+2];
  const GemmOut_t f13 = fvo[fcr_offset1+3];

  // Apply the permutation:

  // [ f00  f10 ]  ->  [ f00  f02 ]
  // [ f01  f11 ]  ->  [ f01  f03 ]
  // [ f02  f12 ]  ->  [ f10  f12 ]
  // [ f03  f13 ]  ->  [ f11  f13 ]

  const GemmOut_t f00p = f00;
  const GemmOut_t f01p = f01;

  const GemmOut_t f02p = f10;
  const GemmOut_t f03p = f11;

  const GemmOut_t f10p = f02;
  const GemmOut_t f11p = f03;

  const GemmOut_t f12p = f12;
  const GemmOut_t f13p = f13;

  // Use "shifter" to move one value to upper half of mantissa.

  const double shifter = (((GMUInt32)1) << GM_TALLY1_MAX_VALUE_BITS);

  // Pack two 25-bit integers into mantissa of double.

  const double d00 = (double)f00p + (double)f02p * shifter;
  const double d01 = (double)f01p + (double)f03p * shifter;

  const double d10 = (double)f10p + (double)f12p * shifter;
  const double d11 = (double)f11p + (double)f13p * shifter;

  // Overwrite block with the new values.
  // All is isolated to a single thread, should be thread safe.

  const size_t dc_offset0 = thread_c * (size_t)(2*nvll);
  const size_t dc_offset1 = thread_c * (size_t)(2*nvll) + nvll;

  const size_t dcr_offset0 = dc_offset0 + 2*thread_r;
  const size_t dcr_offset1 = dc_offset1 + 2*thread_r;

  double* dvo = (double*)vo;

  dvo[dcr_offset0+0] = d00;
  dvo[dcr_offset0+1] = d01;

  dvo[dcr_offset1+0] = d10;
  dvo[dcr_offset1+1] = d11;
}

//-----------------------------------------------------------------------------
/// \brief Swizzle/cast values from cublas call into double complex format.
///
///        The cublas gemm poduces a matrix of scalars of 32 bit size
///        (int32 or float).  However the required format of the metrics
///        is a matrix of double complex values, with each double
///        containing two packed 25-bit integers.
///        This code does an in-place transformation from one to the other.

template<int TC_METHOD>
static void gm_tc_repair_metrics_(
  int nvll,
  int nvl,
  void* vo_ptr,
  TCBufs& tc_bufs,
  GMEnv* env) {

  GMInsist(env && vo_ptr);
  GMInsist(nvll >= 0);
  GMInsist(nvl >= 0);
  GMInsist(nvll <= nvl);

  const int nvll2 = nvll / 2;

  const int threadblocksize = 256;
  const int vll2_threadblocks = gm_ceil_i8(nvll2, threadblocksize);

  gm_tc_repair_metrics_kernel_<typename TCSelector<TC_METHOD>::GemmOut_t><<<
      dim3(vll2_threadblocks, nvl, 1),
      dim3(threadblocksize, 1, 1),
      0,
      env->stream_compute_>>>(nvl, nvll, nvll2, vo_ptr);

  GMEnv_cuda_last_call_succeeded(env);
}

//-----------------------------------------------------------------------------
/// \brief Use a standard GEMM to compute bitwise result: implementation.
///
///        This is the main function to perform the relevant
///        bitwise modified GEMM operation by use of standard GEMM
///        computations, typically using reduced precision arithmetic
///        and associated hardware features.
///
///        This is composed of three steps:
///        1. copy the input matrices into the required matrix format
///        2. apply the GEMM
///        3. adjust the results in-place to the required format.
///        To save on memory, this 3-step process is broken into
///        a sequence of steps as an outer loop.
///        All of these operations are pipelined in a (CUDA) execution
///        stream.
///

template<int TC_METHOD>
static void gm_tc_gemm_start_impl_(
  int m, int n, int k,
  void* dA, int ldda,
  void* dB, int lddb,
  void* dC, int lddc,
  TCBufs& tc_bufs,
  GMEnv* env) {

  GMInsist(ldda == k); // For our purposes, always true
  GMInsist(lddb == k);

  const int nvl = n;
  const int npvfl = k;
  const int I_max = m;
  const int I_max_dim = lddc;
  GMInsist(I_max <= I_max_dim);
  GMInsist(I_max_dim <= nvl);
  // nvll is the effective nvl for left matrix
  // only really need to compute up to I_max, but compute more to
  // satisfy divisibiulity requirements.
  const int nvll = I_max_dim;
  GMInsist((size_t)nvll == gm_gemm_size_required(nvll, env));

  const int num_steps = env->num_tc_steps;

  // Loop over steps of algorithm.
  for (int step_num = 0; step_num < num_steps; ++step_num) {

    // Select the block row of the left and right matrices for this step.
    const int pvfl_min = ((step_num+0) * npvfl) / num_steps;
    const int pvfl_max = ((step_num+1) * npvfl) / num_steps;
    const int npvfl_step = pvfl_max - pvfl_min;

    if (npvfl_step == 0) {  // empty block row
      continue;
    }

    // Convert the input matrices of packed bit values into matrices
    // of values of a type suitable for the GEMM.
    const bool left_matrix = false; // A
    const bool right_matrix = true; // B
    gm_tc_buf_write_<TC_METHOD>(left_matrix, I_max, I_max_dim, nvl, npvfl,
                     npvfl_step, pvfl_min, dA, tc_bufs, env);
    gm_tc_buf_write_<TC_METHOD>(right_matrix, I_max, I_max_dim, nvl, npvfl,
                     npvfl_step, pvfl_min, dB, tc_bufs, env);

    // Perform the GEMM for this pair of block rows; accumulate.
    gm_tc_solve_<TC_METHOD>(
      pvfl_min==0, nvll, nvl, npvfl_step, dA, dB, dC, tc_bufs, env);
  }

  // Revise the results of the GEMMs to be in the needed double complex format.
  gm_tc_repair_metrics_<TC_METHOD>(nvll, nvl, dC, tc_bufs, env);
}

//=============================================================================
// "PUBLIC" FUNCTIONS
//=============================================================================

//-----------------------------------------------------------------------------
/// \brief Divisibility requirement for GEMM.

size_t gm_gemm_size_required(size_t size_requested, GMEnv* const env) {
  GMInsist(env);

  const bool need_divisible_by_4 = env->tc;

  return need_divisible_by_4 ? gm_ceil_i8(size_requested, 4)*4 : size_requested;
}

//-----------------------------------------------------------------------------
/// \brief Use a standard GEMM to compute CoMet metrics bitwise result.

void gm_tc_gemm_start(int m, int n, int k,
                      void* dA, int ldda,
                      void* dB, int lddb,
                      void* dC, int lddc,
                      TCBufs& tc_bufs,
                      GMEnv* env) {
  GMInsist(dA && dB && dC && env);
  GMInsist(m >= 0 && n >= 0 && k >= 0);
  GMInsist(ldda >= 0 && lddb >= 0 && lddc >= 0);
  GMInsist(k <= ldda);
  GMInsist(k <= lddb);
  GMInsist(m <= lddc);
  GMInsist(env->tc >= 1 && env->tc < GM_NUM_TC_METHOD);
  GMInsist(GMEnv_metric_type(env) == GM_METRIC_TYPE_CCC);
  GMInsist(GMEnv_compute_method(env) == GM_COMPUTE_METHOD_GPU);
  // Ensure tensor core hardware is available.
  GMInsistInterface(env, gm_gpu_compute_capability() >= 700 &&
                    "TC option unavailable for this platform/build.");

  // Select required template function instance.

  switch (env->tc) {
    // --------------
    case GM_TC_METHOD_INT8: {
      gm_tc_gemm_start_impl_<GM_TC_METHOD_INT8>(
        m, n, k, dA, ldda, dB, lddb, dC, lddc, tc_bufs,  env);
    } break;
    // --------------
    case GM_TC_METHOD_FLOAT16: {
      gm_tc_gemm_start_impl_<GM_TC_METHOD_FLOAT16>(
        m, n, k, dA, ldda, dB, lddb, dC, lddc, tc_bufs,  env);
    } break;
    // --------------
    default:
      GMInsist(false && "Invalid tc type.");
  } // switch
}

//-----------------------------------------------------------------------------
/// \brief Initialize TCBufs object by allocating memory etc.

void gm_tc_bufs_malloc(int num_vector_local,
                       int num_field_local,
                       int num_packedval_field_local,
                       TCBufs& tc_bufs,
                       GMEnv* env) {
  GMInsist(env);
  GMInsist(num_vector_local >= 0);
  GMInsist(num_packedval_field_local >= 0);
  GMInsist(!tc_bufs.tc_buf_left);
  GMInsist(!tc_bufs.tc_buf_right);

  if (!env->tc) {
    return;
  }

  if (GMEnv_metric_type(env) != GM_METRIC_TYPE_CCC) {
    return;
  }

  GMInsistInterface(env, gm_gpu_compute_capability() >= 700 &&
                    "TC option unavailable for this platform/build.");

  // Calculate sizes.

  const size_t nvl = num_vector_local;
  const size_t npvfl = num_packedval_field_local;
  const size_t npvfl_step_max = gm_ceil_i8(npvfl, env->num_tc_steps);

  const int sizeof_gemm_in_t =
     env->tc == GM_TC_METHOD_INT8 ?
       sizeof(typename TCSelector<GM_TC_METHOD_INT8>::GemmIn_t) :
     env->tc == GM_TC_METHOD_FLOAT16 ?
       sizeof(typename TCSelector<GM_TC_METHOD_FLOAT16>::GemmIn_t) :
     0;
  GMInsist(GM_NUM_TC_METHOD == 3); // this code must be updated if new method

  const size_t nvlX2 = nvl * 2;

  tc_bufs.tc_buf_size = nvlX2 * (npvfl_step_max * 64) * sizeof_gemm_in_t;
  tc_bufs.tc_buf_size = tc_bufs.tc_buf_size ? tc_bufs.tc_buf_size : 1;

  // Allocate buffers.

  hipMalloc(&tc_bufs.tc_buf_left, tc_bufs.tc_buf_size);
  GMEnv_cuda_last_call_succeeded(env);
  env->gpu_mem += tc_bufs.tc_buf_size;
  env->gpu_mem_max = gm_max_i8(env->gpu_mem_max, env->gpu_mem);

  hipMalloc(&tc_bufs.tc_buf_right, tc_bufs.tc_buf_size);
  GMEnv_cuda_last_call_succeeded(env);
  env->gpu_mem += tc_bufs.tc_buf_size;
  env->gpu_mem_max = gm_max_i8(env->gpu_mem_max, env->gpu_mem);

  // Set up cublas handle.

  hipblasStatus_t status_cb = hipblasCreate(&tc_bufs.cublas_handle);
  GMInsist(status_cb == HIPBLAS_STATUS_SUCCESS);

  status_cb = hipblasSetStream(tc_bufs.cublas_handle, env->stream_compute_);
  GMInsist(status_cb == HIPBLAS_STATUS_SUCCESS);

#if __CUDACC_VER_MAJOR__ >= 9
  status_cb = hipblasSetMathMode(tc_bufs.cublas_handle, HIPBLAS_TENSOR_OP_MATH);
  GMInsist(status_cb == HIPBLAS_STATUS_SUCCESS);
#endif
}

//-----------------------------------------------------------------------------
/// \brief Terminate TCBufs object by deallocating memory etc.

void gm_tc_bufs_free(TCBufs& tc_bufs,
                     GMEnv* env) {
  GMInsist(env);
  GMInsist((tc_bufs.tc_buf_left != 0) == (tc_bufs.tc_buf_right != 0));

  if (!tc_bufs.tc_buf_left) {
    return;
  }

  // Free buffers.

  hipFree(tc_bufs.tc_buf_left);
  GMEnv_cuda_last_call_succeeded(env);
  tc_bufs.tc_buf_left = NULL;
  env->gpu_mem -= tc_bufs.tc_buf_size;

  hipFree(tc_bufs.tc_buf_right);
  GMEnv_cuda_last_call_succeeded(env);
  tc_bufs.tc_buf_right = NULL;
  env->gpu_mem -= tc_bufs.tc_buf_size;

  // Free cublas handle.

  hipblasStatus_t status_cb = hipblasDestroy(tc_bufs.cublas_handle);
  GMInsist(status_cb == HIPBLAS_STATUS_SUCCESS);
}

//-----------------------------------------------------------------------------
